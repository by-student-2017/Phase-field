/* 3D semi-implicit spectral
  phase-field crystal code */

#include <stdio.h>
#include <stdlib.h> //rand()
#include <math.h> //mod() and -lm
#include <time.h>

//#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cu_Complex cufftComplex;

//----- ----- ----- ----- ----- ----- -----
void prepare_fft_3d(int Nx, int Ny, int Nz, 
	float dx, float dy, float dz,
	float *kx, float *ky, float *kz, 
	float *k2, float *k4);
//----- ----- -----
void write_vtk_grid_values_3D(int nx, int ny, int nz, 
	float dx, float dy, float dz,
	int istep, float *data1);
//----- ----- ----- ----- ----- ----- -----

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_semi_implicit_time_integration(
	int   Nx,
	int   Ny,
	int   Nz,
	float dtime,
	float tempr,
	float *k2_d,
	float *k4_d,
	hipfftComplex *f_den_d,
	hipfftComplex *f_den3_d
){
	int j, jx, jy, jz;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	jz = blockDim.z*blockIdx.z + threadIdx.z; //<-GPU | CPU -> for(jz=0; jz<nz; jz++){
	j  = (jz*Ny + jy)*Nx + jx; //j = nx*ny*jz + nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	float denom;
	//
	//calculate the value of denominator in Eq.7.10 at every grid points
	denom = 1.0 + dtime*k2_d[j]*(tempr+1.0-2.0*k2_d[j]+k4_d[j]);
	//----- ----- -----
	//calculate the value of phi^(t+1) from Fourier space to real space (inverse FFT transformation)
	f_den_d[j].x = ( f_den_d[j].x - dtime*k2_d[j]*f_den3_d[j].x )/denom; //real part
	f_den_d[j].y = ( f_den_d[j].y - dtime*k2_d[j]*f_den3_d[j].y )/denom; //imaginary part
	
	/* Note: cufftComplex changed between CUDA 1.0 and 1.1.
	dout[idx].x =  d_signal[idx].y; <- dout[idx][0] = d_signal[idx][1];
	dout[idx].y = -d_signal[idx].x; <- dout[idx][1] = d_signal[idx][0]*(-1.0);
	Ref: https://forums.developer.nvidia.com/t/using-cufftcomplex-type-inside-a-kernel-does-it-work/4039 */
}

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	start = clock();
	
	FILE *out1=fopen("final_conf.out","w");
	FILE *out2=fopen("energy.out","w");
	
	//simulation cell parameters
	int Nx=64; //Number of grid points in the x-direction
	int Ny=64; //Number of grid points in the y-direction
	int Nz=2; //Number of grid points in the y-direction
	int NxNyNz=Nx*Ny*Nz; //Total number of grid points in the simulation cell
	
	int BSX=8; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	int BSY=8; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	int BSZ=2; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	
	//The value of pi
	float pix=4.0*atan(1.0);
	
	//The distance between two grid points in x,y,z-direction
	float dx=pix/4.0;
	float dy=pix/4.0;
	float dz=pix/4.0;
	
	//time integration parameters
	//int nstep=20000; //for pfc_3d_v1
	int nstep=200000; //for pfc_3d_v2
	int nprint=5000;
	float dtime=0.05;
	
	//material specific parameters
	//float den0=-0.085; //average density for pfc_3d_v1 (stripe phase)
	float den0=-0.285; //average density for pfc_3d_v2 (triangular phase)
	float tempr=-0.25; //temperature (T-Tm)/Tm, Tm=melting point
	//float tempr0=0.0; //positive value, tempr=tempr+tempr0/isteps;
	float noise=den0*1e-2; //Noise term to modulate the initial density field
	
	int ii; //ii=(i*Ny+j)*Nz+k;
	
	//----- ----- ----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny;
	//
	hipfftComplex *den_d, *f_den_d;
	hipMalloc((void**)&den_d,    sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&f_den_d,  sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *den3_d, *f_den3_d;
	hipMalloc((void**)&den3_d,   sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&f_den3_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *ff_d, *f_ff_d;
	hipMalloc((void**)&ff_d,     sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&f_ff_d,   sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftHandle plan, iplan;
	hipfftPlan3d(&plan,  Nx, Ny, Nz, HIPFFT_C2C);
	hipfftPlan3d(&iplan, Nx, Ny, Nz, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- -----
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float kz[Nz];
	float *kz = (float *)malloc(sizeof(float)*( Nz ));
	//float k2[Nx][Ny][Nz];
	float *k2 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float k4[Nx][Ny][Nz];
	float *k4 = (float *)malloc(sizeof(float)*( NxNyNz ));
	
	//prepare fft (output: kx,ky,kz,k2,k4)
	prepare_fft_3d(Nx,Ny,Nz,dx,dy,dz,kx,ky,kz,k2,k4); //get FFT coefficients
	//----- ----- ----- ----- ----- -----
	
	//----- for cufft
	float *k2_d, *k4_d;
	k2_d  = (float *)malloc(NxNyNz*sizeof(float));
	k4_d  = (float *)malloc(NxNyNz*sizeof(float));
	hipMalloc((void**)&k2_d ,NxNyNz*sizeof(float));
	hipMalloc((void**)&k4_d ,NxNyNz*sizeof(float));
	hipMemcpy(k2_d,k2,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //k2 = k2_h
	hipMemcpy(k4_d,k4,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //k4 = k4_h
	//----- ----- ----- -----
	
	//----- ----- ----- ----- ----- -----
	float _Complex *denc   = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *den3c  = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *f_den  = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *f_ff   = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *ffc    = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	//----- ----- ----- ----- ----- -----
	float *den = (float *)malloc(sizeof(float)*( NxNyNz ));
	float *ff  = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- -----
	
	//if infile=1 read input from file
	int infile = 0;
	FILE *in1;
	int mx,my,mz;
	if(infile==1){
		//open input file
		in1=fopen("g3_3r.inp","r");
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ii=(i*Ny+j)*Nz+k;
					fscanf(in1,"%5d %5d %5d %lf",&mx,&my,&mz,&den[ii]);
					denc[ii] = den[ii];
				}
			}
		}
	}else{
		//initialize density
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ii=(i*Ny+j)*Nz+k;
					den[ii] = den0 + noise*(0.5-(float)rand()/RAND_MAX);
					denc[ii] = den[ii];
				}
			}
		}
	}
	
	//----- ----- ----- ----- ----- -----
	float energy;
	//float ss2[Nx][Ny][Nz];
	float *ss2 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ss4[Nx][Ny][Nz];
	float *ss4 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- -----
	
	int bsx=BSX, bsy=BSY, bsz=BSZ;     //Number of threads
	dim3 blocks(Nx/bsx,Ny/bsy,Nz/bsz); //nx*ny*nz = blocks * threads
	dim3 threads(bsx,bsy,bsz);         //bsx*bsy*bsz <= 1024
	
	//evolve (evolve microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//tempr = tempr + tempr0/istep;
		
		hipMemcpy(den_d,denc,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //denc = denc_h
		
		//take current density filed from real space to Fourier space (forward FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, den_d, f_den_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		//calculate the nonlinear term, phi^3, in Eq.7.10
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ii=(i*Ny+j)*Nz+k;
					den3c[ii]=denc[ii]*denc[ii]*denc[ii];
				}
			}
		}
		hipMemcpy(den3_d,den3c,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //den3c = den3c_h
		
		//take the value of phi^3 from real space to Fourier space (forward FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, den3_d, f_den3_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- -----  on cuda
		Kernel_semi_implicit_time_integration<<<blocks, threads>>>(Nx,Ny,Nz,
			dtime,tempr,
			k2_d,k4_d,
			f_den_d,f_den3_d);
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- -----
		
		//bring back the values of phi^(t+1) from Fourier space to real space (inverse FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, f_den_d, den_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipMemcpy(denc,den_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //denc = denc_h
		
		//print results
		//if print frequency is reached, output the results to file
		if(fmod(istep,nprint)==0){
		
			printf("done step: %5d \n", istep);
			
			//energy calculation
			//calculate the free energy distribution, Eq.7.6
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ii=(i*Ny+j)*Nz+k;
						den[ii] = ( __real__ denc[ii] )/(NxNyNz);
						ss2[ii]=den[ii]*den[ii];
						ss4[ii]=ss2[ii]*ss2[ii];
					}
				}
			}
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ii=(i*Ny+j)*Nz+k;
						f_ff[ii] = 0.5*f_den[ii]*(1.0-2.0*k2[ii]+k4[ii]);
					}
				}
			}
			hipMemcpy(f_ff_d,f_ff,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //f_ff = f_ff_h
			
			//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
			hipfftExecC2C(iplan, f_ff_d, ff_d, HIPFFT_BACKWARD); //IFFT
			hipDeviceSynchronize();
			//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
			hipMemcpy(ffc,ff_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //ffc = ffc_h
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ii=(i*Ny+j)*Nz+k;
						ff[ii] = (( __real__ ffc[ii] )/(NxNyNz)) * den[ii]
						+ 0.5*tempr*ss2[ii]
						+ 0.25*ss4[ii];
					}
				}
			}
			
			//integrate the free energy field
			energy = 0.0;
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ii=(i*Ny+j)*Nz+k;
						energy = energy + ff[ii];
					}
				}
			}
			
			//average free energy density
			energy = energy/(NxNyNz);
			
			//print the average free energy density value to file
			fprintf(out2, "%d %14.6e \n",istep, energy);
			
			//output the results in vtk file format for contour plots to be viewed by using paraview
			write_vtk_grid_values_3D(Nx,Ny,Nz,dx,dy,dz,istep,den);
			
		}
		
		//if intermediate configuration files are required, print the density field to file
		if(istep==nstep){
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ii=(i*Ny+j)*Nz+k;
						fprintf(out1,"%5d %5d %5d %14.6e \n",i,j,k,den[ii]);
					}
				}
			}
		}
		
		//for recycle
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ii=(i*Ny+j)*Nz+k;
					denc[ii] = denc[ii]/(NxNyNz);
				}
			}
		}
		
	}//end of time step (evolve,for)
	
	//calculate the execution time and print it
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %lf \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(den_d);
	hipFree(f_den_d);
	//
	hipFree(den3_d);
	hipFree(f_den3_d);
	//
	hipFree(f_ff_d);
	hipFree(ff_d);
	//
	hipFree(k2_d);
	hipFree(k4_d);
	//----- ----- ----- ----- ----- -----
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(denc);
	free(den3c);
	free(f_den);
	free(f_ff);
	free(ffc);
	free(den);
	free(ff);
	//----- ----- ----- ----- ----- -----
	fclose(out1);
	fclose(out2);
	//----- ----- ----- ----- ----- -----
}
