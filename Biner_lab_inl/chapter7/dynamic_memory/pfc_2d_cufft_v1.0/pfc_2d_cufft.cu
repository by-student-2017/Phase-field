/* 2D semi-implicit spectral
  phase-field crystal code */

#include <stdio.h>
#include <stdlib.h> //rand()
#include <math.h> //mod() and -lm
#include <time.h>

//#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cu_Complex cufftComplex;

void prepare_fft_2d(int Nx, int Ny, 
	float dx, float dy,
	float *kx, float *ky, 
	float *k2, float *k4);

void write_vtk_grid_values_2D(int nx, int ny, 
	float dx, float dy,
	int istep, float *data1);

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	start = clock();
	
	FILE *out1=fopen("final_conf.out","w");
	FILE *out2=fopen("energy.out","w");
	
	//simulation cell parameters
	int Nx=64;
	int Ny=64;
	//int NxNy=Nx*Ny; //Total number of grid points in the simulation cell
	
	//Number of threads, 2^n=<32, BS*BS*1 <= 1024
	int BS=32;
	
	//The value of pi
	float pix=4.0*atan(1.0);
	
	//The distance between two grid points in x,y,z-direction
	float dx=pix/4.0;
	float dy=pix/4.0;
	
	//time integration parameters
	int nstep=200000;
	int nprint=2000;
	float dtime=0.05;
	
	//material specific parameters
	//float den0=-0.085; //average density for pfc_3d_v1 (stripe phase)
	float den0=-0.285; //average density for pfc_3d_v2 (triangular phase)
	float tempr=-0.25; //temperature (T-Tm)/Tm, Tm=melting point
	//float tempr0=0.0; //positive value, tempr=tempr+tempr0/isteps;
	float noise=den0*1e-2; //Noise term to modulate the initial density field
	
	int ii;
	
	//----- ----- ----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny;
	//
	hipfftComplex *den_d, *f_den_d;
	hipMalloc((void**)&den_d,    sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&f_den_d,  sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftComplex *den3_d, *f_den3_d;
	hipMalloc((void**)&den3_d,   sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&f_den3_d, sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftComplex *ff_d, *f_ff_d;
	hipMalloc((void**)&ff_d,     sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&f_ff_d,   sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftHandle plan, iplan;
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float k2[Nx][Ny];
	float *k2 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float k4[Nx][Ny];
	float *k4 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//prepare fft (output: kx,ky,kz,k2,k4)
	prepare_fft_2d(Nx,Ny,dx,dy,kx,ky,k2,k4); //get FFT coefficients
	
	float _Complex *denc   = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *den3c  = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *Nonxc  = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *f_den  = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *f_den3 = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *f_ff   = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *ffc    = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float *den = (float *)malloc(sizeof(float)*( Nx*Ny ));
	float *ff  = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//if infile=1 read input from file
	int infile = 0;
	FILE *in1;
	int mx,my;
	if(infile==1){
		//open input file
		in1=fopen("g3_2r.inp","r");
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				fscanf(in1,"%5d %5d %f",&mx,&my,&den[ii]);
				denc[ii] = den[ii];
			}
		}
	}else{
		//initialize density
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				den[ii] = den0 + noise*(0.5-(float)rand()/RAND_MAX);
				denc[ii] = den[ii];
			}
		}
	}
	
	//float Linx[Nx][Ny];
 	float *Linx  = (float *)malloc(sizeof(float)*( Nx*Ny ));
 	//float denom[Nx][Ny];
 	float *denom = (float *)malloc(sizeof(float)*( Nx*Ny ));
 	//
 	float energy;
	//float ss2[Nx][Ny];
	float *ss2 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ss4[Nx][Ny];
	float *ss4 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(Nx/bs,Ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	//evolve (evolve microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//tempr = tempr + tempr0/istep;
		
		hipMemcpy(den_d,denc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //denc = denc_h
		
		//take current density filed from real space to Fourier space (forward FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, den_d, f_den_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipMemcpy(f_den, f_den_d, Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //f_den = f_den_h
		
		//calculate the value of denominator in Eq.7.10 at every grid points
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				Linx[ii]=-k2[ii]*(tempr+1.0-2.0*k2[ii]+k4[ii]);
				denom[ii]=1.0-dtime*Linx[ii];
			}
		}
		
		//calculate the nonlinear term, phi^3, in Eq.7.10
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				den3c[ii]=denc[ii]*denc[ii]*denc[ii];
			}
		}
		hipMemcpy(den3_d,den3c,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //den3c = den3c_h
		
		//take the value of phi^3 from real space to Fourier space (forward FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, den3_d, f_den3_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipMemcpy(f_den3,f_den3_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //f_den3 = f_den3_h
		
		//calculate the value of phi^(t+1) from Fourier space to real space (inverse FFT transformation)
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				Nonxc[ii]=-k2[ii]*f_den3[ii];
				f_den[ii]=(f_den[ii]+dtime*Nonxc[ii])/denom[ii];
			}
		}
		hipMemcpy(f_den_d,f_den,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //f_den = f_den_h
		
		//bring back the values of phi^(t+1) from Fourier space to real space (inverse FFT transformation)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, f_den_d, den_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipMemcpy(denc,den_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //denc = denc_h
		
		//print results
		//if print frequency is reached, output the results to file
		if(fmod(istep,nprint)==0){
		
			printf("done step: %5d \n", istep);
			
			//energy calculation
			//calculate the free energy distribution, Eq.7.6
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					ii=i*Ny+j;
					den[ii] = ( __real__ denc[ii] )/(Nx*Ny);
					ss2[ii]=den[ii]*den[ii];
					ss4[ii]=ss2[ii]*ss2[ii];
				}
			}
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					ii=i*Ny+j;
					f_ff[ii]=0.5*f_den[ii]*(1.0-2.0*k2[ii]+k4[ii]);
				}
			}
			hipMemcpy(f_ff_d,f_ff,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //f_ff = f_ff_h
			
			//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
			hipfftExecC2C(iplan, f_ff_d, ff_d, HIPFFT_BACKWARD); //IFFT
			hipDeviceSynchronize();
			//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
			hipMemcpy(ffc,ff_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //ffc = ffc_h
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					ii=i*Ny+j;
					ff[ii] = (( __real__ ffc[ii] )/(Nx*Ny)) * den[ii]
					+ 0.5*tempr*ss2[ii]
					+ 0.25*ss4[ii];
				}
			}
			
			//integrate the free energy field
			energy = 0.0;
			
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					ii=i*Ny+j;
					energy = energy + ff[ii];
				}
			}
			
			//average free energy density
			energy = energy/(Nx*Ny);
			
			//print the average free energy density value to file
			fprintf(out2, "%d %14.6e \n",istep, energy);
			
			//output the results in vtk file format for contour plots to be viewed by using paraview
			write_vtk_grid_values_2D(Nx,Ny,dx,dy,istep,den);
			
		}
		
		//if intermediate configuration files are required, print the density field to file
		if(istep==nstep){
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					ii=i*Ny+j;
					fprintf(out1,"%5d %5d %14.6e \n",i,j,den[ii]);
				}
			}
		}
		
		//for recycle
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				denc[ii] = denc[ii]/(Nx*Ny);
			}
		}
		
	}//end of time step (evolve,for)
	
	//calculate the execution time and print it
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %f \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(den_d);
	hipFree(f_den_d);
	//
	hipFree(den3_d);
	hipFree(f_den3_d);
	//
	hipFree(f_ff_d);
	hipFree(ff_d);
	//----- ----- ----- ----- ----- -----
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(denc);
	free(den3c);
	free(Nonxc);
	free(f_den);
	free(f_den3);
	free(f_ff);
	free(ffc);
	free(den);
	free(ff);
	//----- ----- ----- ----- ----- -----
	fclose(out1);
	fclose(out2);
	//----- ----- ----- ----- ----- -----
}
