/* 2D semi-implicit spectral phase-field code
   for solving Cahn-Hilliard equation */

/* This program solves conserved phase-field equation with
   Fourier spactral method by taking into account the effects of
   elastic inhomogeneities and lattice defects, based on
   solution of stress-strain fields ith Green's tensor and
   Fourier transformations. the time integration is
   carried out by using semi-implicit time machining scheme. */

#include <stdio.h>
#include <stdlib.h> //rand() and malloc
#include <math.h> //mod() and -lm
#include <time.h>

//#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
#include <hip/hip_complex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cu_Complex cufftComplex;

//----- ----- ----- ----- ----- ----- -----
void micro_ch_pre_2d(int Nx, int Ny, float c0, float *con);
//----- ----- -----
void prepare_fft_2d(int Nx, int Ny, 
	float dx, float dy,
	float *kx, float *ky, 
	float *k2, float *k4);
//----- ----- -----
float free_energy_ch_2d(float con_ij);
//----- ----- -----
float calculate_energy_2d(int Nx, int Ny, float *con, float grad_coef);
//----- ----- -----
void write_vtk_grid_values_2D(int nx, int ny, 
	float dx, float dy,
	int istep, float *data1);
//----- ----- ----- ----- ----- ----- -----

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_semi_implicit_time_integration(
	int   Nx,
	int   Ny,
	float dtime,
	float coefA,
	float mobility,
	float grad_coef,
	float *k2_d,
	float *k4_d,
	hipfftComplex *conk_d,
	hipfftComplex *dfdconk_d
){
	int j, jx, jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	j  = Nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	float denom;
	//
	denom = 1.0 + dtime*coefA*mobility*grad_coef*k4_d[j];
	conk_d[j].x = ( conk_d[j].x - (dtime*mobility*k2_d[j]*dfdconk_d[j].x) )/denom; //real part
	conk_d[j].y = ( conk_d[j].y - (dtime*mobility*k2_d[j]*dfdconk_d[j].y) )/denom; //imaginary part
	
	/* Note: cufftComplex changed between CUDA 1.0 and 1.1.
	dout[idx].x =  d_signal[idx].y; <- dout[idx][0] = d_signal[idx][1];
	dout[idx].y = -d_signal[idx].x; <- dout[idx][1] = d_signal[idx][0]*(-1.0);
	Ref: https://forums.developer.nvidia.com/t/using-cufftcomplex-type-inside-a-kernel-does-it-work/4039 */
}

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	//(Get initial wall clock time beginning of the execution)
	start = clock();
	
	//simulation cell parameters
	int Nx=64;
	int Ny=64;
	//int NxNy=Nx*Ny; //Total number of grid points in the simulation cell
	
	//Number of threads, 2^n=<32, BS*BS*1 <= 1024
	int BS=32;
	
	//open an output file for writing total bulk energy values
	FILE *out2=fopen("time_energy.out","w");
	
	//The distance between two grid points in x,y-direction
	float dx=1.0; // [nm] unit ?
	float dy=1.0; // [nm] unit ?
	
	//time integration parameters
	int nstep=2000; //Number of time steps
	int nprint=50;  //Print frequency to write the results to file
	float dtime=1.0e-2; //Time increment for numerical integration
	float ttime=0.0;    //Total time
	float coefA=1.0;
	
	//material specific parameters
	
	//Initial concentrations of alloying elements
	float c0=0.40;       //Initial concentraion (20%Cr-containing Fe-Cr alloy
	float mobility=1.0;  //The value of mobility coefficient (dimensionless)
	float grad_coef=0.5; //The value of gradient energy coefficients [J(nm)^2/mol]
	
	float energy=0.0;
	
	int ij; //ij=(i*Ny+j);
	
	//----- ----- ----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny;
	//
	hipfftComplex *con_d, *dfdcon_d;
	hipMalloc((void**)&con_d,     sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdcon_d,  sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftComplex *conk_d, *dfdconk_d;
	hipMalloc((void**)&conk_d,    sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdconk_d, sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftHandle plan, iplan;
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//----- prepare microstructure
	float *con = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	micro_ch_pre_2d(Nx,Ny,c0,con);//Initialize microstructure
	//----- ----- ----- -----
	
	//----- prepare fft (output: kx,ky,kz,k2,k4)
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float k2[Nx][Ny];
	float *k2 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float k4[Nx][Ny];
	float *k4 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//prepare fft (output: kx,ky,kz,k2,k4)
	prepare_fft_2d(Nx,Ny,dx,dy,kx,ky,k2,k4); //Calculate coefficients of Fourier transformation
	//----- ----- ----- -----
	
	//----- for cufft
	float *k2_d, *k4_d;
	k2_d  = (float *)malloc(Nx*Ny*sizeof(float));
	k4_d  = (float *)malloc(Nx*Ny*sizeof(float));
	hipMalloc((void**)&k2_d ,Nx*Ny*sizeof(float));
	hipMalloc((void**)&k4_d ,Nx*Ny*sizeof(float));
	hipMemcpy(k2_d,k2,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k2 = k2_h
	hipMemcpy(k4_d,k4,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k4 = k4_h
	//----- ----- ----- -----
	
	float _Complex *dfdconc = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *conc    = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(Nx/bs,Ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	//evolve (Time evolution of microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//Update the total time
		ttime=ttime+dtime;
		
		//derivative of free energy
		//Calculate the derivative of elastic energy
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ij=i*Ny+j;
				dfdconc[ij] = free_energy_ch_2d(con[ij]);
				//----- ------ ------ ------
				conc[ij] = con[ij];
			}
		}
		
		hipMemcpy(dfdcon_d,dfdconc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //dfdconc = dfdconc_h
		hipMemcpy(con_d,conc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //conc = conc_h
		
		/* Take the values of concentration, derivative of free energy and
		   derivative of elastic energy from real space to Fourier space (forward FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, con_d, conk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, dfdcon_d, dfdconk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		/* Semi-implicit time integration of concentration field at
		   Fourier space (Eq.5.14) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- -----  on cuda
		Kernel_semi_implicit_time_integration<<<blocks, threads>>>(Nx,Ny,
			dtime,coefA,mobility,grad_coef,
			k2_d,k4_d,
			conk_d,dfdconk_d);
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- -----
		
		/* Take concentration field from Fourier space back to
		   real space (inverse FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, conk_d, con_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(conc,con_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //conc = conc_h
		
		//for small deviations
		// For small deviations from max and min values, reset the limits
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ij=i*Ny+j;
				//----- ----- ----- -----
				//con[ij] =  con[ij]/(Nx*Ny);
				con[ij] = ( __real__ conc[ij] )/(Nx*Ny);
				//con[ij] =  creal(conc[ij])/(Nx*Ny); //For #include <_Complex.h>
				//----- ----- ----- -----
				if(con[ij]>=0.9999){
					con[ij]=0.9999;
				}
				if(con[ij]<=0.0001){
					con[ij]=0.0001;
				}
				//----- ----- ----- -----
			}
		}
		
		//print results
		/* If print frequency is reached, output the results to file */
		if(fmod(istep,nprint)==0){
			/* Calculate the total bulk energy and print the result to
			   time_energy.out file for 2D plots */
			energy = calculate_energy_2d(Nx,Ny,con,grad_coef);
			
			//print the average free energy density value to file
			fprintf(out2, "%14.6e %14.6e \n",ttime, energy);
			
			//write vtk file
			/* Write the results in vtk format for contour plots
			   to be viewed by using Paraview */
			write_vtk_grid_values_2D(Nx,Ny,dx,dy,istep,con);
			
			printf("done step: %5d, time: %f \n",istep,ttime*(mobility/(dx*dx)));
			/* The quantities having the dimension of distance were normalized with the
			   magnitude of the Burger's vector, the  quantities having the dimension of
			   energy were normalized with RT, and the time t was normalized with M/(dx^2).
			   The initial concentration was modulated by setting the noise term to
			   0.001 in function micro_ch_pre_2d.c */
		}
		
	}//end of time step (evolve,for)
	
	//calculate the execution time and print it
	/* Calculate the compute time and print it to screen */
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %lf \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(con_d);
	hipFree(dfdcon_d);
	//
	hipFree(conk_d);
	hipFree(dfdconk_d);
	//
	hipFree(k2_d);
	hipFree(k4_d);
	//----- ----- ----- ----- ----- -----
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(con);
	//
	free(conc);
	free(dfdconc);
	//----- ----- ----- ----- ----- -----
}
