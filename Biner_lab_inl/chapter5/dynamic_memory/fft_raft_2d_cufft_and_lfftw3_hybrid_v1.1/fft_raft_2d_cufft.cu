#include "hip/hip_runtime.h"
/* 2D semi-implicit spectral phase-field code 
  for solving precipitation under stress */

/* This program solves Cahn-Hilliard phase-field equation with
   semi-implicit Fourier spectral method by 
   taking into account the effects of elastic inhomogeneities and 
   applied stresses based on solution of stress-strain fields with
   Green's tensor and Fourier transformations.
     The time integration is carried out by using semi-implicit
   time marching scheme. */

#include <stdio.h>
#include <stdlib.h> //rand() and malloc
#include <math.h> //mod() and -lm
#include <time.h>

#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <hip/hip_complex.h>
#define _Complex_I (1.0iF)
#define I i
#undef i
#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <hip/hip_runtime.h> or
  #include "hip/hip_runtime.h"
  #include "" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float hipfftReal;
//typedef hipComplex hipfftComplex;

//----- ----- ----- ----- ----- ----- -----
void micro_ch_pre_2d(int Nx, int Ny, float c0, float *con);
//----- ----- -----
void prepare_fft_2d(int Nx, int Ny, 
	float dx, float dy,
	float *kx, float *ky, 
	float *k2, float *k4);
//----- ----- -----
void green_tensor_2d(int Nx, int Ny,
	float *kx, float *ky,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *tmatx);
//----- ----- -----
float free_energy_ch_2d(float con_ij);
//----- ----- -----
void solve_elasticity_2d(int Nx, int Ny,
	float *tmatx,
	fftw_complex *s11, fftw_complex *s22, fftw_complex *s12,
	fftw_complex *e11, fftw_complex *e22, fftw_complex *e12,
	float *ed11, float *ed22, float *ed12,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea,
	float ei0,
	float *con, float *delsdc);
//----- ----- -----
void write_vtk_grid_values_2D(int nx, int ny, 
	float dx, float dy,
	int istep, float *data1);
//----- ----- ----- ----- ----- ----- -----

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_semi_implicit_time_integration(
	int   Nx,
	int   Ny,
	float dtime,
	float coefA,
	float mobility,
	float grad_coef,
	float *k2_d,
	float *k4_d,
	hipfftComplex *conk_d,
	hipfftComplex *dfdconk_d,
	hipfftComplex *delsdck_d
){
	int j, jx, jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	j  = Ny*jx + jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	float denom;
	//
	denom = 1.0 + dtime*coefA*mobility*grad_coef*k4_d[j];
	conk_d[j].x = ( conk_d[j].x - (dtime*mobility*k2_d[j]*(dfdconk_d[j].x + delsdck_d[j].x)) )/denom;
	//conk_d[j].y = ( conk_d[j].y - (dtime*mobility*k2_d[j]*(dfdconk_d[j].y + delsdck_d[j].y)) )/denom;
}

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	//(Get initial wall clock time beginning of the execution)
	start = clock();
	
	//simulation cell parameters
	int Nx=256;
	int Ny=256;
	
	//Number of threads, 2^n=<32, BS*BS*1 <= 1024
	int BS=32;
	
	//Total number of grid points in the simulation cell
	//int NxNy=Nx*Ny;
	
	//The distance between two grid points in x,y-direction
	float dx=1.0; // [nm] unit ?
	float dy=1.0; // [nm] unit ?
	
	//time integration parameters
	int nstep=5000; //Number of time steps
	int nprint=25;  //Print frequency to write the results to file
	float dtime=5.0e-2; //Time increment for numerical integration
	float ttime=0.0;    //Total time
	float coefA=1.0;
	
	//material specific parameters
	
	//Initial concentrations of alloying elements
	float c0=0.40;       //Initial concentraion
	float mobility=1.0;  //The value of mobility coefficient (dimensionless)
	float grad_coef=0.5; //The value of gradient energy coefficients [J(nm)^2/mol]
	
	//elastic constants
	//Elastic constants of matrix phase
	float cm11=1400.0;
	float cm12= 600.0;
	float cm44= 400.0;
	//
	//Elastic constants of second phase
	float cp11=2.0*cm11;
	float cp12=2.0*cm12;
	float cp44=2.0*cm44;
	
	//eigen strains
	float ei0=0.01; //Maginitude of eigenstrains
	
	//Applied strains
	float ea[3]; //Magnitude of applied strains
	ea[0]=0.00;
	ea[1]=0.01;
	ea[2]=0.00;
	
	int ii;
	
	//----- ----- ----- ----- ----- -----
	const int fftsizex = Nx, fftsizey = Ny;
	//
	hipfftComplex *con_d, *dfdcon_d, *delsdc_d;
	hipMalloc((void**)&con_d,     sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdcon_d,  sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&delsdc_d,  sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftComplex *conk_d, *dfdconk_d, *delsdck_d;
	hipMalloc((void**)&conk_d,    sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdconk_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&delsdck_d, sizeof(hipfftComplex)*Nx*Ny);
	//
	//hipfftComplex *conc_d;
	//hipMalloc((void**)&conc_d,    sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftHandle plan, iplan;
	//hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_R2C);
	//hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2R);
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//----- ----- ----- -----fftw3
	fftw_complex *s11, *s22, *s12;
	 s11 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 s22 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 s12 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//----- ----- ----- -----
	//----- ----- ----- -----fftw3
	fftw_complex *e11, *e22, *e12;
	 e11 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 e22 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 e12 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//----- ----- ----- ----- ----- -----
	
	//float ed11[Nx][Ny];
	float *ed11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ed22[Nx][Ny];
	float *ed22 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ed12[Nx][Ny];
	float *ed12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//Initialize stress & strain componentes
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			ii=i*Ny+j;
			//----- ----- ----- -----
			s11[ii][0] = 0.0;
			s22[ii][0] = 0.0;
			s12[ii][0] = 0.0;
			//
			s11[ii][1] = 0.0;
			s22[ii][1] = 0.0;
			s12[ii][1] = 0.0;
			//----- ----- ----- -----
			e11[ii][0] = 0.0;
			e22[ii][0] = 0.0;
			e12[ii][0] = 0.0;
			//
			e11[ii][1] = 0.0;
			e22[ii][1] = 0.0;
			e12[ii][1] = 0.0;
			//----- ----- ----- -----
			//----- ----- ----- -----
			//Strain components due to lattice defects
			ed11[ii] = 0.0;
			ed22[ii] = 0.0;
			ed12[ii] = 0.0;
			//----- ----- ----- -----
		}
	}
	
	//----- prepare microstructure
	float *con = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	micro_ch_pre_2d(Nx,Ny,c0,con); //Initialize microstructure
	//----- ----- ----- -----
	
	//----- prepare fft (output: kx,ky,kz,k2,k4)
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float k2[Nx][Ny];
	float *k2 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float k4[Nx][Ny];
	float *k4 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	prepare_fft_2d(Nx,Ny,dx,dy,kx,ky,k2,k4); //Calculate coefficients of Fourier transformation
	//----- ----- ----- -----
	
	//----- for cufft
	float *k2_d, *k4_d;
	k2_d  = (float *)malloc(Nx*Ny*sizeof(float));
	k4_d  = (float *)malloc(Nx*Ny*sizeof(float));
	hipMalloc((void**)&k2_d ,Nx*Ny*sizeof(float));
	hipMalloc((void**)&k4_d ,Nx*Ny*sizeof(float));
	hipMemcpy(k2_d,k2,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k2 = k2_h
	hipMemcpy(k4_d,k4,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k4 = k4_h
	//----- ----- ----- -----
	
	//float tmatx[Nx][Ny][2][2][2][2];
	float *tmatx = (float *)malloc(sizeof(float)*( Nx*Ny*2*2*2*2 )); //real part only
	
	//Greens tensor
	green_tensor_2d(Nx,Ny,kx,ky,cm11,cm12,cm44,cp11,cp12,cp44,tmatx); //Calculate Green's tensor
	
	//float *dfdcon = (float *)malloc(sizeof(float)*( Nx*Ny ));
	float *delsdc = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	float _Complex *conc    = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *dfdconc = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *delsdcc = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//
	//float _Complex *conk    = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//float _Complex *dfdconk = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//float _Complex *delsdck = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//
	//float numer, denom;
	//
	//float denom;
	//float _Complex numer;
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(Nx/bs,Ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	//evolve (Time evolution of microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//Update the total time
		ttime=ttime+dtime;
		
		//derivative of elastic energy
		//Calculate the derivative of elastic energy
		solve_elasticity_2d(Nx,Ny,
			tmatx,
			s11,s22,s12,
			e11,e22,e12,
			ed11,ed22,ed12,
			cm11,cm12,cm44,
			cp11,cp12,cp44,
			ea,
			ei0,
			con, delsdc); // Note: tmatx is real part only
		//----- ----- ----- -----
		
		//derivative of free energy and replacement
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//Calculate derivative of free energy
				//dfdcon[ii] = free_energy_ch_2d(con[ii]);
				dfdconc[ii] = free_energy_ch_2d(con[ii]) + 0.0*I;
				//----- ------ ------ ------
				//replace cuda array with host array
				delsdcc[ii] = delsdc[ii];
				conc[ii] = con[ii];
			}
		}
		hipMemcpy(dfdcon_d,dfdconc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //dfdconc = dfdconc_h
		hipMemcpy(delsdc_d,delsdcc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //delsdc = delsdc_h
		hipMemcpy(con_d,conc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //con = con_h
		
		/* Take the values of concentration, derivative of free energy and
		   derivative of elastic energy from real space to Fourier space (forward FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//conk=fft2(con);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_con); //fftw3
		//----- ----- ----- -----
		//hipfftExecR2C(plan, con_d, conk_d); //FFT
		//hipDeviceSynchronize();
		//----- ----- ----- -----
		hipfftExecC2C(plan, con_d, conk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//dfdconk=fft2(dfdcon);      //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_dfdcon); //fftw3
		//----- ----- ----- -----
		//hipfftExecR2C(plan, dfdcon_d, dfdconk_d); //FFT
		//hipDeviceSynchronize();
		//----- ----- ----- -----
		hipfftExecC2C(plan, dfdcon_d, dfdconk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//delsdck=fft2(delsdc);      //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_delsdc); //fftw3
		//----- ----- ----- -----
		//hipfftExecR2C(plan, delsdc_d, delsdck_d); //FFT
		//hipDeviceSynchronize();
		//----- ----- ----- -----
		hipfftExecC2C(plan, delsdc_d, delsdck_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		//hipMemcpy(conk,conk_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //conk = conk_h
		//hipMemcpy(dfdconk,dfdconk_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //dfdconk = dfdconk_h
		//hipMemcpy(delsdck,delsdck_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //delsdck = delsdck_h
		
		/* Semi-implicit time integration of concentration field at
		   Fourier space (Eq.5.50) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- from fftw3
		//for(int i=0;i<Nx;i++){
		//	for(int j=0;j<Ny;j++){
		//		ii=i*Ny+j;
		//		//
		//		denom=1.0+dtime*coefA*mobility*grad_coef*k4[ii];
		//		//
		//		numer=dtime*mobility*k2[ii]*(dfdconk[ii][0]+delsdck[ii][0]);
		//		conk[ii][0]=(conk[ii][0]-numer)/denom;
		//		//
		//		numer=dtime*mobility*k2[ii]*(dfdconk[ii][1]+delsdck[ii][1]);
		//		conk[ii][1]=(conk[ii][1]-numer)/denom;
		//	}
		//}
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- from cufft
		//for(int i=0;i<Nx;i++){
		//	for(int j=0;j<Ny;j++){
		//		ii = i*Ny+j;
		//		//
		//		denom=1.0+dtime*coefA*mobility*grad_coef*k4[ii];
		//		numer=dtime*mobility*k2[ii]*(dfdconk[ii]+delsdck[ii]);
		//		conk[ii]=(conk[ii]-numer)/denom;
		//	}
		//}
		//hipMemcpy(conk_d,conk,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //conk = conk_h
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- on cuda
		Kernel_semi_implicit_time_integration<<<blocks, threads>>>(Nx,Ny,
			dtime,coefA,mobility,grad_coef,
			k2_d,k4_d,
			conk_d,dfdconk_d,delsdck_d);
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- -----
		
		/* Take concentration field from Fourier space back to
		   real space (inverse FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//con=real(ifft2(conk));    //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(iplan_conk); //fftw3
		//----- ----- ----- -----
		//hipfftExecC2R(iplan, conk_d, con_d); //IFFT
		//hipDeviceSynchronize();
		//----- ----- ----- -----
		hipfftExecC2C(iplan, conk_d, con_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		//copy f_d(cuda,device) to F_h(cpu,host)
		//hipMemcpy(con,con_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //con = con_h
		hipMemcpy(conc,con_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //conc = conc_h
		
		//for small deviations
		// For small deviations from max and min values, reset the limits
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//----- ----- ----- -----
				//con[ii] =  con[ii]/(Nx*Ny);
				con[ii] = ( __real__ conc[ii] )/(Nx*Ny);
				//con[ii] =  creal(conc[ii])/(Nx*Ny); //For #include <complex.h>
				//----- ----- ----- -----
				if(con[ii]>=0.9999){
					con[ii]=0.9999;
				}
				if(con[ii]<=0.0001){
					con[ii]=0.0001;
				}
				//----- ----- ----- -----
			}
		}
		
		//print results
		/* If print frequency is reached, output the results to file */
		if(fmod(istep,nprint)==0){
			printf("done step: %5d \n",istep);
			
			//write vtk file
			/* Write the results in vtk format for contour plots
			   to be viewed by using Paraview */
			write_vtk_grid_values_2D(Nx,Ny,dx,dy,istep,con);
		}
		
	}//end of time step (evolve,for)
	
	//calculate the execution time and print it
	/* Calculate the compute time and print it to screen */
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %lf \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(con_d);
	hipFree(dfdcon_d);
	hipFree(delsdc_d);
	//
	hipFree(conk_d);
	hipFree(dfdconk_d);
	hipFree(delsdck_d);
	//
	hipFree(k2_d);
	hipFree(k4_d);
	//
	fftw_free(s11);
	fftw_free(s22);
	fftw_free(s12);
	//
	fftw_free(e11);
	fftw_free(e22);
	fftw_free(e12);
	//----- ----- ----- ----- ----- -----
	free(ed11);
	free(ed22);
	free(ed12);
	//
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(tmatx);
	//
	free(con);
	//
	free(conc);
	free(dfdconc);
	free(delsdcc);
	//
	//free(conk);
	//free(dfdconk);
	//free(delsdck);
	//----- ----- ----- ----- ----- -----
}
