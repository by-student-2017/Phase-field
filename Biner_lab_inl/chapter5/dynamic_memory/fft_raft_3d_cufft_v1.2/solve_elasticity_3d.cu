/* This function evaluates the derivative of elastic energy with
   respect to concentration. First, stress and strain values are
   solved with the iterative algorithm described earlier, 
   then derivative of elastic energy is evaluated for all grid points. */

#include <stdlib.h> //rand() and malloc
#include <math.h>
//#include <fftw3.h>

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cuComplex cufftComplex;

/* Variable and array list
  Nx: Number of grid points in the x-direction
  Ny: Number of grid points in the y-direction
  Nz: Number of grid points in the z-direction
  cm11: C11 component of elasticity matrix for matrix material
  cm12: C12 component of elasticity matrix for matrix material
  cm44: C44 component of elasticity matrix for matrix material
  cp11: C11 component of elasticity matrix for second phase
  cp12: C12 component of elasticity matrix for second phase
  cp44: C44 component of elasticity matrix for second phase
  ed11[Nx][Ny][Nz] to ed13[Nx][Ny][Nz]: Strain component of lattice defects
  ea[6]: Applied strains
  con[Nx][Ny][Nz]: Concentration
  s11[Nx][Ny][Nz] to s13[Nx][Ny][Nz]: Component of stress
  e11[Nx][Ny][Nz] to e13[Nx][Ny][Nz]: Component of strain
  delsdc[Nx][Ny][Nz]: Functional derivative of elastic energy
  //
  omeg11[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg22[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg33[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg12[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg23[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg13[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  tmatx[3][3][3][3]: Green's tensor at i,j,k grid point (real part only)
*/

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_initialization(
	int   Nx, int   Ny, int   Nz,
	float *conr_d,
	float *ei11_d, float *ei22_d, float *ei33_d,
	float *ei12_d, float *ei23_d, float *ei13_d,
	float ei0,
	float *c11_d,  float *c12_d,  float *c44_d,
	float cp11,    float cp12,    float cp44,
	float cm11,    float cm12,    float cm44
){
	int j, jx, jy, jz;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	jz = blockDim.z*blockIdx.z + threadIdx.z; //<-GPU | CPU -> for(jz=0; jz<nz; jz++){
	j  = (jz*Ny + jy)*Nx + jx; //j = nx*ny*jz + nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- -----
	// Calculate the eigenstrains (head ei series)
	ei11_d[j] = ei0*conr_d[j];
	ei22_d[j] = ei0*conr_d[j];
	ei33_d[j] = ei0*conr_d[j];
	//
	//ei12_d[j] = 0.0*conr_d[j];
	//ei23_d[j] = 0.0*conr_d[j];
	//ei13_d[j] = 0.0*conr_d[j];
	ei12_d[j] = 0.0;
	ei23_d[j] = 0.0;
	ei13_d[j] = 0.0;
	//----- ----- ----- ----- -----
	
	/* Calculate the effective elastic constants at 
	   the grid points based on the composition and
	   using Vegard's law */
	//----- ----- ----- ----- -----
	c11_d[j] = conr_d[j]*cp11 + (1.0-conr_d[j])*cm11;
	c12_d[j] = conr_d[j]*cp12 + (1.0-conr_d[j])*cm12;
	c44_d[j] = conr_d[j]*cp44 + (1.0-conr_d[j])*cm44;
	//----- ----- ----- ----- -----
}

void green_tensor2_3D(int Nx, int Ny, int Nz,
	float *kx, float *ky, float *kz,
	float *omeg11, float *omeg22, float *omeg33,
	float *omeg12, float *omeg23, float *omeg13,
	int i, int j, int k,
	float *tmatx);

void solve_elasticity_3d(int Nx, int Ny, int Nz,
	int BSX, int BSY, int BSZ,
	float *kx, float *ky, float *kz,
	float *omeg11, float *omeg22, float *omeg33,
	float *omeg12, float *omeg23, float *omeg13,
	float _Complex *s11, float _Complex *s22, float _Complex *s33,
	float _Complex *s12, float _Complex *s23, float _Complex *s13,
	float _Complex *e11, float _Complex *e22, float _Complex *e33,
	float _Complex *e12, float _Complex *e23, float _Complex *e13,
	float *ed11, float *ed22, float *ed33,
	float *ed12, float *ed23, float *ed13,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea,
	float ei0,
	float *con,  float _Complex *delsdc){
	
	int NxNyNz=Nx*Ny*Nz;
	
	//----- ----- ----- -----
	hipfftComplex *s11_d, *s22_d, *s33_d;
	hipMalloc((void**)&s11_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s22_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s33_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *s12_d, *s23_d, *s13_d;
	hipMalloc((void**)&s12_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s23_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s13_d, sizeof(hipfftComplex)*NxNyNz);
	//----- ----- ----- -----
	hipfftComplex *e11_d, *e22_d, *e33_d;
	hipMalloc((void**)&e11_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e22_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e33_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *e12_d, *e23_d, *e13_d;
	hipMalloc((void**)&e12_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e23_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e13_d, sizeof(hipfftComplex)*NxNyNz);
	//----- ----- ----- -----
	
	//----- ----- ----- -----
	hipfftComplex *s11k_d, *s22k_d, *s33k_d;
	hipMalloc((void**)&s11k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s22k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s33k_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *s12k_d, *s23k_d, *s13k_d;
	hipMalloc((void**)&s12k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s23k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&s13k_d, sizeof(hipfftComplex)*NxNyNz);
	//----- ----- ----- -----
	hipfftComplex *e11k_d, *e22k_d, *e33k_d;
	hipMalloc((void**)&e11k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e22k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e33k_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *e12k_d, *e23k_d, *e13k_d;
	hipMalloc((void**)&e12k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e23k_d, sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&e13k_d, sizeof(hipfftComplex)*NxNyNz);
	//----- ----- ----- -----
	hipfftHandle plan, iplan;
	hipfftPlan3d(&plan,  Nx, Ny, Nz, HIPFFT_C2C);
	hipfftPlan3d(&iplan, Nx, Ny, Nz, HIPFFT_C2C);
	//----- ----- ----- -----
	
	float _Complex *s11k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *s22k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *s33k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *s12k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *s23k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *s13k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	//
	float _Complex *e11k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *e22k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *e33k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *e12k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *e23k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *e13k = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	
	//----- ----- ----- ----- ----- ----- -----
	// eigenstrains (head ei series) components
	//float ei11[Nx][Ny][Nz];
	float *ei11 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei22[Nx][Ny][Nz];
	float *ei22 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei33[Nx][Ny][Nz];
	float *ei33 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//
	//float ei12[Nx][Ny][Nz];
	float *ei12 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei23[Nx][Ny][Nz];
	float *ei23 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei13[Nx][Ny][Nz];
	float *ei13 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- -----
	// elastic modulus components
	//float c11[Nx][Ny][Nz];
	float  *c11 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float c12[Nx][Ny][Nz];
	float  *c12 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float c44[Nx][Ny][Nz];
	float  *c44 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- ----- -----
	
	//----- ----- -----
	int ijk;
	int klij; //For tmatx
	//----- ----- -----
	
	//----- -----
	float *ei11_d, *ei22_d, *ei33_d; // name of dynamic memory for GPU, CUDA, device
	ei11_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	ei22_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	ei33_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	//
	float *ei12_d, *ei23_d, *ei13_d; // name of dynamic memory for GPU, CUDA, device
	ei12_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	ei23_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	ei13_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	//-----
	hipMalloc((void**)&ei11_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei22_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei33_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	//
	hipMalloc((void**)&ei12_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei23_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei13_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	//-----
	hipMemcpy(ei11_d,ei11,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei11 = ei11_h
	hipMemcpy(ei22_d,ei22,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei22 = ei22_h
	hipMemcpy(ei33_d,ei33,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei12 = ei12_h
	//
	hipMemcpy(ei12_d,ei12,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei11 = ei11_h
	hipMemcpy(ei23_d,ei23,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei22 = ei22_h
	hipMemcpy(ei13_d,ei13,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //ei12 = ei12_h
	//----- -----
	float *c11_d, *c12_d, *c44_d; // name of dynamic memory for GPU, CUDA, device
	c11_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	c12_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	c44_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	//-----
	hipMalloc((void**)&c11_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&c12_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&c44_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	//-----
	hipMemcpy(c11_d,c11,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //c11 = c11_h
	hipMemcpy(c12_d,c12,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //c12 = c12_h
	hipMemcpy(c44_d,c44,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //c44 = c44_h
	//----- -----
	float *conr_d;
	conr_d = (float *)malloc(NxNyNz*sizeof(float)); //GPU, CUDA, device
	hipMalloc((void**)&conr_d,NxNyNz*sizeof(float)); // define dynamic memory for GPU (device)
	hipMemcpy(conr_d,con,NxNyNz*sizeof(float),hipMemcpyHostToDevice); //con = con_h
	//----- -----
	
	int bsx=BSX, bsy=BSY, bsz=BSZ;     //Number of threads
	dim3 blocks(Nx/bsx,Ny/bsy,Nz/bsz); //nx*ny*nz = blocks * threads
	dim3 threads(bsx,bsy,bsz);         //bsx*bsy*bsz <= 1024
	
	Kernel_initialization<<<blocks, threads>>>(Nx,Ny,Nz,
		conr_d,
		ei11_d, ei22_d, ei33_d,
		ei12_d, ei23_d, ei13_d,
		ei0,
		c11_d,  c12_d,  c44_d,
		cp11,   cp12,   cp44,
		cm11,   cm12,   cm44);
	hipDeviceSynchronize();
	
	//----- -----
	hipMemcpy(ei11,ei11_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei11 = ei11_h
	hipMemcpy(ei22,ei22_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei22 = ei22_h
	hipMemcpy(ei33,ei33_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei33 = ei33_h
	//
	hipMemcpy(ei12,ei12_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei12 = ei12_h
	hipMemcpy(ei23,ei23_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei23 = ei23_h
	hipMemcpy(ei13,ei13_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //ei13 = ei13_h
	//----- -----
	hipMemcpy(c11,c11_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //c11 = c11_h
	hipMemcpy(c12,c12_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //c12 = c12_h
	hipMemcpy(c44,c44_d,NxNyNz*sizeof(float),hipMemcpyDeviceToHost); //c44 = c44_h
	//----- -----
	
	//for(int i=0;i<Nx;i++){
	//	for(int j=0;j<Ny;j++){
	//		for(int k=0;k<Nz;k++){
	//			ijk=(i*Ny+j)*Nz+k;
	//			//----- ----- ----- ----- -----
	//			// Calculate the eigenstrains (head ei series)
	//			ei11[ijk] = ei0*con[ijk];
	//			ei22[ijk] = ei0*con[ijk];
	//			ei33[ijk] = ei0*con[ijk];
	//			//
	//			ei12[ijk] = 0.0*con[ijk];
	//			ei23[ijk] = 0.0*con[ijk];
	//			ei13[ijk] = 0.0*con[ijk];
	//			//----- ----- ----- ----- -----
	//			
	//			/* Calculate the effective elastic constants at 
	//			   the grid points based on the composition and
	//			   using Vegard's law */
	//			//----- ----- ----- ----- -----
	//			c11[ijk] = con[ijk]*cp11 + (1.0-con[ijk])*cm11;
	//			c12[ijk] = con[ijk]*cp12 + (1.0-con[ijk])*cm12;
	//			c44[ijk] = con[ijk]*cp44 + (1.0-con[ijk])*cm44;
	//			//----- ----- ----- ----- -----
	//		}
	//	}
	//}
	
	/* Note: elastic modulus in this case. */
	//----- ----- ----- -----
	//float c22=c33=c11;
	//float c21=c12;
	//float c31=c13=c12;
	//float c32=c23=c12;
	//float c55=c66=c44;
	//----- ----- ----- -----
	float cm22,cm33;
		cm22=cm33=cm11;
	float cm21;
		cm21=cm12;
	float cm31,cm13;
		cm31=cm13=cm12;
	float cm32,cm23;
		cm32=cm23=cm12;
	float cm55, cm66;
		cm55=cm66=cm44;
	//----- ----- ----- -----
	float cp22, cp33;
		cp22=cp33=cp11;
	float cp21;
		cp21=cp12;
	float cp31, cp13;
		cp31=cp13=cp12;
	float cp32, cp23;
		cp32=cp23=cp12;
	float cp55, cp66;
		cp55=cp66=cp44;
	//----- ----- ----- -----
	float _Complex et21;
	float _Complex et32;
	float _Complex et31;
	//----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- -----
	//float smatx_real[Nx][Ny][Nz][3][3];
	//float *smatx_real = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//
	//float smatx_imag[Nx][Ny][Nz][3][3];
	//float *smatx_imag = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//----- ----- ----- ----- ----- ----- -----
	//float ematx_real[Nx][Ny][Nz][3][3];
	//float *ematx_real = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//
	//float ematx_imag[Nx][Ny][Nz][3][3];
	//float *ematx_imag = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//----- ----- ----- ----- ----- ----- -----
	float _Complex *smatx = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz*3*3 ));
	float _Complex *ematx = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz*3*3 ));
	//----- ----- ----- ----- ----- ----- -----
	
	//float tmatx[3][3][3][3];
	float *tmatx = (float *)malloc(sizeof(float)*( 3*3*3*3 )); //real part only
	
	//----- ----- -----
	float _Complex et11=0.0;
	float _Complex et22=0.0;
	float _Complex et33=0.0;
	//
	float _Complex et12=0.0;
	float _Complex et23=0.0;
	float _Complex et13=0.0;
	//----- ----- -----
	
	//float sum_stress[Nx][Ny][Nz];
	float *sum_stress = (float *)malloc(sizeof(float)*( NxNyNz ));
	
	//----- ----- -----
	//Maximum number of iteration steps
	int niter=10;
	//----- ----- -----
	float old_norm=0.0;
	float normF=0.0;
	//----- ----- -----
	float conver=0.0;
	//----- ----- -----
	//Tolerance value of convergence tests
	float tolerance=0.001;
	//----- ----- -----
	
	/* Solve stress and strain field with 
	   iterative algorithm given in the text */
	for(int iter=0;iter<niter;iter++){
		
		hipMemcpy(s11_d,s11,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s11 = s11_h
		hipMemcpy(s22_d,s22,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s22 = s22_h
		hipMemcpy(s33_d,s33,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s33 = s33_h
		hipMemcpy(s12_d,s12,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s12 = s12_h
		hipMemcpy(s23_d,s23,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s23 = s23_h
		hipMemcpy(s13_d,s13,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //s13 = s13_h
		//
		hipMemcpy(e11_d,e11,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e11 = e11_h
		hipMemcpy(e22_d,e22,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e22 = e22_h
		hipMemcpy(e33_d,e33,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e33 = e33_h
		hipMemcpy(e12_d,e12,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e12 = e12_h
		hipMemcpy(e23_d,e23,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e23 = e23_h
		hipMemcpy(e13_d,e13,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e13 = e13_h
		
		/* Take stress and strain components from real space to
		   Fourier space (forward FFT). Step-a */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		// stress (head s series)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s11_d, s11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s22_d, s22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s33_d, s33k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s12_d, s12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s23_d, s23k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s13_d, s13k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		// strain (head e series)
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e11_d, e11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e22_d, e22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e33_d, e33k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e12_d, e12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e23_d, e23k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e13_d, e13k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(s11k,s11k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s11k = s11k_h
		hipMemcpy(s22k,s22k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s22k = s22k_h
		hipMemcpy(s33k,s33k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s33k = s33k_h
		hipMemcpy(s12k,s12k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s12k = s12k_h
		hipMemcpy(s23k,s23k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s23k = s23k_h
		hipMemcpy(s13k,s13k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //s13k = s13k_h
		//
		hipMemcpy(e11k,e11k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11k = e11k_h
		hipMemcpy(e22k,e22k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22k = e22k_h
		hipMemcpy(e33k,e33k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e33k = e33k_h
		hipMemcpy(e12k,e12k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12k = e12k_h
		hipMemcpy(e23k,e23k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e23k = e23k_h
		hipMemcpy(e13k,e13k_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e13k = e13k_h
		
		//Green operator
		// Calculate strain tensor, Eq.5.46, Step-b
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					
					/* Form stress and strain tensors to be used in 
					   Eq.5.46, Step-b */
					//----- ----- ----- ----- ----- -----
					// stress (smatx, sXY and sXYk)
					//----- ----- ----- ----- ----- -----
					smatx[(ijk*3+0)*3+0]=s11k[ijk];
					smatx[(ijk*3+0)*3+1]=s12k[ijk];
					smatx[(ijk*3+0)*3+2]=s13k[ijk];
					//
					smatx[(ijk*3+1)*3+0]=s12k[ijk];
					smatx[(ijk*3+1)*3+1]=s22k[ijk];
					smatx[(ijk*3+1)*3+2]=s23k[ijk];
					//
					smatx[(ijk*3+2)*3+0]=s13k[ijk];
					smatx[(ijk*3+2)*3+1]=s23k[ijk];
					smatx[(ijk*3+2)*3+2]=s33k[ijk];
					//----- ----- ----- ----- ----- -----
					//
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					//----- ----- ----- ----- ----- -----
					ematx[(ijk*3+0)*3+0]=e11k[ijk];
					ematx[(ijk*3+0)*3+1]=e12k[ijk];
					ematx[(ijk*3+0)*3+2]=e13k[ijk];
					//
					ematx[(ijk*3+1)*3+0]=e12k[ijk];
					ematx[(ijk*3+1)*3+1]=e22k[ijk];
					ematx[(ijk*3+1)*3+2]=e23k[ijk];
					//
					ematx[(ijk*3+2)*3+0]=e13k[ijk];
					ematx[(ijk*3+2)*3+1]=e23k[ijk];
					ematx[(ijk*3+2)*3+2]=e33k[ijk];
					//----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- -----
					green_tensor2_3D(Nx,Ny,Nz,
									kx,ky,kz,
									omeg11,omeg22,omeg33,
									omeg12,omeg23,omeg13,
									i,j,k,
									tmatx);
					//----- ----- ----- ----- ----- -----
					for(int kk=0;kk<3;kk++){
						for(int ll=0;ll<3;ll++){
							for(int ii=0;ii<3;ii++){
								for(int jj=0;jj<3;jj++){
									klij=((kk*3+ll)*3+ii)*3+jj;
									/* Eq.5.46(b): new epsilon(zeta) = epsilon(zeta) - sum( gamma(zeta)*sigma(zeta) )
									   where gamma=tmatx, sigma=smatx
									   Note: tmatx is real part only */
									//----- ----- ----- ----- ----- -----
									//ematx_real[(ijk*3+ii)*3+jj] -= tmatx[klij]*smatx_real[(ijk*3+kk)*3+ll];
									//
									//ematx_imag[(ijk*3+ii)*3+jj] -= tmatx[klij]*smatx_imag[(ijk*3+kk)*3+ll];
									//----- ----- ----- ----- ----- -----
									ematx[(ijk*3+ii)*3+jj] -= tmatx[klij]*smatx[(ijk*3+kk)*3+ll];
									//----- ----- ----- ----- ----- -----
								}//jj
							}//ii
						}//ll
					}//kk
					//----- ----- ----- ----- ----- -----
					
					// Rearrange strain components using symmetry of strain tensor
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					//----- ----- ----- ----- ----- -----
					e11k[ijk]=ematx[(ijk*3+0)*3+0];
					e12k[ijk]=ematx[(ijk*3+0)*3+1];
					e13k[ijk]=ematx[(ijk*3+0)*3+2];
					//
					//e12k[ijk]=ematx[(ijk*3+1)*3+0];
					e22k[ijk]=ematx[(ijk*3+1)*3+1];
					e23k[ijk]=ematx[(ijk*3+1)*3+2];
					//
					//e13k[ijk]=ematx[(ijk*3+2)*3+0];
					//e23k[ijk]=ematx[(ijk*3+2)*3+1];
					e33k[ijk]=ematx[(ijk*3+2)*3+2];
					//----- ----- ----- ----- ----- -----
					
				}//Nz
			}//Ny
		}//Nx
		
		hipMemcpy(e11k_d,e11k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e11k = e11k_h
		hipMemcpy(e22k_d,e22k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e22k = e22k_h
		hipMemcpy(e33k_d,e33k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e33k = e33k_h
		hipMemcpy(e12k_d,e12k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e12k = e12k_h
		hipMemcpy(e23k_d,e23k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e23k = e23k_h
		hipMemcpy(e13k_d,e13k,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //e13k = e13k_h
		
		//From Fourier space to real space
		/* Take strain components from Fourier space back to
		   real space (inverse FFT), Step-c */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e11k_d, e11_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e22k_d, e22_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e33k_d, e33_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e12k_d, e12_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e23k_d, e23_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e13k_d, e13_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(e11,e11_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11 = e11_h
		hipMemcpy(e22,e22_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22 = e22_h
		hipMemcpy(e33,e33_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e33 = e33_h
		hipMemcpy(e12,e12_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12 = e12_h
		hipMemcpy(e23,e23_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e23 = e23_h
		hipMemcpy(e13,e13_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //e13 = e13_h
		
		//Calculate stresses
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					//----- ----- ----- -----
					e11[ijk] /= NxNyNz;
					e22[ijk] /= NxNyNz;
					e33[ijk] /= NxNyNz;
					//
					e12[ijk] /= NxNyNz;
					e23[ijk] /= NxNyNz;
					e13[ijk] /= NxNyNz;
					//----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- ----- ----- -----
					/*s11[ijk][0]=c11[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c13[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c21[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c22[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c23[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c31[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c32[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c33[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);*/
					//
					// c13[ijk]=c12[ijk], c23[ijk]=c13[ijk], c22[ijk]=c11[ijk], c33[ijk]=c11[ijk], etc
					/*s11[ijk][0]=c11[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c12[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c12[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c11[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c12[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s33[ijk][0]=c12[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c11[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);*/
					//
					// Calculate strain (head e series) components
					et11=( ea[0]+e11[ijk]-ei11[ijk]-ed11[ijk] );
					et22=( ea[1]+e22[ijk]-ei22[ijk]-ed22[ijk] );
					et33=( ea[2]+e33[ijk]-ei33[ijk]-ed33[ijk] );
					//
					// Calculate stress (head s series) components
					s11[ijk]=( c11[ijk]*et11 + c12[ijk]*et22 + c12[ijk]*et33 );
					s22[ijk]=( c12[ijk]*et11 + c11[ijk]*et22 + c12[ijk]*et33 );
					s33[ijk]=( c12[ijk]*et11 + c12[ijk]*et22 + c11[ijk]*et33 );
					//
					//s11[ijk][1]=0.0;
					//s22[ijk][1]=0.0;
					//s33[ijk][1]=0.0;
					//----- ----- ----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- ----- ----- -----
					/*s12[ijk][0]=c44[ijk]*(ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk])
							   +c44[ijk]*(ea[3]+e21[ijk][0]-ei21[ijk]-ed21[ijk]);
					s23[ijk][0]=c55[ijk]*(ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk])
							   +c55[ijk]*(ea[4]+e32[ijk][0]-ei32[ijk]-ed32[ijk]);
					s13[ijk][0]=c66[ijk]*(ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk])
							   +c66[ijk]*(ea[5]+e31[ijk][0]-ei31[ijk]-ed31[ijk]);*/
					//
					// c55[ijk]=c44[ijk], c66[ijk]=c44[ijk], etc
					/*s12[ijk][0]=c44[ijk]*(ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk])*2.0;
					  s23[ijk][0]=c44[ijk]*(ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk])*2.0;
					  s13[ijk][0]=c44[ijk]*(ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk])*2.0;*/
					//
					// Calculate strain (head e series) components
					et12=( ea[3]+e12[ijk]-ei12[ijk]-ed12[ijk] );
					et23=( ea[4]+e23[ijk]-ei23[ijk]-ed23[ijk] );
					et13=( ea[5]+e13[ijk]-ei13[ijk]-ed13[ijk] );
					//
					// Calculate stress (head s series) components
					s12[ijk]=c44[ijk]*et12*2.0;
					s23[ijk]=c44[ijk]*et23*2.0;
					s13[ijk]=c44[ijk]*et13*2.0;
					//
					//s12[ijk][1]=0.0;
					//s23[ijk][1]=0.0;
					//s13[ijk][1]=0.0;
					//----- ----- ----- ----- ----- ----- ----- -----
					
					//check convergence
					sum_stress[ijk] = __real__ ( s11[ijk] + s22[ijk] + s33[ijk]
												+s12[ijk] + s23[ijk] + s13[ijk] );
					//normF=norm(sum_stress,2.0);
					normF = normF + sum_stress[ijk]*sum_stress[ijk];
				}
			}
		}
		
		normF=sqrt(normF);
		
		if(iter==1){
			conver=fabs((normF-old_norm)/(old_norm));
			if(conver<=tolerance){
				break;
			}
		}
		old_norm=normF;
		
	}//end iter
	
	//strain energy
	//Calculate functional derivative of elastic energy
	// sum strain components
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				ijk=(i*Ny+j)*Nz+k;
				//----- ----- ----- ----- ----- ----- ----- -----
				// Calculate strain (head e series) components
				et11=( ea[0]+e11[ijk]-ei11[ijk]-ed11[ijk] );
				et22=( ea[1]+e22[ijk]-ei22[ijk]-ed22[ijk] );
				et33=( ea[2]+e33[ijk]-ei33[ijk]-ed33[ijk] );
				//
				et12=( ea[3]+e12[ijk]-ei12[ijk]-ed12[ijk] );
				et23=( ea[4]+e23[ijk]-ei23[ijk]-ed23[ijk] );
				et13=( ea[5]+e13[ijk]-ei13[ijk]-ed13[ijk] );
				//----- ----- ----- ----- ----- ----- ----- -----
				
				//----- ----- ----- ----- ----- ----- ----- -----
				//Functional derivative of the elastic energy with respect to composition
				/* F=(1/2)*sigma[i][j]*(epsilon[i][j] - epsilon0[i][j])
				   sigma[i][j] = C[i][j][k][l]*(epsilon[k][l] - epsilon0[k][l])
				   epsilon0[i][j] is the position- and composition-dependent eigenstrains */
				/* dF/dc = dF/d(con) = (1/2)*( dCijkl/d(con)*etij*etkl + Cijkl*d(etij)/d(con)*etkl + Cijkl*etij*d(etkl)/d(con) )
				   dCijkl/d(con) = (cpijkl - cmijkl), d(etij)/d(con) = -d(eiij)/d(con) = -ei0 */
				/* delsdc[ijk][0]=0.5*( (cp11-cm11)*et11*et11 -c11[ijk]*ei0*et11 -c11[ijk]*et11*ei0
								    +(cp12-cm12)*et11*et22 -c12[ijk]*ei0*et11 -c12[ijk]*et22*ei0
								    +(cp13-cm13)*et11*et33 -c13[ijk]*ei0*et11 -c13[ijk]*et33*ei0
								    //
								    +(cp21-cm21)*et22*et11 -c21[ijk]*ei0*et22 -c21[ijk]*et11*ei0
								    +(cp22-cm22)*et22*et22 -c22[ijk]*ei0*et22 -c22[ijk]*et22*ei0
								    +(cp23-cm23)*et22*et33 -c23[ijk]*ei0*et22 -c23[ijk]*et33*ei0
								    //
								    +(cp31-cm31)*et33*et11 -c31[ijk]*ei0*et33 -c31[ijk]*et11*ei0
								    +(cp32-cm32)*et33*et22 -c32[ijk]*ei0*et33 -c32[ijk]*et22*ei0
								    +(cp33-cm33)*et33*et33 -c33[ijk]*ei0*et33 -c33[ijk]*et33*ei0
								    //
								    +(cp44-cm44)*et12*et12 -c44[ijk]*ei0*et12 -c44[ijk]*et12*ei0
								    +(cp44-cm44)*et21*et21 -c44[ijk]*ei0*et21 -c44[ijk]*et21*ei0
								    //
								    +(cp55-cm55)*et23*et23 -c55[ijk]*ei0*et23 -c55[ijk]*et23*ei0
								    +(cp55-cm55)*et32*et32 -c55[ijk]*ei0*et32 -c55[ijk]*et32*ei0
								    //
								    +(cp66-cm66)*et13*et13 -c66[ijk]*ei0*et13 -c66[ijk]*et13*ei0
								    +(cp66-cm66)*et31*et31 -c66[ijk]*ei0*et31 -c66[ijk]*et31*ei0
								   ); */
				//
				// c13[ijk]=c12[ijk], c23[ijk]=c13[ijk], c22[ijk]=c11[ijk], c33[ijk]=c11[ijk], etc
				et21=et12;
				et32=et23;
				et31=et13;
				//
				delsdc[ijk]=0.5*( (cp11-cm11)*et11*et11 -c11[ijk]*ei0*( et11 + et11 )
								    +(cp12-cm12)*et11*et22 -c12[ijk]*ei0*( et11 + et22 )
								    +(cp13-cm13)*et11*et33 -c12[ijk]*ei0*( et11 + et33 )
								    //
								    +(cp21-cm21)*et22*et11 -c12[ijk]*ei0*( et22 + et11 )
								    +(cp22-cm22)*et22*et22 -c11[ijk]*ei0*( et22 + et22 )
								    +(cp23-cm23)*et22*et33 -c12[ijk]*ei0*( et22 + et33 )
								    //
								    +(cp31-cm31)*et33*et11 -c12[ijk]*ei0*( et33 + et11 )
								    +(cp32-cm32)*et33*et22 -c12[ijk]*ei0*( et33 + et22 )
								    +(cp33-cm33)*et33*et33 -c11[ijk]*ei0*( et33 + et33 )
								    //
								    +(cp44-cm44)*et12*et12 -c44[ijk]*ei0*( et12 + et12 )
								    +(cp44-cm44)*et21*et21 -c44[ijk]*ei0*( et21 + et21 )
								    //
								    +(cp55-cm55)*et23*et23 -c44[ijk]*ei0*( et23 + et23 )
								    +(cp55-cm55)*et32*et32 -c44[ijk]*ei0*( et32 + et32 )
								    //
								    +(cp66-cm66)*et13*et13 -c44[ijk]*ei0*( et13 + et13 )
								    +(cp66-cm66)*et31*et31 -c44[ijk]*ei0*( et31 + et31 )
								   );
				//delsdc[ijk][1]=0.0;
				//----- ----- ----- ----- ----- ----- ----- -----
			}
		}
	}
	
	//----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- -----
	hipFree(s11_d);
	hipFree(s22_d);
	hipFree(s33_d);
	//
	hipFree(s12_d);
	hipFree(s23_d);
	hipFree(s13_d);
	//----- ----- ----- ----- -----
	hipFree(e11_d);
	hipFree(e22_d);
	hipFree(e33_d);
	//
	hipFree(e12_d);
	hipFree(e23_d);
	hipFree(e13_d);
	//----- ----- ----- ----- -----
	hipFree(s11k_d);
	hipFree(s22k_d);
	hipFree(s33k_d);
	//
	hipFree(s12k_d);
	hipFree(s23k_d);
	hipFree(s13k_d);
	//----- ----- ----- ----- -----
	hipFree(e11k_d);
	hipFree(e22k_d);
	hipFree(e33k_d);
	//
	hipFree(e12k_d);
	hipFree(e23k_d);
	hipFree(e13k_d);
	//----- ----- ----- ----- ----- -----
	hipFree(ei11_d);
	hipFree(ei22_d);
	hipFree(ei33_d);
	//
	hipFree(ei12_d);
	hipFree(ei23_d);
	hipFree(ei13_d);
	//----- ----- ----- ----- ----- -----
	hipFree(c11_d);
	hipFree(c12_d);
	hipFree(c44_d);
	//----- ----- ----- ----- ----- -----
	hipFree(conr_d);
	//----- ----- ----- ----- ----- -----
	free(s11k);
	free(s22k);
	free(s33k);
	//
	free(s12k);
	free(s23k);
	free(s13k);
	//----- ----- ----- ----- -----
	free(e11k);
	free(e22k);
	free(e33k);
	//
	free(e12k);
	free(e23k);
	free(e13k);
	//----- ----- ----- ----- ----- -----
	//free(smatx_real);
	//free(smatx_imag);
	//free(ematx_real);
	//free(ematx_imag);
	free(smatx);
	free(ematx);
	//----- ----- ----- ----- -----
	free(sum_stress);
	//----- ----- ----- ----- -----
	free(ei11);
	free(ei22);
	free(ei33);
	//
	free(ei12);
	free(ei23);
	free(ei13);
	//----- ----- ----- ----- -----
	free(c11);
	free(c12);
	free(c44);
	//----- ----- ----- ----- -----
	
	return;
}