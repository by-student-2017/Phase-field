#include "hip/hip_runtime.h"
/* 2D semi-implicit spectral phase-field code 
  for solving precipitation in FeCr alloy */

/* This program solves conserved phase-field equation with
   Fourier spactral method by taking into account the effects of
   elastic inhomogeneities and lattice defects, based on
   solution of stress-strain fields ith Green's tensor and
   Fourier transformations. the time integration is
   carried out by using semi-implicit time machining scheme. */

#include <stdio.h>
#include <stdlib.h> //rand() and malloc
#include <math.h> //mod() and -lm
#include <time.h>

#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <hip/hip_complex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <hip/hip_runtime.h> or
  #include "hip/hip_runtime.h"
  #include "" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float hipfftReal;
//typedef hipComplex hipfftComplex;

void dislo_strain_2d(int Nx, int Ny, int idislo,
	float *ed11, float *ed22, float *ed12);

float FeCr_chem_poten_2d(float cr_ij, float tempr);

void green_tensor_2d(int Nx, int Ny,
	float *kx, float *ky,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *tmatx);

void prepare_fft_2d(int Nx, int Ny, 
	float dx, float dy,
	float *kx, float *ky, 
	float *k2, float *k4);

void solve_elasticity_2d(int Nx, int Ny,
	float *tmatx,
	fftw_complex *s11, fftw_complex *s22, fftw_complex *s12,
	fftw_complex *e11, fftw_complex *e22, fftw_complex *e12,
	float *ed11, float *ed22, float *ed12,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea,
	float ei0,
	float *con, float *delsdc);

void micro_ch_pre_2d(int Nx, int Ny, float c0, float *con);

void write_vtk_grid_values_2D(int nx, int ny, 
	float dx, float dy,
	int istep, float *data1);

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	//(Get initial wall clock time beginning of the execution)
	start = clock();
	
	//simulation cell parameters
	int Nx=128;
	int Ny=128;
	
	//Number of threads, 2^n=<32, BS*BS*1 <= 1024
	int BS=32;
	
	//Total number of grid points in the simulation cell
	//int NxNy=Nx*Ny;
	
	//The distance between two grid points in x,y-direction
	float dx=1.0; // [nm] unit ?
	float dy=1.0; // [nm] unit ?
	
	//time integration parameters
	int nstep=10000; //Number of time steps
	int nprint=50;  //Print frequency to write the results to file
	float dtime=1.0e-2; //Time increment for numerical integration
	float ttime=0.0;    //Total time
	float coefA=2.0;
	
	//material specific parameters
	
	//Initial concentrations of alloying elements
	float c0=0.20;       //Initial concentraion (20%Cr-containing Fe-Cr alloy
	float mobility=1.0;  //The value of mobility coefficient (dimensionless)
	float grad_coef=0.5; //The value of gradient energy coefficients [J(nm)^2/mol]
	
	float tempr=535.0; //Annealing temperature [K]
	float RT=8.314462*tempr; //Gas constant x temperature
	
	//elastic constants
	//Elastic constants of Fe-rich phase [GPa]
	float cm11=233.10e3;
	float cm12=135.44e3;
	float cm44=178.30e3;
	//
	//Elastic constants of Cr-rich phase [GPa]
	float cp11=350.00e3;
	float cp12= 67.80e3;
	float cp44=100.80e3;
	
	//elastic constant of other materials
	//Ref: https://www.jstage.jst.go.jp/article/jsms/69/9/69_657/_pdf
	
	//eigen strains
	//The value of eigenstrains for Cr-rich phase
	float ei0=0.006; //Maginitude of eigenstrains
	
	int ii; //ii=(i*Ny+j);
	
	//----- ----- ----- ----- ----- -----
	const int fftsizex = Nx, fftsizey = Ny;
	//
	hipfftComplex *cr_d, *dfdcr_d, *delsdc_d;
	hipMalloc((void**)&cr_d,      sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdcr_d,   sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&delsdc_d,  sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftComplex *crk_d, *dfdcrk_d, *delsdck_d;
	hipMalloc((void**)&crk_d,     sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&dfdcrk_d,  sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&delsdck_d, sizeof(hipfftComplex)*Nx*Ny);
	//
	hipfftHandle plan, iplan;
	//hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_R2C);
	//hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2R);
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny;
	/* fftw_complex *in, *out; // in[i][0] for real, in[i][1] for imag.
	    in = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	   out = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	   fftw_plan plan, iplan;
	   plan = fftw_plan_dft_2d(fftsize, fftsize, in, out, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	  iplan = fftw_plan_dft_2d(fftsize, fftsize, in, out, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT */
	//
	//array
	//fftw_complex *cr, *crk;
	// cr = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//crk = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//fftw_plan plan_cr, iplan_crk;
	// plan_cr  = fftw_plan_dft_2d(fftsizex, fftsizey, cr, crk, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_crk = fftw_plan_dft_2d(fftsizex, fftsizey, crk,cr,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//
	//array
	//fftw_complex *dfdcr, *dfdcrk;
	// dfdcr = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//dfdcrk = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//fftw_plan plan_dfdcr, iplan_dfdcrk;
	//fftw_plan plan_dfdcr;
	// plan_dfdcr  = fftw_plan_dft_2d(fftsizex, fftsizey, dfdcr, dfdcrk, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_dfdcrk = fftw_plan_dft_2d(fftsizex, fftsizey, dfdcrk, dfdcr, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//
	//array
	//fftw_complex *delsdc, *delsdck;
	// delsdc = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//delsdck = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//fftw_plan plan_delsdc, iplan_delsdck;
	//fftw_plan plan_delsdc;
	// plan_delsdc  = fftw_plan_dft_2d(fftsizex, fftsizey, delsdc, delsdck, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_delsdck = fftw_plan_dft_2d(fftsizex, fftsizey, delsdck, delsdc, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- ----- ----- -----
	
	//----- ----- ----- -----fftw3
	fftw_complex *s11, *s22, *s12;
	 s11 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 s22 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 s12 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//----- ----- ----- -----
	//----- ----- ----- -----fftw3
	fftw_complex *e11, *e22, *e12;
	 e11 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 e22 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	 e12 = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//----- ----- ----- ----- ----- -----
	
	//Initialize stress & strain componentes
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			ii=i*Ny+j;
			//----- ----- ----- -----
			s11[ii][0] = 0.0;
			s22[ii][0] = 0.0;
			s12[ii][0] = 0.0;
			//
			s11[ii][1] = 0.0;
			s22[ii][1] = 0.0;
			s12[ii][1] = 0.0;
			//----- ----- ----- -----
			e11[ii][0] = 0.0;
			e22[ii][0] = 0.0;
			e12[ii][0] = 0.0;
			//
			e11[ii][1] = 0.0;
			e22[ii][1] = 0.0;
			e12[ii][1] = 0.0;
			//----- ----- ----- -----
		}
	}
	
	//Strain components due to lattice defects
	//float ed11[Nx][Ny];
	float *ed11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ed22[Nx][Ny];
	float *ed22 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ed12[Nx][Ny];
	float *ed12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//dislocation eigen strain
	/* idislo=1 for dislocation diploe,
	   idislo=2 for dislocation array */
	int idislo=1;
	dislo_strain_2d(Nx,Ny,idislo,ed11,ed22,ed12);
	
	//Applied strains
	// The components of applied strains
	float ea[3]; //Magnitude of applied strains
	ea[0]=0.0;
	ea[1]=0.0;
	ea[2]=0.0;
	
	//int iflag=1;
	
	//----- prepare microstructure
	float *cr = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	micro_ch_pre_2d(Nx,Ny,c0,cr); //Initialize microstructure
	//----- ----- ----- -----
	
	//----- prepare fft (output: kx,ky,kz,k2,k4)
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float k2[Nx][Ny];
	float *k2 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float k4[Nx][Ny];
	float *k4 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//prepare fft (output: kx,ky,kz,k2,k4)
	prepare_fft_2d(Nx,Ny,dx,dy,kx,ky,k2,k4); //Calculate coefficients of Fourier transformation
	//----- ----- ----- -----
	
	//float tmatx[Nx][Ny][2][2][2][2];
	float *tmatx = (float *)malloc(sizeof(float)*( Nx*Ny*2*2*2*2 )); //real part only
	
	//Greens tensor
	green_tensor_2d(Nx,Ny,kx,ky,cm11,cm12,cm44,cp11,cp12,cp44,tmatx); //Calculate Green's tensor
	
	//float *dfdcr = (float *)malloc(sizeof(float)*( Nx*Ny ));
	float *delsdc = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//
	float _Complex *crc     = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *dfdcrc  = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *delsdcc = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//
	float _Complex *crk     = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *dfdcrk  = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *delsdck = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	
	//float numer, denom;
	//
	float denom;
	float _Complex numer;
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(Nx/bs,Ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	//evolve (Time evolution of microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//Update the total time
		ttime=ttime+dtime;
		
		//derivative of chemical energy
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//Calculate derivative of chemical energy
				dfdcrc[ii]=FeCr_chem_poten_2d(cr[ii],tempr);
			}
		}
		hipMemcpy(dfdcr_d,dfdcrc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //dfdcrc = dfdcrc_h
		
		//derivative of elastic energy
		//Calculate the derivative of elastic energy
		solve_elasticity_2d(Nx,Ny,
			tmatx,
			s11,s22,s12,
			e11,e22,e12,
			ed11,ed22,ed12,
			cm11,cm12,cm44,
			cp11,cp12,cp44,
			ea,
			ei0,
			cr, delsdc);
		// Note: tmatx is real part only. dslsc is output.
		//----- ----- ----- -----
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//replace cuda array with host array
				delsdcc[ii] = delsdc[ii]/RT; //And normalize the derivative elastic energy with RT
				crc[ii] = cr[ii];
			}
		}
		hipMemcpy(delsdc_d,delsdcc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //delsdcc = delsdcc_h
		hipMemcpy(cr_d,crc,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //cr = cr_h
		
		/* Take the values of concentration, derivative of free energy and
		   derivative of elastic energy from real space to Fourier space (forward FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//crk=fft2(cr);              //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_cr);     //fftw3
		//----- ----- ----- -----
		hipfftExecC2C(plan, cr_d, crk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//dfdcrk=fft2(dfdcr);        //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_dfdcr);  //fftw3
		//----- ----- ----- -----
		hipfftExecC2C(plan, dfdcr_d, dfdcrk_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//delsdck=fft2(delsdc);      //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_delsdc); //fftw3
		//----- ----- ----- -----
		hipfftExecC2C(plan, delsdc_d, delsdck_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(crk,crk_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //crk = crk_h
		hipMemcpy(dfdcrk,dfdcrk_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //dfdcrk = dfdcrk_h
		hipMemcpy(delsdck,delsdck_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //delsdck = delsdck_h
		
		/* Semi-implicit time integration of Cr concentration field at
		   Fourier space (Eq.5.50) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- from fftw3
		//for(int i=0;i<Nx;i++){
		//	for(int j=0;j<Ny;j++){
		//		ii=i*Ny+j;
		//		//
		//		denom=1.0+dtime*coefA*mobility*grad_coef*k4[ii];
		//		//
		//		numer=dtime*mobility*k2[ii]*(dfdcrk[ii][0]+delsdck[ii][0]);
		//		crk[ii][0]=(crk[ii][0]-numer)/denom;
		//		//
		//		numer=dtime*mobility*k2[ii]*(dfdcrk[ii][1]+delsdck[ii][1]);
		//		crk[ii][1]=(crk[ii][1]-numer)/denom;
		//	}
		//}
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- from cufft
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii = i*Ny+j;
				//
				denom=1.0+dtime*coefA*mobility*grad_coef*k4[ii];
				numer=dtime*mobility*k2[ii]*(dfdcrk[ii]+delsdck[ii]);
				crk[ii]=(crk[ii]-numer)/denom;
			}
		}
		hipMemcpy(crk_d,crk,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //crk = crk_h
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		/* Take concentration field from Fourier space back to
		   real space (inverse FFT) */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//cr=real(ifft2(crk));     //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(iplan_crk); //fftw3
		//----- ----- ----- -----
		hipfftExecC2C(iplan, crk_d, cr_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		//copy f_d(cuda,device) to F_h(cpu,host)
		//hipMemcpy(cr,cr_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //cr = cr_h
		hipMemcpy(crc,cr_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //crc = crc_h
		
		//for small deviations
		// For small deviations from max and min values, reset the limits
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//----- ----- ----- -----
				//cr[ii] =  cr[ii]/(Nx*Ny);
				cr[ii] = ( __real__ crc[ii] )/(Nx*Ny);
				//cr[ii] =  creal(crc[ii])/(Nx*Ny); //For #include <complex.h>
				//----- ----- ----- -----
				if(cr[ii]>=0.9999){
					cr[ii]=0.9999;
				}
				if(cr[ii]<=0.0001){
					cr[ii]=0.0001;
				}
				//----- ----- ----- -----
			}
		}
		
		//print results
		/* If print frequency is reached, output the results to file */
		if(fmod(istep,nprint)==0){
			//write vtk file
			/* Write the results in vtk format for contour plots
			   to be viewed by using Paraview */
			write_vtk_grid_values_2D(Nx,Ny,dx,dy,istep,cr);
			
			printf("done step: %5d, time: %f \n",istep,ttime*(mobility/(dx*dx)));
			/* The quantities having the dimension of distance were normalized with the
			   magnitude of the Burger's vector, the  quantities having the dimension of
			   energy were normalized with RT, and the time t was normalized with M/(dx^2).
			   The initial concentration was modulated by setting the noise term to
			   0.001 in function micro_ch_pre_2d.c */
		}
		
	}//end of time step (evolve,for)
	
	//calculate the execution time and print it
	/* Calculate the compute time and print it to screen */
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %lf \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(cr_d);
	hipFree(dfdcr_d);
	hipFree(delsdc_d);
	//
	hipFree(crk_d);
	hipFree(dfdcrk_d);
	hipFree(delsdck_d);
	//----- ----- ----- ----- ----- -----
	fftw_free(s11);
	fftw_free(s22);
	fftw_free(s12);
	//
	fftw_free(e11);
	fftw_free(e22);
	fftw_free(e12);
	//----- ----- ----- ----- ----- -----
	free(ed11);
	free(ed22);
	free(ed12);
	//
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(tmatx);
	//
	free(cr);
	//
	free(crc);
	free(dfdcrc);
	free(delsdcc);
	//
	free(crk);
	free(dfdcrk);
	free(delsdck);
	//----- ----- ----- ----- ----- -----
}
