/* This function evaluates the derivative of elastic energy with
   respect to concentration. First, stress and strain values are
   solved with the iterative algorithm described earlier, 
   then derivative of elastic energy is evaluated for all grid points. */

#include <stdlib.h> //rand() and malloc
#include <math.h>
//#include <fftw3.h>

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cuComplex cufftComplex;

/* Variable and array list
  Nx: Number of grid points in the x-direction
  Ny: Number of grid points in the y-direction
  cm11: C11 component of elasticity matrix for matrix material
  cm12: C12 component of elasticity matrix for matrix material
  cm44: C44 component of elasticity matrix for matrix material
  cp11: C11 component of elasticity matrix for second phase
  cp12: C12 component of elasticity matrix for second phase
  cp44: C44 component of elasticity matrix for second phase
  ed11: Strain component of lattice defects
  ed22: Strain component of lattice defects
  ed12: Strain component of lattice defects
  ei0: Magnitude of eigenstrains
  ea[3]: Applied strains
  con[Nx][Ny]: Concentration
  s11[Nx][Ny]: Component of stress
  s22[Nx][Ny]: Component of stress
  s12[Nx][Ny]: Component of stress
  e11[Nx][Ny]: Component of strain
  e22[Nx][Ny]: Component of strain
  e12[Nx][Ny]: Component of strain
  delsdc[Nx][Ny]: Functional derivative of elastic energy
  tmatx[Nx][Ny][2][2][2][2]: Values of Green's tensor at all grid points (real part only)
*/

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_initialization(
	int   Nx, int   Ny,
	float *conr_d,
	float *ei11_d, float *ei22_d, float *ei12_d,
	float ei0,
	float *c11_d,  float *c12_d,  float *c44_d,
	float cp11,    float cp12,    float cp44,
	float cm11,    float cm12,    float cm44
){
	int j, jx, jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	j  = Ny*jx + jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	//
	//Calculate the eigenstrains
	ei11_d[j]=ei0*conr_d[j];
	ei22_d[j]=ei0*conr_d[j];
	//ei12[j]=0.0*conr_d[j];
	ei12_d[j]=0.0;
	
	/* Calculate the effective elastic constants at 
	   the grid points based on the composition and
	   using Vegard's law */
	c11_d[j]=conr_d[j]*cp11+(1.0-conr_d[j])*cm11;
	c12_d[j]=conr_d[j]*cp12+(1.0-conr_d[j])*cm12;
	c44_d[j]=conr_d[j]*cp44+(1.0-conr_d[j])*cm44;
}

void solve_elasticity_2d(int Nx, int Ny, int BS,
	float *tmatx,
	float _Complex *s11, float _Complex *s22, float _Complex *s12,
	float _Complex *e11, float _Complex *e22, float _Complex *e12,
	float *ed11, float *ed22, float *ed12,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea,
	float ei0,
	float *con,  float _Complex *delsdc){
	
	//----- ----- ----- -----
	hipfftComplex *s11_d, *s22_d, *s12_d;
	hipMalloc((void**)&s11_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s22_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s12_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftComplex *e11_d, *e22_d, *e12_d;
	hipMalloc((void**)&e11_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e22_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e12_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	
	//----- ----- ----- -----
	hipfftComplex *s11k_d, *s22k_d, *s12k_d;
	hipMalloc((void**)&s11k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s22k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s12k_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftComplex *e11k_d, *e22k_d, *e12k_d;
	hipMalloc((void**)&e11k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e22k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e12k_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftHandle plan, iplan;
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- -----
	
	float _Complex *s11k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *s22k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *s12k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//
	float _Complex *e11k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *e22k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *e12k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	
	//----- ----- ----- -----
	
	float _Complex *smatx = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny*2*2 ));
	float _Complex *ematx = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny*2*2 ));
	
	//float sum_stress[Nx][Ny];
	float *sum_stress = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	float old_norm=0.0;
	float normF=0.0;
	
	float conver=0.0;
	
	float _Complex et11 = 0.0;
	float _Complex et22 = 0.0;
	float _Complex et12 = 0.0;
	
	int ii, ij;
	
	//Maximum number of iteration steps
	int niter=10;
	
	//Tolerance value of convergence tests
	float tolerance=0.001;
	
	//float ei11[Nx][Ny];
	float *ei11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ei22[Nx][Ny];
	float *ei22 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ei12[Nx][Ny];
	float *ei12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//float c11[Nx][Ny];
	float  *c11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float c12[Nx][Ny];
	float  *c12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float c44[Nx][Ny];
	float  *c44 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//----- -----
	float *ei11_d, *ei22_d, *ei12_d; // name of dynamic memory for GPU, CUDA, device
	ei11_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	ei22_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	ei12_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	//-----
	hipMalloc((void**)&ei11_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei22_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&ei12_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	//-----
	hipMemcpy(ei11_d,ei11,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //ei11 = ei11_h
	hipMemcpy(ei22_d,ei22,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //ei22 = ei22_h
	hipMemcpy(ei12_d,ei12,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //ei12 = ei12_h
	//----- -----
	float *c11_d, *c12_d, *c44_d; // name of dynamic memory for GPU, CUDA, device
	c11_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	c12_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	c44_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	//-----
	hipMalloc((void**)&c11_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&c12_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&c44_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	//-----
	hipMemcpy(c11_d,c11,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //c11 = c11_h
	hipMemcpy(c12_d,c12,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //c12 = c12_h
	hipMemcpy(c44_d,c44,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //c44 = c44_h
	//----- -----
	float *conr_d;
	conr_d = (float *)malloc(Nx*Ny*sizeof(float)); //GPU, CUDA, device
	hipMalloc((void**)&conr_d,Nx*Ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMemcpy(conr_d,con,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //con = con_h
	//----- -----
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(Nx/bs,Ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	Kernel_initialization<<<blocks, threads>>>(Nx,Ny,
		conr_d,
		ei11_d, ei22_d, ei12_d,
		ei0,
		c11_d,  c12_d,  c44_d,
		cp11,   cp12,   cp44,
		cm11,   cm12,   cm44);
	hipDeviceSynchronize();
	
	//----- -----
	hipMemcpy(ei11,ei11_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //ei11 = ei11_h
	hipMemcpy(ei22,ei22_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //ei22 = ei22_h
	hipMemcpy(ei12,ei12_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //ei12 = ei12_h
	//----- -----
	hipMemcpy(c11,c11_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //c11 = c11_h
	hipMemcpy(c12,c12_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //c12 = c12_h
	hipMemcpy(c44,c44_d,Nx*Ny*sizeof(float),hipMemcpyDeviceToHost); //c44 = c44_h
	//----- -----
	
	/* Solve stress and strain field with 
	   iterative algorithm given in the text */
	for(int iter=0;iter<niter;iter++){
		
		hipMemcpy(s11_d,s11,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s11 = s11_h
		hipMemcpy(s22_d,s22,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s22 = s22_h
		hipMemcpy(s12_d,s12,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s12 = s12_h
		//
		hipMemcpy(e11_d,e11,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e11 = e11_h
		hipMemcpy(e22_d,e22,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e22 = e22_h
		hipMemcpy(e12_d,e12,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e12 = e12_h
		
		/* Take stress and strain components from real space to
		   Fourier space (forward FFT). Step-a */
		// stress
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s11_d, s11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s22_d, s22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, s12_d, s12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//
		// strain
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e11_d, e11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e22_d, e22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(plan, e12_d, e12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(s11k,s11k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s11k = s11k_h
		hipMemcpy(s22k,s22k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s22k = s22k_h
		hipMemcpy(s12k,s12k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s12k = s12k_h
		//
		hipMemcpy(e11k,e11k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11k = e11k_h
		hipMemcpy(e22k,e22k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22k = e22k_h
		hipMemcpy(e12k,e12k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12k = e12k_h}
		
		//Green operator
		// Calculate strain tensor, Eq.5.46, Step-b
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ij=i*Ny+j;
				
				//----- ----- ----- ----- ----- 
				/* Form stress and strain tensors to be used in 
				   Eq.5.46, Step-b */
				smatx[(ij*2+0)*2+0] = s11k[ij];
				smatx[(ij*2+0)*2+1] = s12k[ij];
				smatx[(ij*2+1)*2+0] = s12k[ij];
				smatx[(ij*2+1)*2+1] = s22k[ij];
				//
				ematx[(ij*2+0)*2+0] = e11k[ij];
				ematx[(ij*2+0)*2+1] = e12k[ij];
				ematx[(ij*2+1)*2+0] = e12k[ij];
				ematx[(ij*2+1)*2+1] = e22k[ij];
				//----- ----- ----- ----- ----- 
				
				//----- ----- ----- ----- ----- 
				for(int kk=0;kk<2;kk++){
					for(int ll=0;ll<2;ll++){
						for(int ii=0;ii<2;ii++){
							for(int jj=0;jj<2;jj++){
								/* Eq.5.46(b): new epsilon(zeta) = epsilon(zeta) - sum( gamma(zeta)*sigma(zeta) )
								   where gamma=tmatx, sigma=smatx
								   Note: tmatx is real part only */
								ematx[(ij*2+ii)*2+jj] -= tmatx[(((ij*2+kk)*2+ll)*2+ii)*2+jj]*smatx[(ij*2+kk)*2+ll];
							}//jj
						}//ii
					}//ll
				}//kk
				//----- ----- ----- ----- ----- 
				
				//----- ----- ----- ----- ----- 
				// Rearrange strain components using symmetry of strain tensor
				e11k[ij] = ematx[(ij*2+0)*2+0];
				e12k[ij] = ematx[(ij*2+0)*2+1];
				//e12k[ij] = ematx[(ij*2+1)*2+0];
				e22k[ij] = ematx[(ij*2+1)*2+1];
				//----- ----- ----- ----- ----- 
				
			}//Ny
		}//Nx
		
		hipMemcpy(e11k_d,e11k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e11k = e11k_h
		hipMemcpy(e22k_d,e22k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e22k = e22k_h
		hipMemcpy(e12k_d,e12k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e12k = e12k_h
		
		//From Fourier space to real space
		/* Take strain components from Fourier space back to
		   real space (inverse FFT), Step-c */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e11k_d, e11_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e22k_d, e22_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		hipfftExecC2C(iplan, e12k_d, e12_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(e11,e11_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11 = e11_h
		hipMemcpy(e22,e22_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22 = e22_h
		hipMemcpy(e12,e12_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12 = e12_h
		
		//Calculate stresses
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//
				e11[ii] /= (Nx*Ny);
				e22[ii] /= (Nx*Ny);
				e12[ii] /= (Nx*Ny);
				//
				/* s11[ii][0]=c11[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
						  +c12[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]));
				s22[ii][0]=c21[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
						  +c22[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]); */
				// c21[ii]=c12[ii], c22[ii]=c11[ii], etc
				//-----
				s11[ii] = c11[ii]*(ea[0]+e11[ii]-ei11[ii]-ed11[ii])
						 +c12[ii]*(ea[1]+e22[ii]-ei22[ii]-ed22[ii]);
				s22[ii] = c12[ii]*(ea[0]+e11[ii]-ei11[ii]-ed11[ii])
						 +c11[ii]*(ea[1]+e22[ii]-ei22[ii]-ed22[ii]);
				//
				/* s12[ii][0]=c44[ii]*(ea[2]+e12[ii][0]-ei12[ii]-ed12[ii])
						  +c44[ii]*(ea[2]+e21[ii][0]-ei21[ii]-ed21[ii]); */
				// e21[ii]=e12[ii], etc
				//-----
				s12[ii] = c44[ii]*(ea[2]+e12[ii]-ei12[ii]-ed12[ii])*2.0;
				//
				sum_stress[ii] = __real__ (  s11[ii] + s22[ii] + s12[ii] );
				normF = normF + sum_stress[ii]*sum_stress[ii];
			}
		}
		
		normF=sqrt(normF);
		
		if(iter==1){
			conver=fabs((normF-old_norm)/(old_norm));
			if(conver<=tolerance){
				break;
			}
		}
		old_norm=normF;
		
	}//end iter
	
	//strain energy
	//Calculate functional derivative of elastic energy
	// sum strain components
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			ii=i*Ny+j;
			
			//Calculate strain components
			et11=ea[0]+e11[ii]-ei11[ii]-ed11[ii];
			et22=ea[1]+e22[ii]-ei22[ii]-ed22[ii];
			et12=ea[2]+e12[ii]-ei12[ii]-ed12[ii];
			
			//Functional derivative of the elastic energy with respect to composition
			/* F=(1/2)*sigma[i][j]*(epsilon[i][j] - epsilon0[i][j])
			   sigma[i][j] = C[i][j][k][l]*(epsilon[k][l] - epsilon0[k][l])
			   epsilon0[i][j] is the position- and composition-dependent eigenstrains */
			/* dF/dc = dF/d(con) = (1/2)*( dCijkl/d(con)*etij*etkl + Cijkl*d(etij)/d(con)*etkl + Cijkl*etij*d(etkl)/d(con) )
			   dCijkl/d(con) = (cpijkl - cmijkl), d(etij)/d(con) = -d(eiij)/d(con) = -ei0 */
			// cp21=cp12, cp22=cp11, et21=et12, etc
			/* delsdc[ii][0]=0.5*( (cp11-cm11)*et11*et11 -c11[ii]*et11*ei0 -c11[ii]*et11*ei0
							   +(cp12-cm12)*et11*et22 -c12[ii]*et11*ei0 -c12[ii]*et22*ei0
							   //
							   +(cp21-cm21)*et22*et11 -c21[ii]*et22*ei0 -c21[ii]*et11*ei0
							   +(cp22-cm22)*et22*et22 -c22[ii]*et22*ei0 -c22[ii]*et22*ei0
							   //
							   +(cp44-cm44)*et12*et12 -c44[ii]*et12*ei0 -c44[ii]*et21*ei0
							   +(cp44-cm44)*et21*et21 -c44[ii]*et21*ei0 -c44[ii]*et12*ei0
							  ); */
			delsdc[ii]=0.5*(et11*( (cp12-cm12)*et22 + (cp11-cm11)*et11 - c12[ii]*ei0 - c11[ii]*ei0 )
							   -ei0*(     c12[ii]*et22 +     c11[ii]*et11 )
						      +et22*( (cp11-cm11)*et22 + (cp12-cm12)*et11 - c12[ii]*ei0 - c11[ii]*ei0 )
							   -ei0*(     c11[ii]*et22 +     c12[ii]*et11 )
							   +2.0*(cp44-cm44)*et12*et12 -4.0*ei0*c44[ii]*et12
							  );
			//delsdc[ii][1]=0.0;
		}
	}
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(s11_d);
	hipFree(s22_d);
	hipFree(s12_d);
	//
	hipFree(e11_d);
	hipFree(e22_d);
	hipFree(e12_d);
	//
	hipFree(s11k_d);
	hipFree(s22k_d);
	hipFree(s12k_d);
	//
	hipFree(e11k_d);
	hipFree(e22k_d);
	hipFree(e12k_d);
	//
	hipFree(ei11_d);
	hipFree(ei22_d);
	hipFree(ei12_d);
	//
	hipFree(c11_d);
	hipFree(c12_d);
	hipFree(c44_d);
	//
	hipFree(conr_d);
	//----- ----- ----- ----- ----- -----
	free(s11k);
	free(s22k);
	free(s12k);
	//
	free(e11k);
	free(e22k);
	free(e12k);
	//----- ----- ----- ----- ----- -----
	free(smatx);
	free(ematx);
	//
	free(sum_stress);
	//
	free(ei11);
	free(ei22);
	free(ei12);
	//
	free(c11);
	free(c12);
	free(c44);
	//----- ----- ----- ----- ----- -----
	
	return;
}