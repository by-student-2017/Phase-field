/* This function evaluates the derivative of elastic energy with
   respect to concentration. First, stress and strain values are
   solved with the iterative algorithm described earlier, 
   then derivative of elastic energy is evaluated for all grid points. */

#include <stdlib.h> //rand() and malloc
#include <math.h>
//#include <fftw3.h>

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cuComplex cufftComplex;

/* Variable and array list
  Nx: Number of grid points in the x-direction
  Ny: Number of grid points in the y-direction
  cm11: C11 component of elasticity matrix for matrix material
  cm12: C12 component of elasticity matrix for matrix material
  cm44: C44 component of elasticity matrix for matrix material
  cp11: C11 component of elasticity matrix for second phase
  cp12: C12 component of elasticity matrix for second phase
  cp44: C44 component of elasticity matrix for second phase
  ed11: Strain component of lattice defects
  ed22: Strain component of lattice defects
  ed12: Strain component of lattice defects
  ei0: Magnitude of eigenstrains
  ea[3]: Applied strains
  con[Nx][Ny]: Concentration
  s11[Nx][Ny]: Component of stress
  s22[Nx][Ny]: Component of stress
  s12[Nx][Ny]: Component of stress
  e11[Nx][Ny]: Component of strain
  e22[Nx][Ny]: Component of strain
  e12[Nx][Ny]: Component of strain
  delsdc[Nx][Ny]: Functional derivative of elastic energy
  tmatx[Nx][Ny][2][2][2][2]: Values of Green's tensor at all grid points (real part only)
*/

void solve_elasticity_2d(int Nx, int Ny,
	float *tmatx,
	float _Complex *s11, float _Complex *s22, float _Complex *s12,
	float _Complex *e11, float _Complex *e22, float _Complex *e12,
	float *ed11, float *ed22, float *ed12,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea,
	float ei0,
	float *con,  float _Complex *delsdc){
	
	//----- ----- ----- -----
	hipfftComplex *s11_d, *s22_d, *s12_d;
	hipMalloc((void**)&s11_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s22_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s12_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftComplex *e11_d, *e22_d, *e12_d;
	hipMalloc((void**)&e11_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e22_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e12_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	
	//----- ----- ----- -----
	hipfftComplex *s11k_d, *s22k_d, *s12k_d;
	hipMalloc((void**)&s11k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s22k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&s12k_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftComplex *e11k_d, *e22k_d, *e12k_d;
	hipMalloc((void**)&e11k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e22k_d, sizeof(hipfftComplex)*Nx*Ny);
	hipMalloc((void**)&e12k_d, sizeof(hipfftComplex)*Nx*Ny);
	//----- ----- ----- -----
	hipfftHandle plan, iplan;
	hipfftPlan2d(&plan,  Nx, Ny, HIPFFT_C2C);
	hipfftPlan2d(&iplan, Nx, Ny, HIPFFT_C2C);
	//----- ----- ----- -----
	
	float _Complex *s11k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *s22k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *s12k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	//
	float _Complex *e11k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *e22k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	float _Complex *e12k = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny ));
	
	//----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny;
	
	//fftw_complex *s11k, *s22k, *s12k;
	//s11k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//s22k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//s12k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	
	//fftw_plan plan_s11, plan_s22, plan_s12;
	// plan_s11  = fftw_plan_dft_2d(fftsizex, fftsizey, s11, s11k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	// plan_s22  = fftw_plan_dft_2d(fftsizex, fftsizey, s22, s22k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	// plan_s12  = fftw_plan_dft_2d(fftsizex, fftsizey, s12, s12k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//----- ----- ----- -----
	//fftw_complex *e11k, *e22k, *e12k;
	//e11k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//e22k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	//e12k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * fftsizex*fftsizey);
	
	//fftw_plan plan_e11, iplan_e11k;
	// plan_e11  = fftw_plan_dft_2d(fftsizex, fftsizey, e11, e11k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_e11k = fftw_plan_dft_2d(fftsizex, fftsizey, e11k, e11, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//
	//fftw_plan plan_e22, iplan_e22k;
	// plan_e22  = fftw_plan_dft_2d(fftsizex, fftsizey, e22, e22k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_e22k = fftw_plan_dft_2d(fftsizex, fftsizey, e22k, e22, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//
	//fftw_plan plan_e12, iplan_e12k;
	// plan_e12  = fftw_plan_dft_2d(fftsizex, fftsizey, e12, e12k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//iplan_e12k = fftw_plan_dft_2d(fftsizex, fftsizey, e12k, e12, FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	
	//float smatx_real[Nx][Ny][2][2];
	//float *smatx_real = (float *)malloc(sizeof(float)*( Nx*Ny*2*2 ));
	//float ematx_real[Nx][Ny][2][2];
	//float *ematx_real = (float *)malloc(sizeof(float)*( Nx*Ny*2*2 ));
	//
	//float smatx_imag[Nx][Ny][2][2];
	//float *smatx_imag = (float *)malloc(sizeof(float)*( Nx*Ny*2*2 ));
	//float ematx_imag[Nx][Ny][2][2];
	//float *ematx_imag = (float *)malloc(sizeof(float)*( Nx*Ny*2*2 ));
	//
	float _Complex *smatx = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny*2*2 ));
	float _Complex *ematx = (float _Complex *)malloc(sizeof(float _Complex)*( Nx*Ny*2*2 ));
	
	//float sum_stress[Nx][Ny];
	float *sum_stress = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	float old_norm=0.0;
	float normF=0.0;
	
	float conver=0.0;
	
	float _Complex et11 = 0.0;
	float _Complex et22 = 0.0;
	float _Complex et12 = 0.0;
	
	int ii, ij;
	
	//Maximum number of iteration steps
	int niter=10;
	
	//Tolerance value of convergence tests
	float tolerance=0.001;
	
	//float ei11[Nx][Ny];
	float *ei11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ei22[Nx][Ny];
	float *ei22 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float ei12[Nx][Ny];
	float *ei12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	//float c11[Nx][Ny];
	float  *c11 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float c12[Nx][Ny];
	float  *c12 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	//float c44[Nx][Ny];
	float  *c44 = (float *)malloc(sizeof(float)*( Nx*Ny ));
	
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			ii=i*Ny+j;
			
			//Calculate the eigenstrains
			ei11[ii]=ei0*con[ii];
			ei22[ii]=ei0*con[ii];
			ei12[ii]=0.0*con[ii];
			
			/* Calculate the effective elastic constants at 
			   the grid points based on the composition and
			   using Vegard's law */
			c11[ii]=con[ii]*cp11+(1.0-con[ii])*cm11;
			c12[ii]=con[ii]*cp12+(1.0-con[ii])*cm12;
			c44[ii]=con[ii]*cp44+(1.0-con[ii])*cm44;
		}
	}
	
	/* Solve stress and strain field with 
	   iterative algorithm given in the text */
	for(int iter=0;iter<niter;iter++){
		
		hipMemcpy(s11_d,s11,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s11 = s11_h
		hipMemcpy(s22_d,s22,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s22 = s22_h
		hipMemcpy(s12_d,s12,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //s12 = s12_h
		//
		hipMemcpy(e11_d,e11,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e11 = e11_h
		hipMemcpy(e22_d,e22,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e22 = e22_h
		hipMemcpy(e12_d,e12,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e12 = e12_h
		
		/* Take stress and strain components from real space to
		   Fourier space (forward FFT). Step-a */
		// stress
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//s11k=fft2(s11);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_s11); //fftw3
		//----- 
		hipfftExecC2C(plan, s11_d, s11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//s22k=fft2(s22);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_s22); //fftw3
		//----- 
		hipfftExecC2C(plan, s22_d, s22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//s12k=fft2(s12);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_s12); //fftw3
		//----- 
		hipfftExecC2C(plan, s12_d, s12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//
		// strain
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e11k=fft2(e11);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_e11); //fftw3
		//----- 
		hipfftExecC2C(plan, e11_d, e11k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e22k=fft2(e22);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_e22); //fftw3
		//----- 
		hipfftExecC2C(plan, e22_d, e22k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e12k=fft2(e12);         //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(plan_e12); //fftw3
		//----- 
		hipfftExecC2C(plan, e12_d, e12k_d, HIPFFT_FORWARD); //FFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(s11k,s11k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s11k = s11k_h
		hipMemcpy(s22k,s22k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s22k = s22k_h
		hipMemcpy(s12k,s12k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //s12k = s12k_h
		//
		hipMemcpy(e11k,e11k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11k = e11k_h
		hipMemcpy(e22k,e22k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22k = e22k_h
		hipMemcpy(e12k,e12k_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12k = e12k_h
		
		/* Form stress and strain tensors to be used in 
		   Eq.5.46, Step-b */
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//smatx_real[(ii*2+0)*2+0] = __real__ s11k[ii];
				//smatx_real[(ii*2+0)*2+1] = __real__ s12k[ii];
				//smatx_real[(ii*2+1)*2+0] = __real__ s12k[ii];
				//smatx_real[(ii*2+1)*2+1] = __real__ s22k[ii];
				//
				//smatx_imag[(ii*2+0)*2+0] = __imag__ s11k[ii];
				//smatx_imag[(ii*2+0)*2+1] = __imag__ s12k[ii];
				//smatx_imag[(ii*2+1)*2+0] = __imag__ s12k[ii];
				//smatx_imag[(ii*2+1)*2+1] = __imag__ s22k[ii];
				//
				//ematx_real[(ii*2+0)*2+0] = __real__ e11k[ii];
				//ematx_real[(ii*2+0)*2+1] = __real__ e12k[ii];
				//ematx_real[(ii*2+1)*2+0] = __real__ e12k[ii];
				//ematx_real[(ii*2+1)*2+1] = __real__ e22k[ii];
				//
				//ematx_imag[(ii*2+0)*2+0] = __imag__ e11k[ii];
				//ematx_imag[(ii*2+0)*2+1] = __imag__ e12k[ii];
				//ematx_imag[(ii*2+1)*2+0] = __imag__ e12k[ii];
				//ematx_imag[(ii*2+1)*2+1] = __imag__ e22k[ii];
				//
				smatx[(ii*2+0)*2+0] = s11k[ii];
				smatx[(ii*2+0)*2+1] = s12k[ii];
				smatx[(ii*2+1)*2+0] = s12k[ii];
				smatx[(ii*2+1)*2+1] = s22k[ii];
				//
				ematx[(ii*2+0)*2+0] = e11k[ii];
				ematx[(ii*2+0)*2+1] = e12k[ii];
				ematx[(ii*2+1)*2+0] = e12k[ii];
				ematx[(ii*2+1)*2+1] = e22k[ii];
			}
		}
		
		//Green operator
		// Calculate strain tensor, Eq.5.46, Step-b
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ij=i*Ny+j;
				//
				for(int kk=0;kk<2;kk++){
					for(int ll=0;ll<2;ll++){
						for(int ii=0;ii<2;ii++){
							for(int jj=0;jj<2;jj++){
								/* Eq.5.46(b): new epsilon(zeta) = epsilon(zeta) - sum( gamma(zeta)*sigma(zeta) )
								   where gamma=tmatx, sigma=smatx
								   Note: tmatx is real part only */
								//ematx_real[(ij*2+ii)*2+jj]=ematx_real[(ij*2+ii)*2+jj]
								//	-tmatx[(((ij*2+kk)*2+ll)*2+ii)*2+jj]*smatx_real[(ij*2+kk)*2+ll];
								//
								//ematx_imag[(ij*2+ii)*2+jj]=ematx_imag[(ij*2+ii)*2+jj]
								//	-tmatx[(((ij*2+kk)*2+ll)*2+ii)*2+jj]*smatx_imag[(ij*2+kk)*2+ll];
								ematx[(ij*2+ii)*2+jj]=ematx[(ij*2+ii)*2+jj]
									-tmatx[(((ij*2+kk)*2+ll)*2+ii)*2+jj]*smatx[(ij*2+kk)*2+ll];
							}//jj
						}//ii
					}//ll
				}//kk
				//
			}//Ny
		}//Nx
		
		// Rearrange strain components using symmetry of strain tensor
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				
				//e11k[ii][0]=ematx_real[(ii*2+0)*2+0];
				//e12k[ii][0]=ematx_real[(ii*2+0)*2+1];
				//e12k[ii][0]=ematx_real[(ii*2+1)*2+0];
				//e22k[ii][0]=ematx_real[(ii*2+1)*2+1];
				//
				//e11k[ii][1]=ematx_imag[(ii*2+0)*2+0];
				//e12k[ii][1]=ematx_imag[(ii*2+0)*2+1];
				//e12k[ii][1]=ematx_imag[(ii*2+1)*2+0];
				//e22k[ii][1]=ematx_imag[(ii*2+1)*2+1];
				//
				//e11k[ii] = ematx_real[(ii*2+0)*2+0] + ematx_imag[(ii*2+0)*2+0]*I;
				//e12k[ii] = ematx_real[(ii*2+0)*2+1] + ematx_imag[(ii*2+0)*2+1]*I;
				//e12k[ii] = ematx_real[(ii*2+1)*2+0] + ematx_imag[(ii*2+1)*2+0]*I;
				//e22k[ii] = ematx_real[(ii*2+1)*2+1] + ematx_imag[(ii*2+1)*2+1]*I;
				//
				e11k[ii] = ematx[(ii*2+0)*2+0];
				e12k[ii] = ematx[(ii*2+0)*2+1];
				//e12k[ii] = ematx[(ii*2+1)*2+0];
				e22k[ii] = ematx[(ii*2+1)*2+1];
			}
		}
		
		hipMemcpy(e11k_d,e11k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e11k = e11k_h
		hipMemcpy(e22k_d,e22k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e22k = e22k_h
		hipMemcpy(e12k_d,e12k,Nx*Ny*sizeof(float _Complex),hipMemcpyHostToDevice); //e12k = e12k_h
		
		//From Fourier space to real space
		/* Take strain components from Fourier space back to
		   real space (inverse FFT), Step-c */
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e11=real(ifft2(e11k));    //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(iplan_e11k); //fftw3
		//-----
		hipfftExecC2C(iplan, e11k_d, e11_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e22=real(ifft2(e22k));    //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(iplan_e22k); //fftw3
		//-----
		hipfftExecC2C(iplan, e22k_d, e22_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		//e12=real(ifft2(e12k));    //Octave or Matlab
		//----- ----- ----- -----
		//fftw_execute(iplan_e12k); //fftw3
		//-----
		hipfftExecC2C(iplan, e12k_d, e12_d, HIPFFT_BACKWARD); //IFFT
		hipDeviceSynchronize();
		//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
		
		hipMemcpy(e11,e11_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e11 = e11_h
		hipMemcpy(e22,e22_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e22 = e22_h
		hipMemcpy(e12,e12_d,Nx*Ny*sizeof(float _Complex),hipMemcpyDeviceToHost); //e12 = e12_h
		
		//Calculate stresses
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				//
				//e11[ii][0]=e11[ii][0]/(fftsizex*fftsizey);
				//e22[ii][0]=e22[ii][0]/(fftsizex*fftsizey);
				//e12[ii][0]=e12[ii][0]/(fftsizex*fftsizey);
				//
				//e11[ii][1]=e11[ii][1]/(fftsizex*fftsizey);
				//e22[ii][1]=e22[ii][1]/(fftsizex*fftsizey);
				//e12[ii][1]=e12[ii][1]/(fftsizex*fftsizey);
				//
				e11[ii] = e11[ii]/(Nx*Ny);
				e22[ii] = e22[ii]/(Nx*Ny);
				e12[ii] = e12[ii]/(Nx*Ny);
				//
				/* s11[ii][0]=c11[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
						  +c12[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]));
				s22[ii][0]=c21[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
						  +c22[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]); */
				// c21[ii]=c12[ii], c22[ii]=c11[ii], etc
				//-----
				//s11[ii][0]=c11[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
				//		  +c12[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]);
				//s22[ii][0]=c12[ii]*(ea[0]+e11[ii][0]-ei11[ii]-ed11[ii])
				//		  +c11[ii]*(ea[1]+e22[ii][0]-ei22[ii]-ed22[ii]);
				//s11[ii][1]=0.0;
				//s22[ii][1]=0.0;
				//
				/* s12[ii][0]=c44[ii]*(ea[2]+e12[ii][0]-ei12[ii]-ed12[ii])
						  +c44[ii]*(ea[2]+e21[ii][0]-ei21[ii]-ed21[ii]); */
				// e21[ii]=e12[ii], etc
				//-----
				//s12[ii][0]=c44[ii]*(ea[2]+e12[ii][0]-ei12[ii]-ed12[ii])*2.0;
				//
				//s12[ii][1]=0.0;
				//
				s11[ii] = c11[ii]*(ea[0]+e11[ii]-ei11[ii]-ed11[ii])
						 +c12[ii]*(ea[1]+e22[ii]-ei22[ii]-ed22[ii]);
				s22[ii] = c12[ii]*(ea[0]+e11[ii]-ei11[ii]-ed11[ii])
						 +c11[ii]*(ea[1]+e22[ii]-ei22[ii]-ed22[ii]);
				s12[ii] = c44[ii]*(ea[2]+e12[ii]-ei12[ii]-ed12[ii])*2.0;
			}
		}
		
		//check convergence
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				sum_stress[ii] = ( __real__ s11[ii] + __real__ s22[ii] + __real__ s12[ii] );
			}
		}
		
		//normF=norm(sum_stress,2.0);
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				ii=i*Ny+j;
				normF = normF + sum_stress[ii]*sum_stress[ii];
			}
		}
		normF=sqrt(normF);
		
		if(iter==1){
			conver=fabs((normF-old_norm)/(old_norm));
			if(conver<=tolerance){
				break;
			}
		}
		old_norm=normF;
		
	}//end iter
	
	//strain energy
	//Calculate functional derivative of elastic energy
	// sum strain components
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			ii=i*Ny+j;
			
			//Calculate strain components
			et11=ea[0]+e11[ii]-ei11[ii]-ed11[ii];
			et22=ea[1]+e22[ii]-ei22[ii]-ed22[ii];
			et12=ea[2]+e12[ii]-ei12[ii]-ed12[ii];
			
			//Functional derivative of the elastic energy with respect to composition
			/* F=(1/2)*sigma[i][j]*(epsilon[i][j] - epsilon0[i][j])
			   sigma[i][j] = C[i][j][k][l]*(epsilon[k][l] - epsilon0[k][l])
			   epsilon0[i][j] is the position- and composition-dependent eigenstrains */
			/* dF/dc = dF/d(con) = (1/2)*( dCijkl/d(con)*etij*etkl + Cijkl*d(etij)/d(con)*etkl + Cijkl*etij*d(etkl)/d(con) )
			   dCijkl/d(con) = (cpijkl - cmijkl), d(etij)/d(con) = -d(eiij)/d(con) = -ei0 */
			// cp21=cp12, cp22=cp11, et21=et12, etc
			/* delsdc[ii][0]=0.5*( (cp11-cm11)*et11*et11 -c11[ii]*et11*ei0 -c11[ii]*et11*ei0
							   +(cp12-cm12)*et11*et22 -c12[ii]*et11*ei0 -c12[ii]*et22*ei0
							   //
							   +(cp21-cm21)*et22*et11 -c21[ii]*et22*ei0 -c21[ii]*et11*ei0
							   +(cp22-cm22)*et22*et22 -c22[ii]*et22*ei0 -c22[ii]*et22*ei0
							   //
							   +(cp44-cm44)*et12*et12 -c44[ii]*et12*ei0 -c44[ii]*et21*ei0
							   +(cp44-cm44)*et21*et21 -c44[ii]*et21*ei0 -c44[ii]*et12*ei0
							  ); */
			delsdc[ii]=0.5*(et11*( (cp12-cm12)*et22 + (cp11-cm11)*et11 - c12[ii]*ei0 - c11[ii]*ei0 )
							   -ei0*(     c12[ii]*et22 +     c11[ii]*et11 )
						      +et22*( (cp11-cm11)*et22 + (cp12-cm12)*et11 - c12[ii]*ei0 - c11[ii]*ei0 )
							   -ei0*(     c11[ii]*et22 +     c12[ii]*et11 )
							   +2.0*(cp44-cm44)*et12*et12 -4.0*ei0*c44[ii]*et12
							  );
			//delsdc[ii][1]=0.0;
		}
	}
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(s11k_d);
	hipFree(s22k_d);
	hipFree(s12k_d);
	//
	hipFree(e11k_d);
	hipFree(e22k_d);
	hipFree(e12k_d);
	//----- ----- ----- ----- ----- -----
	free(s11k);
	free(s22k);
	free(s12k);
	//
	free(e11k);
	free(e22k);
	free(e12k);
	//----- ----- ----- ----- ----- -----
	//free(smatx_real);
	//free(smatx_imag);
	//free(ematx_real);
	//free(ematx_imag);
	free(smatx);
	free(ematx);
	//
	free(sum_stress);
	//
	free(ei11);
	free(ei22);
	free(ei12);
	//
	free(c11);
	free(c12);
	free(c44);
	//----- ----- ----- ----- ----- -----
	
	return;
}