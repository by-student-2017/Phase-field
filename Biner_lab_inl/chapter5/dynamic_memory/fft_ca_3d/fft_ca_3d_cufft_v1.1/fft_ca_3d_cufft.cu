/* 3D semi-implicit spectral phase-field code
   for solving Allen-Cahn equation */

/* This program solves conserved phase-field equation with
   Fourier spactral method by taking into account the effects of
   elastic inhomogeneities and lattice defects, based on
   solution of stress-strain fields ith Green's tensor and
   Fourier transformations. the time integration is
   carried out by using semi-implicit time machining scheme. */

#include <stdio.h>
#include <stdlib.h> //rand() and malloc
#include <math.h> //mod() and -lm
#include <time.h>

//#include <fftw3.h>
//gcc test.c -lfftw3
//#include <mpi.h> //mpi version
//#include <fftw3-mpi.h> //mpi version

/* Memo for complex type
  "float __complex__ " is old version
  "float _Complex " is new version */
//#include <complex.h>
//#include <cuComplex.h>
//#define _Complex_I (1.0iF)
//#define I i
//#undef i
//#undef j
//Ref: http://nalab.mind.meiji.ac.jp/~mk/labo/text/complex-c.pdf (Japanese)

#include <hip/hip_runtime.h>   //GPU
/* #include <cuda.h> or
  #include "cuda_runtime.h"
  #include "device_launch_parameters.h" */
//----- ----- -----
#include <hipfft/hipfft.h> //FFT (GPU)

//typedef float cufftReal;
//typedef cu_Complex cufftComplex;

//----- ----- ----- ----- ----- ----- -----
void init_grain_micro_3d(int Nx, int Ny, int Nz,
	float dx, float dy, float dz,
	int iflag, int ngrain,
	float *etas, int *glist);
//----- ----- -----
float free_energy_fd_ca_3d(int i, int j, int k,
	int Nx, int Ny, int Nz,
	int ngrain, float *etas, float *eta, int igrain);
//----- ----- -----
void prepare_fft_3d(int Nx, int Ny, int Nz, 
	float dx, float dy, float dz,
	float *kx, float *ky, float *kz, 
	float *k2, float *k4);
//----- ----- -----
void write_vtk_grid_values_3D(int nx, int ny, int nz, 
	float dx, float dy, float dz,
	int istep, float *data1);
//----- ----- ----- ----- ----- ----- -----

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel_semi_implicit_time_integration(
	int   Nx,
	int   Ny,
	int   Nz,
	float dtime,
	float coefA,
	float mobil,
	float grcoef,
	float *k2_d,
	hipfftComplex *etak_d,
	hipfftComplex *dfdetak_d
){
	int j, jx, jy, jz;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	jz = blockDim.z*blockIdx.z + threadIdx.z; //<-GPU | CPU -> for(jz=0; jz<nz; jz++){
	j  = (jz*Ny + jy)*Nx + jx; //j = nx*ny*jz + nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	float denom;
	//
	denom = 1.0 + dtime*coefA*mobil*grcoef*k2_d[j];
	etak_d[j].x = ( etak_d[j].x - (dtime*mobil*dfdetak_d[j].x) )/denom; //real part
	etak_d[j].y = ( etak_d[j].y - (dtime*mobil*dfdetak_d[j].y) )/denom; //imaginary part
	
	/* Note: cufftComplex changed between CUDA 1.0 and 1.1.
	dout[idx].x =  d_signal[idx].y; <- dout[idx][0] = d_signal[idx][1];
	dout[idx].y = -d_signal[idx].x; <- dout[idx][1] = d_signal[idx][0]*(-1.0);
	Ref: https://forums.developer.nvidia.com/t/using-cufftcomplex-type-inside-a-kernel-does-it-work/4039 */
}

int main(){
	clock_t start, end;
	float compute_time;
	
	//get initial wall time
	//(Get initial wall clock time beginning of the execution)
	start = clock();
	
	//open an output file for writing total bulk energy values
	FILE *out2=fopen("area_frac.out","w");
	
	//simulation cell parameters (These values are dummy)
	int Nx=64; //Number of grid points in the x-direction
	int Ny=64; //Number of grid points in the y-direction
	int Nz=2; //Number of grid points in the y-direction
	
	int BSX=8; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	int BSY=8; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	int BSZ=2; //Number of threads, 2^n=<32, BSX*BSY*BSZ <= 1024
	
	int ngrain=2;
	
	//The distance between two grid points in x,y-direction
	float dx=0.5; //Grid spacing between two grid pints in x-direction
	float dy=0.5; //Grid spacing between two grid pints in y-direction
	float dz=0.5; //Grid spacing between two grid pints in z-direction
	
	//time integration parameters
	int nstep=100000; //Number of time integration steps
	int nprint=100; //Output frequency to write the results to file
	float dtime=0.005; //Time increment for the numerical integration
	float ttime=0.0;   //Total time
	float coefA=1.0;
	
	//material specific parameters
	
	//Initial concentrations of alloying elements
	float mobil=5.0;  //The value of mobility coefficient
	float grcoef=0.1; //The value of gradient energy coefficients [J(nm)^2/mol]
	
	//----- ----- ----- -----
	int ijk; //ijk=(i*Ny+j)*Nz+k;
	//----- ----- ----- -----
	
	/* Generate initial grain microstructure
	   iflag=1 is for bi-crystal and
	   iflag=2 is for polycrystal */
	int iflag=2;
	if(iflag==2){
		//read polycrystal microstructure
		FILE *in=fopen("grain_25.inp","r");
		fscanf(in,"%5d %5d %5d %5d ",&Nx,&Ny,&Nz,&ngrain);
		fclose(in);
	}
	//----- ----- ----- -----
	int NxNyNz=Nx*Ny*Nz; //Total number of grid points in the simulation cell
	float *etas = (float *)malloc(sizeof(float)*( NxNyNz*ngrain ));
	int   *glist = (int *)malloc(sizeof(int)*( ngrain ));
	//----- ----- ----- -----
	if(iflag==2){
		FILE *in=fopen("grain_25.inp","r");
		fscanf(in,"%5d %5d %5d %5d ",&Nx,&Ny,&Nz,&ngrain);
		//
		int nline=1;
		int ri, rj, rk, rigrain;
		float reta;
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					fscanf(in,"%5d %5d %5d %5d %f",&ri,&rj,&rk,&rigrain,&reta);
					//----- ----- ----- -----
					if( i != ri ){ printf("Don't match x data, Line %5d \n",nline); exit(1); }
					if( j != rj ){ printf("Don't match y data, LIne %5d \n",nline); exit(1); }
					if( k != rk ){ printf("Don't match z data, LIne %5d \n",nline); exit(1); }
					nline = nline + 1;
					//----- ----- ----- -----
					ijk=(i*Ny+j)*Nz+k;
					etas[ijk*ngrain+rigrain]=reta;
					//----- ----- ----- -----
				}
			}
		}
		fclose(in);
		//initialize glist
		for(int igrain=0;igrain<ngrain;igrain++){
			glist[igrain]=1.0;
		}
	}
	if(iflag==1){ init_grain_micro_3d(Nx,Ny,Nz,dx,dy,dz,iflag,ngrain,etas,glist); }
	
	//----- ----- ----- ----- ----- -----
	//const int fftsizex = Nx, fftsizey = Ny, fftsizez = Nz;
	//
	hipfftComplex *eta_d, *dfdeta_d;
	hipMalloc((void**)&eta_d,     sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&dfdeta_d,  sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftComplex *etak_d, *dfdetak_d;
	hipMalloc((void**)&etak_d,    sizeof(hipfftComplex)*NxNyNz);
	hipMalloc((void**)&dfdetak_d, sizeof(hipfftComplex)*NxNyNz);
	//
	hipfftHandle plan, iplan;
	hipfftPlan3d(&plan,  Nx, Ny, Nz, HIPFFT_C2C);
	hipfftPlan3d(&iplan, Nx, Ny, Nz, HIPFFT_C2C);
	//----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- ----- ----- -----
	//float kx[Nx];
	float *kx = (float *)malloc(sizeof(float)*( Nx ));
	//float ky[Ny];
	float *ky = (float *)malloc(sizeof(float)*( Ny ));
	//float kz[Ny];
	float *kz = (float *)malloc(sizeof(float)*( Nz ));
	//float k2[Nx][Ny][Nz];
	float *k2 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float k4[Nx][Ny][Nz];
	float *k4 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- ----- ----- ----- -----
	
	//prepare fft (output: kx,ky,kz,k2,k4)
	prepare_fft_3d(Nx,Ny,Nz,dx,dy,dz,kx,ky,kz,k2,k4); //get FFT coefficients
	
	//----- for cufft
	float *k2_d, *k4_d;
	k2_d  = (float *)malloc(Nx*Ny*sizeof(float));
	k4_d  = (float *)malloc(Nx*Ny*sizeof(float));
	hipMalloc((void**)&k2_d ,Nx*Ny*sizeof(float));
	hipMalloc((void**)&k4_d ,Nx*Ny*sizeof(float));
	hipMemcpy(k2_d,k2,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k2 = k2_h
	hipMemcpy(k4_d,k4,Nx*Ny*sizeof(float),hipMemcpyHostToDevice); //k4 = k4_h
	//----- ----- ----- -----
	
	float *eta  = (float *)malloc(sizeof(float)*( NxNyNz ));
	float *eta2 = (float *)malloc(sizeof(float)*( NxNyNz ));
	
	//----- ----- ----- -----
	float denom;
	float _Complex numer;
	//----- ----- ----- -----
	float grain_sum;
	//----- ----- ----- -----
	int ncount;
	//----- ----- ----- -----
	
	float _Complex *dfdetac = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	float _Complex *etac    = (float _Complex *)malloc(sizeof(float _Complex)*( NxNyNz ));
	
	int bsx=BSX, bsy=BSY, bsz=BSZ;     //Number of threads
	dim3 blocks(Nx/bsx,Ny/bsy,Nz/bsz); //nx*ny*nz = blocks * threads
	dim3 threads(bsx,bsy,bsz);         //bsx*bsy*bsz <= 1024
	
	//evolve (Time evolution of microstructure)
	for(int istep=0;istep<=nstep;istep++){
		
		//Update the total time
		ttime=ttime+dtime;
		
		//Loop over each grain
		for(int igrain=0;igrain<ngrain;igrain++){
			
			/* If glist is equal to one, which indicates that
			   the current grain area fraction is greater than 0.001,
			   continue the calculation. Otherwise, the current grain
			   does not exist anymore */
			if(glist[igrain]==1){
				
				/* Assign order parameters to temporary array eta[Nx][Ny] from
				   the common array etas[Nx][Ny][ngrain] for the current grain */
				for(int i=0;i<Nx;i++){
					for(int j=0;j<Ny;j++){
						for(int k=0;k<Nz;k++){
							ijk=(i*Ny+j)*Nz+k;
							eta[ijk] = etas[ijk*ngrain+igrain];
							//-----
							etac[ijk] = eta[ijk];
						}
					}
				}
				
				//derivative of free energy
				//Calculate the derivative of elastic energy
				for(int i=0;i<Nx;i++){
					for(int j=0;j<Ny;j++){
						for(int k=0;k<Nz;k++){
							ijk=(i*Ny+j)*Nz+k;
							dfdetac[ijk] = free_energy_fd_ca_3d(i,j,k,Nx,Ny,Nz,ngrain,etas,eta,igrain);
						}
					}
				}
				
				hipMemcpy(dfdeta_d,dfdetac,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //dfdetac = dfdetac_h
				hipMemcpy(eta_d,etac,NxNyNz*sizeof(float _Complex),hipMemcpyHostToDevice); //etac = etac_h
				
				/* Take the values of concentration, derivative of free energy and
				   derivative of elastic energy from real space to Fourier space (forward FFT) */
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				hipfftExecC2C(plan, eta_d, etak_d, HIPFFT_FORWARD); //FFT
				hipDeviceSynchronize();
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				hipfftExecC2C(plan, dfdeta_d, dfdetak_d, HIPFFT_FORWARD); //FFT
				hipDeviceSynchronize();
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				
				/* Semi-implicit time integration of concentration field at
				   Fourier space (Eq.5.14) */
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- on cuda
				Kernel_semi_implicit_time_integration<<<blocks, threads>>>(Nx,Ny,Nz,
					dtime,coefA,mobil,grcoef,
					k2_d,
					etak_d,dfdetak_d);
				hipDeviceSynchronize();
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				
				/* Take concentration field from Fourier space back to
				   real space (inverse FFT) */
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				hipfftExecC2C(iplan, etak_d, eta_d, HIPFFT_BACKWARD); //IFFT
				hipDeviceSynchronize();
				//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
				
				hipMemcpy(etac,eta_d,NxNyNz*sizeof(float _Complex),hipMemcpyDeviceToHost); //etac = etac_h
				
				//for small deviations
				// For small deviations from max and min values, reset the limits
				grain_sum=0.0;
				for(int i=0;i<Nx;i++){
					for(int j=0;j<Ny;j++){
						for(int k=0;k<Nz;k++){
							ijk=(i*Ny+j)*Nz+k;
							//----- ----- ----- -----
							//eta[ijk] =  eta[ijk]/(NxNyNz);
							eta[ijk] = ( __real__ etac[ijk] )/(NxNyNz);
							//eta[ijk] =  creal(etac[ijk])/(NxNyNz); //For #include <_Complex.h>
							//----- ----- ----- -----
							if(eta[ijk]>=0.9999){
								eta[ijk]=0.9999;
							}
							if(eta[ijk]<=0.0001){
								eta[ijk]=0.0001;
							}
							//----- ----- ----- -----
							/* Calculate the total area of the current grain,
							   also return the order parameter values from
							   the temporary array eta[Nx][Ny] to 
							   common array etas[Nx][Ny][ngrain] */
							grain_sum = grain_sum + eta[ijk];
							etas[ijk*ngrain+igrain] = eta[ijk];
							//----- ----- ----- -----
						}
					}
				}
				
				//Check volume fraction of current grain
				/* Check the area fraction of the current grain.
				   If it is less than 0.001, set the its value in 
				   glist[ngrain] as zero which indicates that it is extinct. 
				   Also print message "grain # is eliminated" to screen. */
				grain_sum = grain_sum/NxNyNz;
				
				if(grain_sum<=0.001){
					glist[igrain]=0;
					printf("grain: No. %3d is eliminated \n",igrain);
				}
				
			}//end if(glist
		}//end igrain
		
		//print results
		// If print frequency reached, print the results to file
		if(fmod(istep,nprint)==0){
			//write vtk file & calculate are function of grains
			/* Prepare the data to be written to vtk file and
			   calculate the area fraction of each grain and
			   print them to file area_fract.out. */
			for(int i=0;i<Nx;i++){
				for(int j=0;j<Ny;j++){
					for(int k=0;k<Nz;k++){
						ijk=(i*Ny+j)*Nz+k;
						eta2[ijk]=0.0;
					}
				}
			}
			fprintf(out2, "%14.6e ",ttime);
			
			for(int igrain=0;igrain<ngrain;igrain++){
				ncount=0;
				for(int i=0;i<Nx;i++){
					for(int j=0;j<Ny;j++){
						for(int k=0;k<Nz;k++){
							ijk=(i*Ny+j)*Nz+k;
							//eta2[ijk]=eta2[ijk]+etas[ijk*ngrain+igrain]*etas[ijk*ngrain+igrain];
							eta2[ijk]=eta2[ijk]+etas[ijk*ngrain+igrain]*etas[ijk*ngrain+igrain]*igrain;
							if(etas[ijk*ngrain+igrain]>=0.5){
								ncount=ncount+1;
							}//end if
						}//end for(k
					}//end for(j
				}//end for(i
				ncount=ncount/NxNyNz;
				fprintf(out2, "%5d ",ncount);
			}//end igrain
			fprintf(out2, "\n");
			
			//write vtk file
			/* Write the results in vtk format for contour plots
			   to be viewed by using Paraview */
			write_vtk_grid_values_3D(Nx,Ny,Nz,dx,dy,dz,istep,eta2);
			
			printf("done step: %5d \n",istep);
		}//end if
	}//end for(istep
	
	//calculate the execution time and print it
	/* Calculate the compute time and print it to screen */
	end = clock();
	compute_time = ((float) (end - start)) / CLOCKS_PER_SEC;
	printf("Compute Time: %f \n", compute_time);
	
	//----- ----- ----- ----- ----- -----
	hipfftDestroy(plan);
	hipfftDestroy(iplan);
	//----- ----- ----- ----- ----- -----
	hipFree(eta_d);
	hipFree(dfdeta_d);
	//
	hipFree(etak_d);
	hipFree(dfdetak_d);
	//----- ----- ----- ----- ----- -----
	free(kx);
	free(ky);
	free(k2);
	free(k4);
	//
	free(etas);
	free(glist);
	free(eta2);
	free(eta);
	//
	free(etac);
	free(dfdetac);
	//----- ----- ----- ----- ----- -----
}
