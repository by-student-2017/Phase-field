#include "hip/hip_runtime.h"
/* This function evaluates the derivative of elastic energy with
   respect to concentration. First, stress and strain values are
   solved with the iterative algorithm described earlier, 
   then derivative of elastic energy is evaluated for all grid points. */

#include <stdlib.h> //rand() and malloc
#include <math.h>
#include <fftw3.h>

/* Variable and array list
  Nx: Number of grid points in the x-direction
  Ny: Number of grid points in the y-direction
  Nz: Number of grid points in the z-direction
  cm11: C11 component of elasticity matrix for matrix material
  cm12: C12 component of elasticity matrix for matrix material
  cm44: C44 component of elasticity matrix for matrix material
  cp11: C11 component of elasticity matrix for second phase
  cp12: C12 component of elasticity matrix for second phase
  cp44: C44 component of elasticity matrix for second phase
  ed11[Nx][Ny][Nz] to ed13[Nx][Ny][Nz]: Strain component of lattice defects
  ea[6]: Applied strains
  con[Nx][Ny][Nz]: Concentration
  s11[Nx][Ny][Nz] to s13[Nx][Ny][Nz]: Component of stress
  e11[Nx][Ny][Nz] to e13[Nx][Ny][Nz]: Component of strain
  delsdc[Nx][Ny][Nz]: Functional derivative of elastic energy
  //
  omeg11[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg22[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg33[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg12[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg23[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  omeg13[Nx][Ny][Nz]: Coefficient needed for the Green's tensor
  tmatx[3][3][3][3]: Green's tensor at i,j,k grid point (real part only)
*/

void green_tensor2_3D(int Nx, int Ny, int Nz,
	float *kx, float *ky, float *kz,
	float *omeg11, float *omeg22, float *omeg33,
	float *omeg12, float *omeg23, float *omeg13,
	int i, int j, int k,
	float *tmatx);

void solve_elasticity_3d(int Nx, int Ny, int Nz,
	float *kx, float *ky, float *kz,
	float *omeg11, float *omeg22, float *omeg33,
	float *omeg12, float *omeg23, float *omeg13,
	fftw_complex *s11, fftw_complex *s22, fftw_complex *s33,
	fftw_complex *s12, fftw_complex *s23, fftw_complex *s13,
	fftw_complex *e11, fftw_complex *e22, fftw_complex *e33,
	fftw_complex *e12, fftw_complex *e23, fftw_complex *e13,
	float *ed11, float *ed22, float *ed33,
	float *ed12, float *ed23, float *ed13,
	float cm11, float cm12, float cm44,
	float cp11, float cp12, float cp44,
	float *ea, float ei0,
	float *con, float *delsdc){
	
	//----- ----- ----- -----
	const int fftsizex = Nx, fftsizey = Ny, fftsizez = Nz;
	const int scale=fftsizex*fftsizey*fftsizez;
	float fftw3d_scale = (float)scale;
	
	//stress (head s series) components
	//----- ----- ----- -----
	fftw_complex *s11k, *s22k, *s33k;
	 s11k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 s22k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 s33k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	//
	fftw_complex *s12k, *s23k, *s13k;
	 s12k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 s23k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 s13k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	//----- ----- ----- -----
	fftw_plan plan_s11, plan_s22, plan_s33;
	 plan_s11  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s11, s11k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	 plan_s22  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s22, s22k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	 plan_s33  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s33, s33k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//
	fftw_plan plan_s12, plan_s23, plan_s13;
	 plan_s12  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s12, s12k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	 plan_s23  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s23, s23k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	 plan_s13  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, s13, s13k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	//----- ----- ----- -----
	
	//strain (head e series) components
	//----- ----- ----- -----
	fftw_complex *e11k, *e22k, *e33k;
	 e11k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 e22k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 e33k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	//
	fftw_complex *e12k, *e23k, *e13k;
	 e12k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 e23k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	 e13k = (fftw_complex *)fftw_malloc(sizeof(fftw_complex) * scale);
	//----- ----- ----- -----
	fftw_plan plan_e11, iplan_e11k;
	 plan_e11  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e11, e11k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e11k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e11k,e11,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	fftw_plan plan_e22, iplan_e22k;
	 plan_e22  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e22, e22k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e22k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e22k,e22,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	fftw_plan plan_e33, iplan_e33k;
	 plan_e33  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e33, e33k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e33k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e33k,e33,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	//----- ----- ----- -----
	fftw_plan plan_e12, iplan_e12k;
	 plan_e12  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e12, e12k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e12k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e12k,e12,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	fftw_plan plan_e23, iplan_e23k;
	 plan_e23  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e23, e23k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e23k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e23k,e23,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	fftw_plan plan_e13, iplan_e13k;
	 plan_e13  = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e13, e13k, FFTW_FORWARD,  FFTW_ESTIMATE); //For forward FFT
	iplan_e13k = fftw_plan_dft_3d(fftsizex, fftsizey, fftsizez, e13k,e13,  FFTW_BACKWARD, FFTW_ESTIMATE); //For inverse FFT
	//----- ----- ----- -----
	
	int NxNyNz=Nx*Ny*Nz;
	
	//----- ----- ----- ----- ----- ----- -----
	// eigenstrains (head ei series) components
	//float ei11[Nx][Ny][Nz];
	float *ei11 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei22[Nx][Ny][Nz];
	float *ei22 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei33[Nx][Ny][Nz];
	float *ei33 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//
	//float ei12[Nx][Ny][Nz];
	float *ei12 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei23[Nx][Ny][Nz];
	float *ei23 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float ei13[Nx][Ny][Nz];
	float *ei13 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- -----
	// elastic modulus components
	//float c11[Nx][Ny][Nz];
	float  *c11 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float c12[Nx][Ny][Nz];
	float  *c12 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//float c44[Nx][Ny][Nz];
	float  *c44 = (float *)malloc(sizeof(float)*( NxNyNz ));
	//----- ----- ----- ----- ----- ----- -----
	
	//----- ----- -----
	int ijk;
	int klij; //For tmatx
	//----- ----- -----
	
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				ijk=(i*Ny+j)*Nz+k;
				//----- ----- ----- ----- -----
				// Calculate the eigenstrains (head ei series)
				ei11[ijk] = ei0*con[ijk];
				ei22[ijk] = ei0*con[ijk];
				ei33[ijk] = ei0*con[ijk];
				//
				ei12[ijk] = 0.0*con[ijk];
				ei23[ijk] = 0.0*con[ijk];
				ei13[ijk] = 0.0*con[ijk];
				//----- ----- ----- ----- -----
				
				/* Calculate the effective elastic constants at 
				   the grid points based on the composition and
				   using Vegard's law */
				//----- ----- ----- ----- -----
				c11[ijk] = con[ijk]*cp11 + (1.0-con[ijk])*cm11;
				c12[ijk] = con[ijk]*cp12 + (1.0-con[ijk])*cm12;
				c44[ijk] = con[ijk]*cp44 + (1.0-con[ijk])*cm44;
				//----- ----- ----- ----- -----
			}
		}
	}
	
	/* Note: elastic modulus in this case. */
	//----- ----- ----- -----
	//float c22=c33=c11;
	//float c21=c12;
	//float c31=c13=c12;
	//float c32=c23=c12;
	//float c55=c66=c44;
	//----- ----- ----- -----
	float cm22,cm33;
		cm22=cm33=cm11;
	float cm21;
		cm21=cm12;
	float cm31,cm13;
		cm31=cm13=cm12;
	float cm32,cm23;
		cm32=cm23=cm12;
	float cm55, cm66;
		cm55=cm66=cm44;
	//----- ----- ----- -----
	float cp22, cp33;
		cp22=cp33=cp11;
	float cp21;
		cp21=cp12;
	float cp31, cp13;
		cp31=cp13=cp12;
	float cp32, cp23;
		cp32=cp23=cp12;
	float cp55, cp66;
		cp55=cp66=cp44;
	//----- ----- ----- -----
	float et21;
	float et32;
	float et31;
	//----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- -----
	//float smatx_real[Nx][Ny][Nz][3][3];
	float *smatx_real = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//
	//float smatx_imag[Nx][Ny][Nz][3][3];
	float *smatx_imag = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//----- ----- ----- ----- ----- ----- -----
	//float ematx_real[Nx][Ny][Nz][3][3];
	float *ematx_real = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//
	//float ematx_imag[Nx][Ny][Nz][3][3];
	float *ematx_imag = (float *)malloc(sizeof(float)*( NxNyNz*3*3 ));
	//----- ----- ----- ----- ----- ----- -----
	
	//float tmatx[3][3][3][3];
	float *tmatx = (float *)malloc(sizeof(float)*( 3*3*3*3 )); //real part only
	
	//----- ----- -----
	float et11=0.0;
	float et22=0.0;
	float et33=0.0;
	//
	float et12=0.0;
	float et23=0.0;
	float et13=0.0;
	//----- ----- -----
	
	//float sum_stress[Nx][Ny][Nz];
	float *sum_stress = (float *)malloc(sizeof(float)*( NxNyNz ));
	
	//----- ----- -----
	//Maximum number of iteration steps
	int niter=10;
	//----- ----- -----
	float old_norm=0.0;
	float normF=0.0;
	//----- ----- -----
	float conver=0.0;
	//----- ----- -----
	//Tolerance value of convergence tests
	float tolerance=0.001;
	//----- ----- -----
	
	/* Solve stress and strain field with 
	   iterative algorithm given in the text */
	for(int iter=0;iter<niter;iter++){
		
		/* Take stress and strain components from real space to
		   Fourier space (forward FFT). Step-a */
		//----- ----- ----- -----
		// stress (head s series)
		//s11k=fft3(s11);
		fftw_execute(plan_s11);
		//s22k=fft3(s22);
		fftw_execute(plan_s22);
		//s33k=fft3(s33);
		fftw_execute(plan_s33);
		//
		//s12k=fft3(s12);
		fftw_execute(plan_s12);
		//s23k=fft3(s23);
		fftw_execute(plan_s23);
		//s13k=fft3(s13);
		fftw_execute(plan_s13);
		//----- ----- ----- -----
		// strain (head e series)
		//e11k=fft3(e11);
		fftw_execute(plan_e11);
		//e22k=fft3(e22);
		fftw_execute(plan_e22);
		//e33k=fft3(e33);
		fftw_execute(plan_e33);
		//
		//e12k=fft3(e12);
		fftw_execute(plan_e12);
		//e23k=fft3(e23);
		fftw_execute(plan_e23);
		//e13k=fft3(e13);
		fftw_execute(plan_e13);
		//----- ----- ----- -----
		
		/* Form stress and strain tensors to be used in 
		   Eq.5.46, Step-b */
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					//----- ----- ----- ----- ----- -----
					// stress (smatx, sXY and sXYk)
					smatx_real[(ijk*3+0)*3+0]=s11k[ijk][0];
					smatx_real[(ijk*3+0)*3+1]=s12k[ijk][0];
					smatx_real[(ijk*3+0)*3+2]=s13k[ijk][0];
					//
					smatx_real[(ijk*3+1)*3+0]=s12k[ijk][0];
					smatx_real[(ijk*3+1)*3+1]=s22k[ijk][0];
					smatx_real[(ijk*3+1)*3+2]=s23k[ijk][0];
					//
					smatx_real[(ijk*3+2)*3+0]=s13k[ijk][0];
					smatx_real[(ijk*3+2)*3+1]=s23k[ijk][0];
					smatx_real[(ijk*3+2)*3+2]=s33k[ijk][0];
					//----- ----- ----- -----
					smatx_imag[(ijk*3+0)*3+0]=s11k[ijk][1];
					smatx_imag[(ijk*3+0)*3+1]=s12k[ijk][1];
					smatx_imag[(ijk*3+0)*3+2]=s13k[ijk][1];
					//
					smatx_imag[(ijk*3+1)*3+0]=s12k[ijk][1];
					smatx_imag[(ijk*3+1)*3+1]=s22k[ijk][1];
					smatx_imag[(ijk*3+1)*3+2]=s23k[ijk][1];
					//
					smatx_imag[(ijk*3+2)*3+0]=s13k[ijk][1];
					smatx_imag[(ijk*3+2)*3+1]=s23k[ijk][1];
					smatx_imag[(ijk*3+2)*3+2]=s33k[ijk][1];
					//----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					ematx_real[(ijk*3+0)*3+0]=e11k[ijk][0];
					ematx_real[(ijk*3+0)*3+1]=e12k[ijk][0];
					ematx_real[(ijk*3+0)*3+2]=e13k[ijk][0];
					//
					ematx_real[(ijk*3+1)*3+0]=e12k[ijk][0];
					ematx_real[(ijk*3+1)*3+1]=e22k[ijk][0];
					ematx_real[(ijk*3+1)*3+2]=e23k[ijk][0];
					//
					ematx_real[(ijk*3+2)*3+0]=e13k[ijk][0];
					ematx_real[(ijk*3+2)*3+1]=e23k[ijk][0];
					ematx_real[(ijk*3+2)*3+2]=e33k[ijk][0];
					//----- ----- ----- -----
					ematx_imag[(ijk*3+0)*3+0]=e11k[ijk][1];
					ematx_imag[(ijk*3+0)*3+1]=e12k[ijk][1];
					ematx_imag[(ijk*3+0)*3+2]=e13k[ijk][1];
					//
					ematx_imag[(ijk*3+1)*3+0]=e12k[ijk][1];
					ematx_imag[(ijk*3+1)*3+1]=e22k[ijk][1];
					ematx_imag[(ijk*3+1)*3+2]=e23k[ijk][1];
					//
					ematx_imag[(ijk*3+2)*3+0]=e13k[ijk][1];
					ematx_imag[(ijk*3+2)*3+1]=e23k[ijk][1];
					ematx_imag[(ijk*3+2)*3+2]=e33k[ijk][1];
					//----- ----- ----- ----- ----- -----
				}
			}
		}
		
		//Green operator
		// Calculate strain tensor, Eq.5.46, Step-b
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					//----- ----- ----- ----- ----- -----
					green_tensor2_3D(Nx,Ny,Nz,
									kx,ky,kz,
									omeg11,omeg22,omeg33,
									omeg12,omeg23,omeg13,
									i,j,k,
									tmatx);
					//----- ----- ----- ----- ----- -----
					for(int kk=0;kk<3;kk++){
						for(int ll=0;ll<3;ll++){
							for(int ii=0;ii<3;ii++){
								for(int jj=0;jj<3;jj++){
									klij=((kk*3+ll)*3+ii)*3+jj;
									/* Eq.5.46(b): new epsilon(zeta) = epsilon(zeta) - sum( gamma(zeta)*sigma(zeta) )
									   where gamma=tmatx, sigma=smatx
									   Note: tmatx is real part only */
									//----- ----- ----- ----- ----- -----
									ematx_real[(ijk*3+ii)*3+jj] -= tmatx[klij]*smatx_real[(ijk*3+kk)*3+ll];
									//
									ematx_imag[(ijk*3+ii)*3+jj] -= tmatx[klij]*smatx_imag[(ijk*3+kk)*3+ll];
									//----- ----- ----- ----- ----- -----
								}//jj
							}//ii
						}//ll
					}//kk
					//----- ----- ----- ----- ----- -----
				}//Nz
			}//Ny
		}//Nx
		
		// Rearrange strain components using symmetry of strain tensor
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					e11k[ijk][0]=ematx_real[(ijk*3+0)*3+0];
					e12k[ijk][0]=ematx_real[(ijk*3+0)*3+1];
					e13k[ijk][0]=ematx_real[(ijk*3+0)*3+2];
					//
					e12k[ijk][0]=ematx_real[(ijk*3+1)*3+0];
					e22k[ijk][0]=ematx_real[(ijk*3+1)*3+1];
					e23k[ijk][0]=ematx_real[(ijk*3+1)*3+2];
					//
					e13k[ijk][0]=ematx_real[(ijk*3+2)*3+0];
					e23k[ijk][0]=ematx_real[(ijk*3+2)*3+1];
					e33k[ijk][0]=ematx_real[(ijk*3+2)*3+2];
					//----- ----- ----- -----
					e11k[ijk][1]=ematx_imag[(ijk*3+0)*3+0];
					e12k[ijk][1]=ematx_imag[(ijk*3+0)*3+1];
					e13k[ijk][1]=ematx_imag[(ijk*3+0)*3+2];
					//
					e12k[ijk][1]=ematx_imag[(ijk*3+1)*3+0];
					e22k[ijk][1]=ematx_imag[(ijk*3+1)*3+1];
					e23k[ijk][1]=ematx_imag[(ijk*3+1)*3+2];
					//
					e13k[ijk][1]=ematx_imag[(ijk*3+2)*3+0];
					e23k[ijk][1]=ematx_imag[(ijk*3+2)*3+1];
					e33k[ijk][1]=ematx_imag[(ijk*3+2)*3+2];
					//----- ----- ----- ----- ----- -----
				}
			}
		}
		
		//From Fourier space to real space
		/* Take strain components from Fourier space back to
		   real space (inverse FFT), Step-c */
		//----- ----- ----- -----
		//e11=real(ifft3(e11k));
		fftw_execute(iplan_e11k);
		//e22=real(ifft3(e22k));
		fftw_execute(iplan_e22k);
		//e33=real(ifft3(e33k));
		fftw_execute(iplan_e33k);
		//
		//e12=real(ifft3(e12k));
		fftw_execute(iplan_e12k);
		//e23=real(ifft3(e23k));
		fftw_execute(iplan_e23k);
		//e13=real(ifft3(e13k));
		fftw_execute(iplan_e13k);
		//----- ----- ----- -----
		
		//Calculate stresses
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					//----- ----- ----- ----- ----- -----
					// strain (ematx, eXY and eXYk)
					//----- ----- ----- -----
					e11[ijk][0] /= fftw3d_scale;
					e22[ijk][0] /= fftw3d_scale;
					e33[ijk][0] /= fftw3d_scale;
					//
					e12[ijk][0] /= fftw3d_scale;
					e23[ijk][0] /= fftw3d_scale;
					e13[ijk][0] /= fftw3d_scale;
					//----- ----- ----- -----
					e11[ijk][1] /= fftw3d_scale;
					e22[ijk][1] /= fftw3d_scale;
					e33[ijk][1] /= fftw3d_scale;
					//
					e12[ijk][1] /= fftw3d_scale;
					e23[ijk][1] /= fftw3d_scale;
					e13[ijk][1] /= fftw3d_scale;
					//----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- ----- ----- -----
					/*s11[ijk][0]=c11[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c13[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c21[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c22[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c23[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c31[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c32[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c33[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);*/
					//
					// c13[ijk]=c12[ijk], c23[ijk]=c13[ijk], c22[ijk]=c11[ijk], c33[ijk]=c11[ijk], etc
					/*s11[ijk][0]=c11[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c12[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s22[ijk][0]=c12[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c11[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c12[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);
					s33[ijk][0]=c12[ijk]*(ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk])
							   +c12[ijk]*(ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk])
							   +c11[ijk]*(ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk]);*/
					//
					// Calculate strain (head e series) components
					et11=( ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk] );
					et22=( ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk] );
					et33=( ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk] );
					//
					// Calculate stress (head s series) components
					s11[ijk][0]=( c11[ijk]*et11 + c12[ijk]*et22 + c12[ijk]*et33 );
					s22[ijk][0]=( c12[ijk]*et11 + c11[ijk]*et22 + c12[ijk]*et33 );
					s33[ijk][0]=( c12[ijk]*et11 + c12[ijk]*et22 + c11[ijk]*et33 );
					//
					s11[ijk][1]=0.0;
					s22[ijk][1]=0.0;
					s33[ijk][1]=0.0;
					//----- ----- ----- ----- ----- ----- ----- -----
					
					//----- ----- ----- ----- ----- ----- ----- -----
					/*s12[ijk][0]=c44[ijk]*(ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk])
							   +c44[ijk]*(ea[3]+e21[ijk][0]-ei21[ijk]-ed21[ijk]);
					s23[ijk][0]=c55[ijk]*(ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk])
							   +c55[ijk]*(ea[4]+e32[ijk][0]-ei32[ijk]-ed32[ijk]);
					s13[ijk][0]=c66[ijk]*(ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk])
							   +c66[ijk]*(ea[5]+e31[ijk][0]-ei31[ijk]-ed31[ijk]);*/
					//
					// c55[ijk]=c44[ijk], c66[ijk]=c44[ijk], etc
					/*s12[ijk][0]=c44[ijk]*(ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk])*2.0;
					  s23[ijk][0]=c44[ijk]*(ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk])*2.0;
					  s13[ijk][0]=c44[ijk]*(ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk])*2.0;*/
					//
					// Calculate strain (head e series) components
					et12=( ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk] );
					et23=( ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk] );
					et13=( ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk] );
					//
					// Calculate stress (head s series) components
					s12[ijk][0]=c44[ijk]*et12*2.0;
					s23[ijk][0]=c44[ijk]*et23*2.0;
					s13[ijk][0]=c44[ijk]*et13*2.0;
					//
					s12[ijk][1]=0.0;
					s23[ijk][1]=0.0;
					s13[ijk][1]=0.0;
					//----- ----- ----- ----- ----- ----- ----- -----
				}
			}
		}
		
		//check convergence
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					sum_stress[ijk] = ( s11[ijk][0] + s22[ijk][0] + s33[ijk][0]
									   +s12[ijk][0] + s23[ijk][0] + s13[ijk][0] );
				}
			}
		}
		
		//normF=norm(sum_stress,2.0);
		for(int i=0;i<Nx;i++){
			for(int j=0;j<Ny;j++){
				for(int k=0;k<Nz;k++){
					ijk=(i*Ny+j)*Nz+k;
					normF = normF + sum_stress[ijk]*sum_stress[ijk];
				}
			}
		}
		normF=sqrt(normF);
		
		if(iter==1){
			conver=fabs((normF-old_norm)/(old_norm));
			if(conver<=tolerance){
				break;
			}
		}
		old_norm=normF;
		
	}//end iter
	
	//strain energy
	//Calculate functional derivative of elastic energy
	// sum strain components
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				ijk=(i*Ny+j)*Nz+k;
				//----- ----- ----- ----- ----- ----- ----- -----
				// Calculate strain (head e series) components
				et11=( ea[0]+e11[ijk][0]-ei11[ijk]-ed11[ijk] );
				et22=( ea[1]+e22[ijk][0]-ei22[ijk]-ed22[ijk] );
				et33=( ea[2]+e33[ijk][0]-ei33[ijk]-ed33[ijk] );
				//
				et12=( ea[3]+e12[ijk][0]-ei12[ijk]-ed12[ijk] );
				et23=( ea[4]+e23[ijk][0]-ei23[ijk]-ed23[ijk] );
				et13=( ea[5]+e13[ijk][0]-ei13[ijk]-ed13[ijk] );
				//----- ----- ----- ----- ----- ----- ----- -----
				
				//----- ----- ----- ----- ----- ----- ----- -----
				//Functional derivative of the elastic energy with respect to composition
				/* F=(1/2)*sigma[i][j]*(epsilon[i][j] - epsilon0[i][j])
				   sigma[i][j] = C[i][j][k][l]*(epsilon[k][l] - epsilon0[k][l])
				   epsilon0[i][j] is the position- and composition-dependent eigenstrains */
				/* dF/dc = dF/d(con) = (1/2)*( dCijkl/d(con)*etij*etkl + Cijkl*d(etij)/d(con)*etkl + Cijkl*etij*d(etkl)/d(con) )
				   dCijkl/d(con) = (cpijkl - cmijkl), d(etij)/d(con) = -d(eiij)/d(con) = -ei0 */
				/* delsdc[ijk][0]=0.5*( (cp11-cm11)*et11*et11 -c11[ijk]*ei0*et11 -c11[ijk]*et11*ei0
								    +(cp12-cm12)*et11*et22 -c12[ijk]*ei0*et11 -c12[ijk]*et22*ei0
								    +(cp13-cm13)*et11*et33 -c13[ijk]*ei0*et11 -c13[ijk]*et33*ei0
								    //
								    +(cp21-cm21)*et22*et11 -c21[ijk]*ei0*et22 -c21[ijk]*et11*ei0
								    +(cp22-cm22)*et22*et22 -c22[ijk]*ei0*et22 -c22[ijk]*et22*ei0
								    +(cp23-cm23)*et22*et33 -c23[ijk]*ei0*et22 -c23[ijk]*et33*ei0
								    //
								    +(cp31-cm31)*et33*et11 -c31[ijk]*ei0*et33 -c31[ijk]*et11*ei0
								    +(cp32-cm32)*et33*et22 -c32[ijk]*ei0*et33 -c32[ijk]*et22*ei0
								    +(cp33-cm33)*et33*et33 -c33[ijk]*ei0*et33 -c33[ijk]*et33*ei0
								    //
								    +(cp44-cm44)*et12*et12 -c44[ijk]*ei0*et12 -c44[ijk]*et12*ei0
								    +(cp44-cm44)*et21*et21 -c44[ijk]*ei0*et21 -c44[ijk]*et21*ei0
								    //
								    +(cp55-cm55)*et23*et23 -c55[ijk]*ei0*et23 -c55[ijk]*et23*ei0
								    +(cp55-cm55)*et32*et32 -c55[ijk]*ei0*et32 -c55[ijk]*et32*ei0
								    //
								    +(cp66-cm66)*et13*et13 -c66[ijk]*ei0*et13 -c66[ijk]*et13*ei0
								    +(cp66-cm66)*et31*et31 -c66[ijk]*ei0*et31 -c66[ijk]*et31*ei0
								   ); */
				//
				// c13[ijk]=c12[ijk], c23[ijk]=c13[ijk], c22[ijk]=c11[ijk], c33[ijk]=c11[ijk], etc
				et21=et12;
				et32=et23;
				et31=et13;
				//
				delsdc[ijk]=0.5*( (cp11-cm11)*et11*et11 -c11[ijk]*ei0*( et11 + et11 )
								    +(cp12-cm12)*et11*et22 -c12[ijk]*ei0*( et11 + et22 )
								    +(cp13-cm13)*et11*et33 -c12[ijk]*ei0*( et11 + et33 )
								    //
								    +(cp21-cm21)*et22*et11 -c12[ijk]*ei0*( et22 + et11 )
								    +(cp22-cm22)*et22*et22 -c11[ijk]*ei0*( et22 + et22 )
								    +(cp23-cm23)*et22*et33 -c12[ijk]*ei0*( et22 + et33 )
								    //
								    +(cp31-cm31)*et33*et11 -c12[ijk]*ei0*( et33 + et11 )
								    +(cp32-cm32)*et33*et22 -c12[ijk]*ei0*( et33 + et22 )
								    +(cp33-cm33)*et33*et33 -c11[ijk]*ei0*( et33 + et33 )
								    //
								    +(cp44-cm44)*et12*et12 -c44[ijk]*ei0*( et12 + et12 )
								    +(cp44-cm44)*et21*et21 -c44[ijk]*ei0*( et21 + et21 )
								    //
								    +(cp55-cm55)*et23*et23 -c44[ijk]*ei0*( et23 + et23 )
								    +(cp55-cm55)*et32*et32 -c44[ijk]*ei0*( et32 + et32 )
								    //
								    +(cp66-cm66)*et13*et13 -c44[ijk]*ei0*( et13 + et13 )
								    +(cp66-cm66)*et31*et31 -c44[ijk]*ei0*( et31 + et31 )
								   );
				//delsdc[ijk][1]=0.0;
				//----- ----- ----- ----- ----- ----- ----- -----
			}
		}
	}
	
	//----- ----- ----- ----- -----
	fftw_destroy_plan(plan_s11);
	fftw_destroy_plan(plan_s22);
	fftw_destroy_plan(plan_s33);
	//
	fftw_destroy_plan(plan_s12);
	fftw_destroy_plan(plan_s23);
	fftw_destroy_plan(plan_s13);
	//----- ----- ----- ----- -----
	 fftw_destroy_plan(plan_e11);
	fftw_destroy_plan(iplan_e11k);
	 fftw_destroy_plan(plan_e22);
	fftw_destroy_plan(iplan_e22k);
	 fftw_destroy_plan(plan_e33);
	fftw_destroy_plan(iplan_e33k);
	//
	 fftw_destroy_plan(plan_e12);
	fftw_destroy_plan(iplan_e12k);
	 fftw_destroy_plan(plan_e23);
	fftw_destroy_plan(iplan_e23k);
	 fftw_destroy_plan(plan_e13);
	fftw_destroy_plan(iplan_e13k);
	//----- ----- ----- ----- -----
	fftw_free(s11k);
	fftw_free(s22k);
	fftw_free(s33k);
	//
	fftw_free(s12k);
	fftw_free(s23k);
	fftw_free(s13k);
	//----- ----- ----- ----- -----
	fftw_free(e11k);
	fftw_free(e22k);
	fftw_free(e33k);
	//
	fftw_free(e12k);
	fftw_free(e23k);
	fftw_free(e13k);
	//----- ----- ----- ----- ----- -----
	free(smatx_real);
	free(smatx_imag);
	free(ematx_real);
	free(ematx_imag);
	//----- ----- ----- ----- -----
	free(sum_stress);
	//----- ----- ----- ----- -----
	free(ei11);
	free(ei22);
	free(ei33);
	//
	free(ei12);
	free(ei23);
	free(ei13);
	//----- ----- ----- ----- -----
	free(c11);
	free(c12);
	free(c44);
	//----- ----- ----- ----- -----
	
	return;
}