/* This function initializes the microstructure for
   given average composition modulated with
   a noise term to account the thermal fluctuations in
   Cahn-Hilliard equation. */


#include <hip/hip_runtime.h>
#include <stdlib.h> //rand()
//#include <fftw3.h>

/* Variable and array list
  Nx: Number of grid points in the x-direction
  Ny: Number of grid points in the y-direction
  Nz: Number of grid points in the z-direction
  c0: Average alloy composition
  con[Nx][Ny][Nz]: Concentration field for un-optimized mode (iflag=1)
*/

void micro_ch_pre_3d(int Nx, int Ny, int Nz, float c0, float *con){
	
	// Total number of grid points in the simulation cell
	//int NxNy=Nx*Ny;
	
	//Set the magnitude of the noise term for fluctuations
	float noise=0.02;
	
	int ijk;
	
	//Introduce random flucturation to concentration, con[Nx][Ny]
	for(int i=0;i<Nx;i++){
		for(int j=0;j<Ny;j++){
			for(int k=0;k<Nz;k++){
				ijk=(i*Ny+j)*Nz+k;
				con[ijk] = c0 + noise*(0.5-(float)rand()/RAND_MAX);
			}
		}
	}
	
	return;
}