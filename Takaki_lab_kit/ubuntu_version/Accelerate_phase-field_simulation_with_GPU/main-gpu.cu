/* Program : 2D Phase-Field Simulation for 
   Spinodal Decomposition in Fe-Cr Alloy by GPU Computation.
   
   Programmer : Akinori Yamanaka (original version)
   Place : Depertment of Mechanical and Control Engineering Tokyo Institute of Technology
   Date : 7th, July, 2010 
   
   Modified version: By Student
   Place: 2-1 Hirosawa, Wako, Saitama, 351-0198, Japan
   Date: 11th/May/2023
*/

#include <stdio.h>  //printf()
#include <stdlib.h> //rand() and malloc()
#include <math.h>   //mod() and -lm
//#include <cuda.h>
#include "hip/hip_runtime.h"


#define NX 256 //Number of grid points in the x-direction
#define NY 256 //Number of grid points in the y-direction

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel
(
	float *f,float *fn, int nx, int ny,
	float  rr, float temp,float L0,
	float kapa_c,float da,float db,float dt,float dx,float dy
)
{
	int j, jx, jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.y*blockIdx.y + threadIdx.y;
	jy = blockDim.x*blockIdx.x + threadIdx.x;
	j  = nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	float fcc, fce, fcw, fcs, fcn, fcnw, fcne, fcsw, fcse, fcww, fcee, fcnn, fcss, 
		  mu_chc, mu_chw, mu_che, mu_chn, mu_chs, mu_suc, mu_suw, mu_sue, mu_sun, 
		  mu_sus, mu_c, mu_w, mu_e, mu_n, mu_s,  
		  nab_mu, dfmdx, dfmdy, dab = db/da, mcc, dmc, dfdt ;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	/* Consider up to the second nearest neighbor. Therefore, 
	13 difference grid points are used. (#1 to #13)
	   The if statement is used because of periodic boundary conditions.*/
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #1
	fcc = f[j]; // The value of concentration at current (jx,jy) grid point
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #2
	if(jx == 0)    fcw = f[j+nx-1];
	else           fcw = f[j   -1];
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #3
	if(jx == nx-1) fce = f[j-nx+1];
	else           fce = f[j   +1];
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #4
	if(jy == 0)    fcs = f[j+nx*ny-nx];
	else           fcs = f[j      -nx];
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #5
	if(jy == ny-1) fcn = f[j-nx*ny+nx];
	else           fcn = f[j      +nx];
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #6
	if(jx == 0 && jy == ny-1)     { fcnw = f[nx-1];}
	else if(jx == 0 && jy  < ny-1){ fcnw = f[j+nx    +nx-1];}
	else if(jx  > 0 && jy == ny-1){ fcnw = f[j-nx*ny +nx-1];}
	else                          { fcnw = f[j       +nx-1];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #7
	if(jx == nx-1 && jy  < ny-1)     { fcne = f[j-nx    +nx+1];}
	else if(jx  < nx-1 && jy == ny-1){ fcne = f[j-nx*ny +nx+1];}
	else if(jx == nx-1 && jy == ny-1){ fcne = f[0];}
	else                             { fcne = f[j       +nx+1];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #8
	if(jx == 0 && jy >  0)     { fcsw = f[j+nx    -nx-1];}
	else if(jx  > 0 && jy == 0){ fcsw = f[j+nx*ny -nx-1];}
	else if(jx == 0 && jy == 0){ fcsw = f[nx*ny-1];} 
	else                       { fcsw = f[j       -nx-1];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #9
	if(jx == nx-1 && jy == 0)     {fcse = f[nx*ny-1-nx+1];}
	else if(jx == nx-1 && jy  > 0){ fcse = f[j-nx    -nx+1];}
	else if(jx <  nx-1 && jy == 0){ fcse = f[j+nx*ny -nx+1];}
	else                          { fcse = f[j       -nx+1];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #10
	if(jx == 0) {fcww = f[j+nx-2];}
	else if(jx == 1){ fcww = f[j+nx-2];}
	else            { fcww = f[j   -2];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #11
	if(jx == nx - 2)     { fcee = f[j-nx+2];}
	else if(jx == nx - 1){ fcee = f[j-nx+2];}
	else                 { fcee = f[j   +2];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #12
	if(jy == ny - 2)      { fcnn = f[j-nx*ny+nx+nx];}
	else if(jy == ny - 1) { fcnn = f[j-nx*ny+nx+nx];}
	else                  { fcnn = f[j      +nx+nx];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #13
	if(jy == 0)      { fcss = f[j+nx*ny-nx-nx];}
	else if(jy == 1) { fcss = f[j+nx*ny-nx-nx];}
	else             { fcss = f[j      -nx-nx];}
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term1 = Atomic_interaction*(1-2*f) + RT*{log(f) - log(1-f)}
	mu_chc = L0*(1.0-2.0*fcc)+rr*temp*(log(fcc)-log(1.0-fcc));
	mu_chw = L0*(1.0-2.0*fcw)+rr*temp*(log(fcw)-log(1.0-fcw));
	mu_che = L0*(1.0-2.0*fce)+rr*temp*(log(fce)-log(1.0-fce));
	mu_chn = L0*(1.0-2.0*fcn)+rr*temp*(log(fcn)-log(1.0-fcn));
	mu_chs = L0*(1.0-2.0*fcs)+rr*temp*(log(fcs)-log(1.0-fcs));
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term2 = -gradient_energy_coefficient * Laplacian(f)
	mu_suc = -kapa_c*(fce +fcw +fcn +fcs -4.0*fcc)/dx/dx;
	mu_suw = -kapa_c*(fcc +fcww+fcnw+fcsw-4.0*fcw)/dx/dx;
	mu_sue = -kapa_c*(fcee+fcc +fcne+fcse-4.0*fce)/dx/dx;
	mu_sun = -kapa_c*(fcne+fcnw+fcnn+fcc -4.0*fcn)/dx/dx;
	mu_sus = -kapa_c*(fcse+fcsw+fcc +fcss-4.0*fcs)/dx/dx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// mu = dG/df = term1 + term2
	mu_c = mu_chc + mu_suc; // at current (jx,jy) grid point
	mu_w = mu_chw + mu_suw; // at (jx-1,jy) grid point
	mu_e = mu_che + mu_sue; // at (jx+1,jy) grid point
	mu_n = mu_chn + mu_sun; // at (jx,jy+1) grid point
	mu_s = mu_chs + mu_sus; // at (jx,jy-1) grid point
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Laplacian(mu) = d^2(mu)/dx^2 + d^2(mu)/dy^2
	nab_mu = (mu_w + mu_e + mu_n + mu_s -4.0*mu_c)/dx/dx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dx) * d(mu)/dx
	dfmdx = ((mu_w-mu_e)*(fcw-fce))/(4.0*dx*dx);
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dy) * d(mu)/dy
	dfmdy = ((mu_n-mu_s)*(fcn-fcs))/(4.0*dy*dy);
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Mobility, M = { (D_A/RT)*c + (D_B/RT)*(1-c) }*c*(1-c)
	//             = (D_a/RT)*{f + (D_B/D_A)*(1-f)}*f*(1-f)
	mcc = (da/rr/temp)*(fcc+dab*(1.0-fcc))*fcc*(1.0-fcc);
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// dM/df
	dmc = (da/rr/temp)*((1.0-dab)*fcc*(1.0-fcc)
		+(fcc+dab*(1.0-fcc))*(1.0-2.0*fcc));
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// df/dt = M*Laplacian(f) + (dM/df)*( (df/dx) * d(mu)/dx + (df/dy) * d(mu)/dy )
	dfdt = mcc*nab_mu + dmc*(dfmdx+dfmdy);
	fn[j] = f[j]+dfdt*dt;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
}

void update(float **f, float **fn)
{
	float *tmp = *f;
	*f  = *fn;
	*fn = tmp;
}

void write_vtk_grid_values_2D(int Nx, int Ny, float dx, float dy, int istep, float *data1);

int main(int argc, char** argv)
{
	float *f_d, *fn_d; // name of dynamic memory for GPU, CUDA, device
	float *F_h;        // name of dynamic memory for CPU
	int nx = NX, ny = NY;
	
	int nstep=10000;    //Number of time integration steps
	int nprint=1000;    //Output frequency to write the results to file
	float Lx = 3.0e-07, // Simulation length in x-direction [micro m]
		  Ly = 3.0e-07, // Simulation length in y-direction [micro m]
		  dx = Lx/(float)nx, // Grid spacing between two grid pints in x-direction [nm]
		  dy = Ly/(float)ny, // Grid spacing between two grid pints in y-direction [nm]
		  c_0 = 0.4,    // Initial concentration (atomic fraction)
		  rr = 8.314,   // Gas constant [J/(mol*K)]
		  temp = 673.0, // Temperature [K]
		  L0 = 21020.8-9.31889*temp, // Atomic interaction [J/mol]
		  kapa_c = 1.2e-14,  // The value of gradient energy coefficients [J*m^2/mol]
		  da = 1.0e-04*exp(-294000.0/rr/temp), // Self-diffusion coefficient [m^2/s] (Fe)
		  db = 2.0e-05*exp(-308000.0/rr/temp), // Self-diffusion coefficient [m^2/s] (Cr)
		  dt = (dx*dx/da)*0.1; // Time increment for the numerical integration [dimensionless]
	
	//CUT_DEVICE_INIT(argc, argv);
	
	f_d  = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	fn_d = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	
	hipMalloc((void**)&f_d ,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&fn_d,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	
	F_h  = (float *)malloc(nx*ny*sizeof(float));   // define dynamic memory for CPU (host)
	
	// Initialize the concentration filed F_h with random modulation
	for(int jy=0; jy<ny ; jy++){
		for(int jx=0; jx<nx ; jx++){
			int j = nx*jy + jx;
			float r = (float)rand()/(float)(RAND_MAX);
			F_h[j] = c_0 + 0.01*r;
		}
	}//on CPU calculation
	
	//copy F_h(cpu,host) to f_d(cuda,device)
	hipMemcpy(f_d,F_h,nx*ny*sizeof(float),hipMemcpyHostToDevice);
	
	int BS=16; // Number of threads, 16 or 32
	dim3 blocks(nx/BS,ny/BS,1); //nx*ny = blocks * threads
	dim3 threads(BS,BS,1);      //BS*BS*1 <= 1024
	
	//unsigned int timer;
	//cutCreateTimer(&timer);
	//cutResetTimer(timer);
	//cutStartTimer(timer);
	
	for(int istep=0; istep<=nstep ; istep++){
		//calculate subroutine "Kernel" on GPU
		Kernel<<<blocks, threads>>>(f_d,fn_d,nx,ny,rr,temp,L0,kapa_c,da,db,dt,dx,dy);
		//cudaThreadSynchronize();
		
		// replace f_d with new f_d (=fn_d)
		update(&f_d,&fn_d);
		//
		if(istep%nprint == 0){
			//copy f_d(cuda,device) to F_h(cpu,host)
			hipMemcpy(F_h,f_d,nx*ny*sizeof(float),hipMemcpyDeviceToHost);
			
			//output vtk format
			write_vtk_grid_values_2D(nx,ny,dx,dy,istep,F_h);
			
			//show current step
			fprintf(stderr,"istep = %5d \n",istep);
		}
		//
	}
	//cutStopTimer(timer);
	//float calc_time = cutGetTimerValue(timer)*1.0e-03;
	//printf("Calculation Time = %9.3e [sec]\n",calc_time);
	
	hipFree(f_d);
	hipFree(fn_d);
	
	free(F_h);
	
	return 0;
}
