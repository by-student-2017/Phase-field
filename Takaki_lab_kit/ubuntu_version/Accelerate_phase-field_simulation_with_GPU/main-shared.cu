#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include <cuda.h>
#include "hip/hip_runtime.h"


#define NX 256 //Number of grid points in the x-direction
#define NY 256 //Number of grid points in the y-direction

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel
(
	float *f, float *fn, int nx, int ny,
	float  rr, float temp, float L0,
	float  kapa_c, float da, float db, float dt, float dx, float dy
)
{
	int j, jx, jy;
	float  fcc, fce, fcw, fcs, fcn, fcnw, fcne, fcsw, fcse, fcww, fcee, fcnn, fcss, 
		   mu_chc, mu_chw, mu_che, mu_chn, mu_chs,
		   mu_suc, mu_suw, mu_sue, mu_sun, mu_sus, 
		   mu_c, mu_w, mu_e, mu_n, mu_s, 
		   nab_mu, dfmdx, dfmdy, dab = db/da, mcc, dmc, dfdt ;
	
	int joff;
	int J0,J1,J2,J3,J4,J5,J6,J7,J8,J9,J10,J11;
	//int thread_x = 16, thread_y = 16; // 16=BS (16 kB before GF100 Core)
	
	__shared__ float fs[16+4][16+4];
	
	jx = threadIdx.x + 2;
	jy = threadIdx.y + 2;
	joff = nx*(blockDim.y*blockIdx.y) + blockDim.x*blockIdx.x;
	j = joff + nx*threadIdx.y + threadIdx.x;
	
	fcc = f[j];
	fs[jx][jy] = fcc;
	
	if(blockIdx.y == 0) {J0 = nx*(ny-1)+blockDim.x*blockIdx.x + threadIdx.x,
						J4 = J0 - nx;} 
	else                {J0 =  j - nx, 
						J4 = J0 - nx;}
	
	if(blockIdx.y == gridDim.y - 1) {J1 = blockDim.x*blockIdx.x + threadIdx.x, 
						J5 = J1 + nx;} 
	else				{J1 =  j + nx, 
						J5 = J1 + nx;}
	
	if(blockIdx.x == 0) {J2 = joff + nx*threadIdx.x + nx - 1,
						J6 = J2 - 1;}
	else				{J2 = joff + nx*threadIdx.x - 1, 
						J6 = J2 - 1;}
	
	if(blockIdx.x == gridDim.x - 1) {J3 = joff + nx*threadIdx.x + 15 - nx + 1,
						J7 = J3 + 1;}
	else				{J3 = joff + nx*threadIdx.x + 16,
						J7 = J3 + 1;}
	
		 if(blockIdx.x == 0 && blockIdx.y == gridDim.y - 1) { J8 = blockDim.x*16 -1;}
	else if(blockIdx.x  > 0 && blockIdx.y == gridDim.y - 1) { J8 = J1 - 1 ;}
	else if(blockIdx.x == 0 && blockIdx.y  < gridDim.y - 1) { J8 = j + nx + nx -1;}
	else                                                    { J8 = j + nx -1 ;}
	
		 if(blockIdx.x == gridDim.x - 1 && blockIdx.y == gridDim.y - 1) { J9 = 0 ;}
	else if(blockIdx.x  < gridDim.x - 1 && blockIdx.y == gridDim.y - 1) { J9 = J1 + 1 ;}
	else if(blockIdx.x == gridDim.x - 1 && blockIdx.y  < gridDim.y - 1) { J9 = j  + 1 ;}
	else                                                                { J9 = j + nx +1 ;}
	
		 if(blockIdx.x  > 0 && blockIdx.y == 0) { J10 = J0 - 1 ;}
	else if(blockIdx.x == 0 && blockIdx.y  > 0) { J10 =  j -1  ;}
	else if(blockIdx.x == 0 && blockIdx.y == 0) { J10 = nx*blockDim.x*blockDim.y - 1 ;}
	else                                        { J10 = j - nx - 1 ;}
	
		 if(blockIdx.x == gridDim.x -1 && blockIdx.y == 0) { J11 = nx*blockDim.x*blockDim.y -1 - nx + 1;}
	else if(blockIdx.x  < gridDim.x -1 && blockIdx.y == 0) { J11 = J0 + 1  ;}
	else if(blockIdx.x == gridDim.x -1 && blockIdx.y  > 0) { J11 =  j - nx - nx + 1 ;}
	else                                                   { J11 = j - nx + 1 ;}
	
	if(threadIdx.y ==  0){ fs[jx][ 1] = f[J0], fs[jx][ 0] = f[J4];}
	if(threadIdx.y ==  1){ fs[ 1][jx] = f[J2], fs[ 0][jx] = f[J6];}
	if(threadIdx.y ==  2){ fs[18][jx] = f[J3], fs[19][jx] = f[J7];}
	if(threadIdx.y == 15){ fs[jx][18] = f[J1], fs[jx][19] = f[J5];}
	if(threadIdx.x ==  0 && threadIdx.y == 15) {fs[ 1][18] = f[J8];}
	if(threadIdx.x == 15 && threadIdx.y == 15) {fs[18][18] = f[J9];}
	if(threadIdx.x ==  0 && threadIdx.y ==  0) {fs[ 1][ 1] = f[J10];}
	if(threadIdx.x == 15 && threadIdx.y ==  0) {fs[18][ 1] = f[J11];}
	
	__syncthreads(); // Wait until all data is secured
	
	fcc  = fs[jx  ][jy  ];
	fcw  = fs[jx-1][jy  ];
	fce  = fs[jx+1][jy  ];
	fcn  = fs[jx  ][jy+1];
	fcs  = fs[jx  ][jy-1];
	
	fcww = fs[jx-2][jy  ];
	fcee = fs[jx+2][jy  ];
	fcnn = fs[jx  ][jy+2];
	fcss = fs[jx  ][jy-2];
	
	fcnw = fs[jx-1][jy+1];
	fcne = fs[jx+1][jy+1];
	fcsw = fs[jx-1][jy-1];
	fcse = fs[jx+1][jy-1];
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term1 = Atomic_interaction*(1-2*f) + RT*{log(f) - log(1-f)}
	mu_chc = L0*(1.0-2.0*fcc)+rr*temp*(log(fcc)-log(1.0-fcc));
	mu_chw = L0*(1.0-2.0*fcw)+rr*temp*(log(fcw)-log(1.0-fcw));
	mu_che = L0*(1.0-2.0*fce)+rr*temp*(log(fce)-log(1.0-fce));
	mu_chn = L0*(1.0-2.0*fcn)+rr*temp*(log(fcn)-log(1.0-fcn));
	mu_chs = L0*(1.0-2.0*fcs)+rr*temp*(log(fcs)-log(1.0-fcs));
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term2 = -gradient_energy_coefficient * Laplacian(f)
	mu_suc = -kapa_c*(fce +fcw +fcn +fcs -4.0*fcc)/dx/dx;
	mu_suw = -kapa_c*(fcc +fcww+fcnw+fcsw-4.0*fcw)/dx/dx;
	mu_sue = -kapa_c*(fcee+fcc +fcne+fcse-4.0*fce)/dx/dx;
	mu_sun = -kapa_c*(fcne+fcnw+fcnn+fcc -4.0*fcn)/dx/dx;
	mu_sus = -kapa_c*(fcse+fcsw+fcc +fcss-4.0*fcs)/dx/dx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// mu = dG/df = term1 + term2
	mu_c = mu_chc + mu_suc; // at current (jx,jy) grid point
	mu_w = mu_chw + mu_suw; // at (jx-1,jy) grid point
	mu_e = mu_che + mu_sue; // at (jx+1,jy) grid point
	mu_n = mu_chn + mu_sun; // at (jx,jy+1) grid point
	mu_s = mu_chs + mu_sus; // at (jx,jy-1) grid point
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Laplacian(mu) = d^2(mu)/dx^2 + d^2(mu)/dy^2
	nab_mu = (mu_w + mu_e + mu_n + mu_s -4.0*mu_c)/dx/dx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dx) * d(mu)/dx
	dfmdx = ((mu_w-mu_e)*(fcw-fce))/(4.0*dx*dx);
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dy) * d(mu)/dy
	dfmdy = ((mu_n-mu_s)*(fcn-fcs))/(4.0*dx*dx);
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Mobility, M = { (D_A/RT)*c + (D_B/RT)*(1-c) }*c*(1-c)
	//             = (D_a/RT)*{f + (D_B/D_A)*(1-f)}*f*(1-f)
	mcc = (da/rr/temp)*(fcc+dab*(1.0-fcc))*fcc*(1.0-fcc); 
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// dM/df
	dmc = (da/rr/temp)*((1.0-dab)*fcc*(1.0-fcc)
					  +(fcc+dab*(1.0-fcc))*(1.0-2.0*fcc));
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// df/dt = M*Laplacian(f) + (dM/df)*( (df/dx) * d(mu)/dx + (df/dy) * d(mu)/dy )
	dfdt = mcc*nab_mu + dmc*(dfmdx+dfmdy); 
	fn[j] = f[j]+dfdt*dt;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
}

void update(float **f, float **fn)
{
	float *tmp = *f;
	*f  = *fn;
	*fn = tmp;
}

void write_vtk_grid_values_2D(int Nx, int Ny, float dx, float dy, int istep, float *data1);

int main(int argc, char** argv)
{
	float *f_d, *fn_d; // name of dynamic memory for GPU, CUDA, device
	float *F_h;        // name of dynamic memory for CPU
	int nx = NX, ny = NY;
	
	int nstep=10000;    //Number of time integration steps
	int nprint=1000;    //Output frequency to write the results to file
	float Lx = 3.0e-07, // Simulation length in x-direction [micro m]
		  Ly = 3.0e-07, // Simulation length in y-direction [micro m]
		  dx = Lx/(float)nx, // Grid spacing between two grid pints in x-direction [nm]
		  dy = Ly/(float)ny, // Grid spacing between two grid pints in y-direction [nm]
		  c_0 = 0.4,    // Initial concentration (atomic fraction)
		  rr = 8.314,   // Gas constant [J/(mol*K)]
		  temp = 673.0, // Temperature [K]
		  L0 = 21020.8-9.31889*temp, // Atomic interaction [J/mol]
		  kapa_c = 1.2e-14,  // The value of gradient energy coefficients [J*m^2/mol]
		  da = 1.0e-04*exp(-294000.0/rr/temp), // Self-diffusion coefficient [m^2/s] (Fe)
		  db = 2.0e-05*exp(-308000.0/rr/temp), // Self-diffusion coefficient [m^2/s] (Cr)
		  dt = (dx*dx/da)*0.1; // Time increment for the numerical integration [dimensionless]
	
	//CUT_DEVICE_INIT(argc, argv);
	
	f_d  = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	fn_d = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	
	hipMalloc((void**)&f_d ,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&fn_d,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	
	F_h  = (float *)malloc(nx*ny*sizeof(float));   // define dynamic memory for CPU (host)
	
	// Initialize the concentration filed F_h with random modulation
	for(int jy=0; jy<ny ; jy++){
		for(int jx=0; jx<nx ; jx++){
			int j = nx*jy + jx;
			float r = (float)rand()/(float)(RAND_MAX);
			F_h[j] = c_0 + 0.01*r;
		}
	}//on CPU calculation
	
	//copy F_h(cpu,host) to f_d(cuda,device)
	hipMemcpy(f_d,F_h,nx*ny*sizeof(float),hipMemcpyHostToDevice);
	
	int BS=16; // Number of threads
	dim3 blocks(nx/BS,ny/BS,1); //nx*ny = blocks * threads
	dim3 threads(BS,BS,1);      //BS*BS*1 <= 1024
	
	//unsigned int timer;
	//cutCreateTimer(&timer);
	//cutResetTimer(timer);
	//cutStartTimer(timer);
	
	for(int istep=0; istep<=nstep ; istep++){
		//calculate subroutine "Kernel" on GPU
		Kernel<<<blocks, threads>>>(f_d,fn_d,nx,ny,rr,temp,L0,kapa_c,da,db,dt,dx,dy);
		//cudaThreadSynchronize();
		
		// replace f_d with new f_d (=fn_d)
		update(&f_d,&fn_d);
		
		if(istep%nprint == 0){
			//copy f_d(cuda,device) to F_h(cpu,host)
			hipMemcpy(F_h,f_d,nx*ny*sizeof(float),hipMemcpyDeviceToHost);
			
			//output vtk format
			write_vtk_grid_values_2D(nx,ny,dx,dy,istep,F_h);
			
			//show current step
			fprintf(stderr,"nstep = %5d \n",istep);
		}
	}
	//cutStopTimer(timer);
	//float calc_time = cutGetTimerValue(timer)*1.0e-03;
	//printf("Calculation Time = %9.3e [sec]\n",calc_time);
	
	hipFree(f_d);
	hipFree(fn_d);
	
	free(F_h);
	
	return 0;
}
