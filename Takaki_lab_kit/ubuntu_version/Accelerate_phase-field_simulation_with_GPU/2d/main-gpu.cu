#include "hip/hip_runtime.h"
/* Program : 2D Phase-Field Simulation for 
   Spinodal Decomposition in Fe-Cr Alloy by GPU Computation.
   
   Programmer : Akinori Yamanaka (original version)
   Place : Depertment of Mechanical and Control Engineering Tokyo Institute of Technology
   Date : 7th, July, 2010 
   
   Modified version: By Student
   Place: 2-1 Hirosawa, Wako, Saitama, 351-0198, Japan
   Date: 12th/May/2023
   Test: Ubuntu 22.04 LTS
   
   Compling: nvcc -O2 main-gpu.cu write_vtk_grid_values_2D.cu -o main-gpu.exe -arch=native -lm --std 'c++17'
   Run: ./main-gpu.exe
   ParaView: time_XX.vtk
*/

#include <stdio.h>  //printf()
#include <stdlib.h> //rand() and malloc()
#include <math.h>   //mod() and -lm
//----- ----- -----
#include <hip/hip_runtime.h>   //GPU
/* #include <hip/hip_runtime.h> or
  #include "hip/hip_runtime.h"
  #include "" */
//----- ----- -----

#define BS 32        //Number of threads, 2^n=<32, BS*BS*1 <= 1024
#define TIMES 2
//----- ----- -----
#define NX 256*TIMES //Number of grid points in the x-direction
#define NY 256*TIMES //Number of grid points in the y-direction

// Define subroutine "Kernel" for GPU (Device) calculation in detail
__global__ void Kernel
(
	float *f, 
	float *fn,
	int    nx,
	int    ny,
	float  rr,
	float  temp,
	float  L0,
	float  kapa_c,
	float  Da,
	float  Db,
	float  dt,
	float  dx,
	float  dy
)
{
	int j, jx, jy;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	jx = blockDim.y*blockIdx.y + threadIdx.y; //<-GPU | CPU -> for(jx=0; jx<nx; jx++){
	jy = blockDim.x*blockIdx.x + threadIdx.x; //<-GPU | CPU -> for(jy=0; jy<ny; jy++){
	j  = nx*jy + jx;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	float  fcc,
		   fce,  fcw,  fcs,  fcn,
		   //----- ----- -----
		   fcnw, fcne,
		   fcsw, fcse,
		   //----- ----- -----
		   fcww, fcee, fcnn, fcss,
		   //----- ----- ----- ----- ----- -----
		   RT = rr*temp,
		   //----- ----- ----- ----- ----- -----
		   mu_chc,
		   mu_chw, mu_che, mu_chn, mu_chs,
		   //----- ----- -----
		   mu_suc,
		   mu_suw, mu_sue, mu_sun, mu_sus, 
		   //----- ----- -----
		   mu_c,
		   mu_w, mu_e, mu_n, mu_s, 
		   //----- ----- ----- ----- ----- -----
		   nab_mu, 
		   dfmdx, dfmdy, 
		   //----- ----- -----
		   Dab = Db/Da, 
		   mcc, dmc,
		   //----- ----- -----
		   dfdt ;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	/* Consider up to the second nearest neighbor. Therefore, 
	13 difference grid points are used. (#1 to #13)
	   The if statement is used because of periodic boundary conditions. */
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #1 (center: fcc)
	fcc = f[j];
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #2 (center: fcc, and + w)
	if(jx == 0)    fcw = f[j+nx-1];    //boundary condition at west edge
	else           fcw = f[j   -1];    //non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #3 (center: fcc, and + e)
	if(jx == nx-1) fce = f[j-nx+1];    //boundary condition at east edge
	else           fce = f[j   +1];    //non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #4 (center: fcc, and + s)
	if(jy == 0)    fcs = f[j+nx*(+ny-1)]; //boundary condition at south edge
	else           fcs = f[j+nx*(   -1)]; //non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #5 (center: fcc, and + n)
	if(jy == ny-1) fcn = f[j+nx*(-ny+1)]; //boundary condition at north edge
	else           fcn = f[j+nx*(   +1)]; //non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #6 (center: fcc, and + n + w)(#5 and #2)
	/* e.g., "if(jx == 0 && jy == ny-1)" is f[ (j+nx*(-ny+1)) + (j+nx-1) - j] = f[j + nx*(-ny+1) +  nx-1] using above #5 and #2 condition.
	         "if(jx == 0 && jy  < ny-1)" is f[ j+nx*(   +1) + (j+nx-1) - j]   = f[j + nx*(   +1) +  nx-1] using above #5 and #2 condition. */
		 if(jx == 0 && jy == ny-1)   { fcnw = f[         nx-1];} // =f[j + nx*(-ny+1) +  nx-1] = f[nx*(ny-1)-nx*(ny-1)+nx-1] = f[nx-1]
	else if(jx == 0 && jy  < ny-1)   { fcnw = f[j+nx    +nx-1];} // =f[j + nx*(   +1) +  nx-1] = f[j+nx    +nx-1]
	else if(jx  > 0 && jy == ny-1)   { fcnw = f[j-nx*ny +nx-1];} // =f[j + nx*(-ny+1) +    -1] = f[j-nx*ny +nx-1]
	else                             { fcnw = f[j       +nx-1];} // =f[j + nx*(   +1) +    -1] = f[j       +nx-1]
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #7 (center: fcc, and + n + e)(#5 and #3)
		 if(jx == nx-1 && jy  < ny-1){ fcne = f[j+nx    -nx+1];} // =f[j + nx*(   +1) + -nx+1] = f[j+nx    -nx+1]
	else if(jx  < nx-1 && jy == ny-1){ fcne = f[j-nx*ny +nx+1];} // =f[j + nx*(-ny+1) +     1] = f[j-nx*ny +nx+1]
	else if(jx == nx-1 && jy == ny-1){ fcne = f[            0];} // =f[j + nx*(-ny+1) + -nx+1] = f[nx*(ny-1)+nx-1+nx*(-ny+1)-(nx-1)]=f[0]
	else                             { fcne = f[j       +nx+1];} // =f[j + nx*(   +1) +     1] = f[j       +nx+1]
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #8 (center: fcc, and + s + w)(#4 and #2)
		 if(jx == 0 && jy >  0)      { fcsw = f[j-nx    +nx-1];} // =f[j + nx*(   -1) +  nx-1] = f[j-nx    +nx-1]
	else if(jx  > 0 && jy == 0)      { fcsw = f[j+nx*ny -nx-1];} // =f[j + nx*(+ny-1) +    -1] = f[j+nx*ny -nx-1]
	else if(jx == 0 && jy == 0)      { fcsw = f[      nx*ny-1];} // =f[j + nx-1       + nx*(+ny-1)] = f[j+nx-1+nx*ny-nx] (and j= nx*jy + jx= nx*0 + 0 = 0)
	else                             { fcsw = f[j       -nx-1];} // =f[j + nx*(   -1) +    -1] = f[j       -nx-1]
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #9 (center: fcc, and + s + e)(#4 and #3)
		 if(jx == nx-1 && jy == 0)   { fcse = f[nx*ny-1 -nx+1];} // =f[j + nx*(+ny-1) + -nx+1] = f[nx-1+nx*ny-nx -nx+1]
	else if(jx == nx-1 && jy  > 0)   { fcse = f[j-nx    -nx+1];} // =f[j + nx*(   -1) + -nx+1] = f[j-nx    -nx+1]
	else if(jx <  nx-1 && jy == 0)   { fcse = f[j+nx*ny -nx+1];} // =f[j + nx*(+ny-1) +     1] = f[j+nx*ny -nx+1]
	else                             { fcse = f[j       -nx+1];} // =f[j + nx*(   -1) +     1] = f[j       -nx+1]
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #10 (center: fcc, and + w + w)
		 if(jx == 0)     { fcww = f[j+nx-2];}    // edge(west)
	else if(jx == 1)     { fcww = f[j+nx-2];}    // edge(west,one inside)
	else                 { fcww = f[j   -2];}    // non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #11 (center: fcc, and + e + e)
		 if(jx == nx - 2){ fcee = f[j-nx+2];}    // edge(east)
	else if(jx == nx - 1){ fcee = f[j-nx+2];}    // edge(east, one inside)
	else                 { fcee = f[j   +2];}    // non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #12 (center: fcc, and + n + n)
		 if(jy == ny - 2){ fcnn = f[j+nx*(-ny+2)];} // edge(north)
	else if(jy == ny - 1){ fcnn = f[j+nx*(-ny+2)];} // edge(north, one inside)
	else                 { fcnn = f[j+nx*(   +2)];} // non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- #13 (center: fcc, and + s + s)
		 if(jy == 0)     { fcss = f[j+nx*(+ny-2)];} // edge(south)
	else if(jy == 1)     { fcss = f[j+nx*(+ny-2)];} // edge(south, one inside)
	else                 { fcss = f[j+nx*(   -2)];} // non edge
	//----- ----- ----- ----- ----- ----- ----- ----- ----- -----
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term1 = Atomic_interaction*(1-2*f) + RT*{log(f) - log(1-f)}
	mu_chc = L0*(1.0-2.0*fcc) + RT*( log(fcc) - log(1.0-fcc) ); //center: fcc
	mu_chw = L0*(1.0-2.0*fcw) + RT*( log(fcw) - log(1.0-fcw) ); //center: fcw
	mu_che = L0*(1.0-2.0*fce) + RT*( log(fce) - log(1.0-fce) ); //center: fce
	mu_chn = L0*(1.0-2.0*fcn) + RT*( log(fcn) - log(1.0-fcn) ); //center: fcn
	mu_chs = L0*(1.0-2.0*fcs) + RT*( log(fcs) - log(1.0-fcs) ); //center: fcs
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// term2 = -gradient_energy_coefficient * Laplacian(f)
	mu_suc = -kapa_c*( (fce  + fcw  -2.0*fcc)/(dx*dx) + (fcn  + fcs  -2.0*fcc)/(dy*dy) ); //center: fcc
	mu_suw = -kapa_c*( (fcc  + fcww -2.0*fcw)/(dx*dx) + (fcnw + fcsw -2.0*fcw)/(dy*dy) ); //fcc=fcwe, fcnw=fcwn, fcsw=fcws, //center: fcw
	mu_sue = -kapa_c*( (fcee + fcc  -2.0*fce)/(dx*dx) + (fcne + fcse -2.0*fce)/(dy*dy) ); //fcc=fcew, fcne=fcen, fcse=fces, //center: fce
	mu_sun = -kapa_c*( (fcne + fcnw -2.0*fcn)/(dx*dx) + (fcnn + fcc  -2.0*fcn)/(dy*dy) ); //fcc=fcns, //center: fcn
	mu_sus = -kapa_c*( (fcse + fcsw -2.0*fcs)/(dx*dx) + (fcc  + fcss -2.0*fcs)/(dy*dy) ); //fcc=fcsn, //center: fcs
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// mu = dG/df = term1 + term2
	mu_c = mu_chc + mu_suc; // at current (jx,jy) grid point, //center: fcc
	mu_w = mu_chw + mu_suw; // at (jx-1,jy) grid point, //center: fcw
	mu_e = mu_che + mu_sue; // at (jx+1,jy) grid point, //center: fce
	mu_n = mu_chn + mu_sun; // at (jx,jy+1) grid point, //center: fcn
	mu_s = mu_chs + mu_sus; // at (jx,jy-1) grid point, //center: fcs
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Laplacian(mu) = d^2(mu)/dx^2 + d^2(mu)/dy^2
	nab_mu = (mu_w + mu_e -2.0*mu_c)/(dx*dx)  // d^2(mu)/dx^2
		   + (mu_n + mu_s -2.0*mu_c)/(dy*dy); // d^2(mu)/dy^2
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dx) * d(mu)/dx, (x is related with w and e), (the center is fc.)
	dfmdx = ( (fcw - fce)/(2.0*dx) * (mu_w - mu_e)/(2.0*dx) );
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// (df/dy) * d(mu)/dy, (y is related with n and s), (the center is fc.)
	dfmdy = ( (fcn - fcs)/(2.0*dy) * (mu_n - mu_s)/(2.0*dy) );
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// Mobility, M = { (D_A/RT)*c + (D_B/RT)*(1-c) }*c*(1-c)
	//             = (D_a/RT)*{f + (D_B/D_A)*(1-f)}*f*(1-f)
	mcc = (Da/RT)*(fcc+Dab*(1.0-fcc))*fcc*(1.0-fcc); 
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// dM/df
	dmc = (Da/RT)*((1.0-Dab)*fcc*(1.0-fcc) + (fcc+Dab*(1.0-fcc))*(1.0-2.0*fcc)); 
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
	// df/dt = M*Laplacian(f) + (dM/df)*( (df/dx) * d(mu)/dx + (df/dy) * d(mu)/dy )
	dfdt = mcc*nab_mu + dmc*(dfmdx + dfmdy); 
	fn[j] = f[j] + dfdt*dt;
	//----- ----- ----- ----- ----- ----- ----- ----- ----- ----- 
}

void update(float **f, float **fn)
{
	float *tmp = *f;
	*f  = *fn;
	*fn = tmp;
}

void write_vtk_grid_values_2D(int Nx, int Ny, float dx, float dy, int istep, float *data1);

int main(int argc, char** argv)
{
	float *f_d, *fn_d; // name of dynamic memory for GPU, CUDA, device
	float *F_h;        // name of dynamic memory for CPU
	int nx = NX, ny = NY;
	int times = TIMES;
	
	int nstep=10000;    //Number of time integration steps
	int nprint=1000;    //Output frequency to write the results to file
	//----- ----- ----- -----
	float Lx = 3.0e-07*times, // Simulation length in x-direction [micro m]
		  Ly = 3.0e-07*times, // Simulation length in y-direction [micro m]
		  //----- ----- ----- -----
		  dx = Lx/(float)nx, // Grid spacing between two grid pints in x-direction [nm]
		  dy = Ly/(float)ny, // Grid spacing between two grid pints in y-direction [nm]
		  //----- ----- ----- -----
		  c_0 = 0.4,    // Initial concentration (atomic fraction)
		  //----- ----- ----- -----
		  rr = 8.314,   // Gas constant [J/(mol*K)]
		  temp = 673.0, // Temperature [K]
		  RT = rr*temp,
		  //----- ----- ----- -----
		  L0 = 21020.8-9.31889*temp, // Atomic interaction [J/mol]
		  kapa_c = 1.2e-14,  // The value of gradient energy coefficients [J*m^2/mol]
		  //----- ----- ----- -----
		  Da = 1.0e-04*exp(-294000.0/RT), // Self-diffusion coefficient [m^2/s] (Fe)
		  Db = 2.0e-05*exp(-308000.0/RT), // Self-diffusion coefficient [m^2/s] (Cr)
		  //----- ----- ----- -----
		  dt = (dx*dx/Da)*0.1; // Time increment for the numerical integration [dimensionless]
	
	//----- ----- ----- -----start:(This part is not really needed.)----- ----- ----- ----
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("--------------------------------------------------\n");
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("--------------------------------------------------\n");
	}
	//----- ----- ----- -----end:(This part is not really needed.)----- ----- ----- ----
	
	f_d  = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	fn_d = (float *)malloc(nx*ny*sizeof(float)); //GPU, CUDA, device
	
	hipMalloc((void**)&f_d ,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	hipMalloc((void**)&fn_d,nx*ny*sizeof(float)); // define dynamic memory for GPU (device)
	
	F_h  = (float *)malloc(nx*ny*sizeof(float));   // define dynamic memory for CPU (host)
	
	// Initialize the concentration filed F_h with random modulation
	for(int jy=0; jy<ny ; jy++){
		for(int jx=0; jx<nx ; jx++){
			int j = nx*jy + jx;
			float r = (float)rand()/(float)(RAND_MAX);
			F_h[j] = c_0 + 0.01*r;
		}
	}//on CPU calculation
	
	//copy F_h(cpu,host) to f_d(cuda,device)
	hipMemcpy(f_d,F_h,nx*ny*sizeof(float),hipMemcpyHostToDevice);
	
	int bs=BS; // Number of threads, 16 or 32
	dim3 blocks(nx/bs,ny/bs,1); //nx*ny = blocks * threads
	dim3 threads(bs,bs,1);      //bs*bs*1 <= 1024
	
	//----- ----- ----- -----start:(This part is not really needed.)----- ----- ----- ----
	//Set recording time
	hipEvent_t start, stop;
	
	//Initialization
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//Start recording time
	hipEventRecord(start);
	//----- ----- ----- -----end:(This part is not really needed.)----- ----- ----- ----
	
	for(int istep=0; istep<=nstep ; istep++){
		//calculate subroutine "Kernel" on GPU
		Kernel<<<blocks, threads>>>(f_d,fn_d,nx,ny,rr,temp,L0,kapa_c,Da,Db,dt,dx,dy);
		hipDeviceSynchronize(); //<- new version | old version -> hipDeviceSynchronize();
		
		// replace f_d with new f_d (=fn_d)
		update(&f_d,&fn_d);
		
		if(istep%nprint == 0){
			//copy f_d(cuda,device) to F_h(cpu,host)
			hipMemcpy(F_h,f_d,nx*ny*sizeof(float),hipMemcpyDeviceToHost);
			
			//output vtk format
			write_vtk_grid_values_2D(nx,ny,dx,dy,istep,F_h);
			
			//show current step
			fprintf(stderr,"istep = %5d \n",istep);
		}
		//
	}
	
	//----- ----- ----- -----start:(This part is not really needed.)----- ----- ----- ----
	//Stop recording time
	hipEventRecord(stop);
	
	//Wait all event
	hipEventSynchronize(stop);
	
	//calculate time. time is [ms] unit.
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	//Show computing time
	printf("Calculation Time = %9.3f [sec] \n",milliseconds*1.0e-03);
	
	//End processing
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//----- ----- ----- -----end:(This part is not really needed.)----- ----- ----- ----
	
	hipFree(f_d);
	hipFree(fn_d);
	
	free(F_h);
	
	return 0;
}
